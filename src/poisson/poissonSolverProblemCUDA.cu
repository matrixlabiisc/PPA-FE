#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2022 The Regents of the University of Michigan and DFT-FE
// authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Gourab Panigrahi
//

#include <poissonSolverProblemCUDA.h>


namespace dftfe
{
  //
  // constructor
  //
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::poissonSolverProblemCUDA(
    const MPI_Comm &mpi_comm)
    : mpi_communicator(mpi_comm)
    , n_mpi_processes(dealii::Utilities::MPI::n_mpi_processes(mpi_comm))
    , this_mpi_process(dealii::Utilities::MPI::this_mpi_process(mpi_comm))
    , pcout(std::cout,
            (dealii::Utilities::MPI::this_mpi_process(mpi_comm) == 0))
  {
    d_isMeanValueConstraintComputed = false;
    d_isGradSmearedChargeRhs        = false;
    d_isStoreSmearedChargeRhs       = false;
    d_isReuseSmearedChargeRhs       = false;
    d_isFastConstraintsInitialized  = false;
    d_rhoValuesPtr                  = NULL;
    d_atomsPtr                      = NULL;
    d_smearedChargeValuesPtr        = NULL;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::clear()
  {
    d_diagonalA.reinit(0);
    d_rhsSmearedCharge.reinit(0);
    d_meanValueConstraintVec.reinit(0);
    d_cellShapeFunctionGradientIntegralFlattened.clear();
    d_isMeanValueConstraintComputed = false;
    d_isGradSmearedChargeRhs        = false;
    d_isStoreSmearedChargeRhs       = false;
    d_isReuseSmearedChargeRhs       = false;
    d_isFastConstraintsInitialized  = false;
    d_rhoValuesPtr                  = NULL;
    d_atomsPtr                      = NULL;
    d_smearedChargeValuesPtr        = NULL;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::reinit(
    const dealii::MatrixFree<3, double> &    matrixFreeData,
    distributedCPUVec<double> &              x,
    const dealii::AffineConstraints<double> &constraintMatrix,
    const unsigned int                       matrixFreeVectorComponent,
    const unsigned int matrixFreeQuadratureComponentRhsDensity,
    const unsigned int matrixFreeQuadratureComponentAX,
    const std::map<dealii::types::global_dof_index, double> &atoms,
    const std::map<dealii::CellId, std::vector<double>> &smearedChargeValues,
    const unsigned int smearedChargeQuadratureId,
    const std::map<dealii::CellId, std::vector<double>> &rhoValues,
    hipblasHandle_t &                                     cublasHandle,
    const bool                                           isComputeDiagonalA,
    const bool         isComputeMeanValueConstraint,
    const bool         smearedNuclearCharges,
    const bool         isRhoValues,
    const bool         isGradSmearedChargeRhs,
    const unsigned int smearedChargeGradientComponentId,
    const bool         storeSmearedChargeRhs,
    const bool         reuseSmearedChargeRhs,
    const bool         reinitializeFastConstraints)
  {
    int this_process;
    MPI_Comm_rank(mpi_communicator, &this_process);
    MPI_Barrier(mpi_communicator);
    double time = MPI_Wtime();

    d_matrixFreeDataPtr = &matrixFreeData;
    d_xPtr              = &x;
    d_xDevice.reinit(d_xPtr->get_partitioner(), 1);
    cudaUtils::copyHostVecToCUDAVec<double>(d_xPtr->begin(),
                                            d_xDevice.begin(),
                                            d_xDevice.locallyOwnedDofsSize());

    d_constraintMatrixPtr       = &constraintMatrix;
    d_matrixFreeVectorComponent = matrixFreeVectorComponent;
    d_matrixFreeQuadratureComponentRhsDensity =
      matrixFreeQuadratureComponentRhsDensity;
    d_matrixFreeQuadratureComponentAX = matrixFreeQuadratureComponentAX;
    d_rhoValuesPtr                    = isRhoValues ? &rhoValues : NULL;
    d_atomsPtr                        = smearedNuclearCharges ? NULL : &atoms;
    d_smearedChargeValuesPtr =
      smearedNuclearCharges ? &smearedChargeValues : NULL;
    d_smearedChargeQuadratureId        = smearedChargeQuadratureId;
    d_isGradSmearedChargeRhs           = isGradSmearedChargeRhs;
    d_smearedChargeGradientComponentId = smearedChargeGradientComponentId;
    d_isStoreSmearedChargeRhs          = storeSmearedChargeRhs;
    d_isReuseSmearedChargeRhs          = reuseSmearedChargeRhs;
    d_cublasHandlePtr                  = &cublasHandle;
    d_nLocalCells                      = d_matrixFreeDataPtr->n_macro_cells();
    d_xLenLocalDof                     = d_xDevice.locallyOwnedDofsSize();
    d_xLen = d_xDevice.locallyOwnedDofsSize() + d_xDevice.ghostFlattenedSize();

    AssertThrow(
      storeSmearedChargeRhs == false || reuseSmearedChargeRhs == false,
      dealii::ExcMessage(
        "DFT-FE Error: both store and reuse smeared charge rhs cannot be true at the same time."));

    if (isComputeMeanValueConstraint)
      {
        computeMeanValueConstraint();
        d_isMeanValueConstraintComputed = true;
      }

    if (isComputeDiagonalA)
      computeDiagonalA();

    if (!d_isFastConstraintsInitialized || reinitializeFastConstraints)
      {
        d_constraintsInfo.initialize(matrixFreeData.get_vector_partitioner(
                                       matrixFreeVectorComponent),
                                     constraintMatrix);

        d_isFastConstraintsInitialized = true;
      }

    constraintsTotalPotentialInfo.initialize(
      d_matrixFreeDataPtr->get_vector_partitioner(d_matrixFreeVectorComponent),
      *d_constraintMatrixPtr);
    constraintsTotalPotentialInfo.precomputeMaps(
      d_matrixFreeDataPtr->get_vector_partitioner(d_matrixFreeVectorComponent),
      d_xPtr->get_partitioner(),
      1);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::copyXfromDeviceToHost()
  {
    cudaUtils::copyCUDAVecToHostVec<double>(d_xDevice.begin(),
                                            d_xPtr->begin(),
                                            d_xLen);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::distributeX()
  {
    constraintsTotalPotentialInfo.distribute(d_xDevice, 1);

    if (d_isMeanValueConstraintComputed)
      meanValueConstraintDistribute(d_xDevice);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  distributedGPUVec<double> &
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::getX()
  {
    return d_xDevice;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::computeRhs(
    distributedCPUVec<double> &rhs)
  {
    dealii::DoFHandler<3>::active_cell_iterator subCellPtr;
    rhs.reinit(*d_xPtr);
    rhs = 0;

    if (d_isStoreSmearedChargeRhs)
      {
        d_rhsSmearedCharge.reinit(*d_xPtr);
        d_rhsSmearedCharge = 0;
      }

    const dealii::DoFHandler<3> &dofHandler =
      d_matrixFreeDataPtr->get_dof_handler(d_matrixFreeVectorComponent);

    const unsigned int     dofs_per_cell = dofHandler.get_fe().dofs_per_cell;
    dealii::Vector<double> elementalRhs(dofs_per_cell);
    std::vector<dealii::types::global_dof_index> local_dof_indices(
      dofs_per_cell);
    typename dealii::DoFHandler<3>::active_cell_iterator
      cell = dofHandler.begin_active(),
      endc = dofHandler.end();


    distributedCPUVec<double> tempvec;
    tempvec.reinit(rhs);
    tempvec = 0.0;
    d_constraintsInfo.distribute(tempvec);

    dealii::FEEvaluation<3, FEOrderElectro, FEOrderElectro + 1> fe_eval(
      *d_matrixFreeDataPtr,
      d_matrixFreeVectorComponent,
      d_matrixFreeQuadratureComponentAX);

    int isPerformStaticCondensation = (tempvec.linfty_norm() > 1e-10) ? 1 : 0;

    MPI_Bcast(&isPerformStaticCondensation, 1, MPI_INT, 0, mpi_communicator);

    if (isPerformStaticCondensation == 1)
      {
        dealii::VectorizedArray<double> quarter =
          dealii::make_vectorized_array(1.0 / (4.0 * M_PI));
        for (unsigned int macrocell = 0;
             macrocell < d_matrixFreeDataPtr->n_macro_cells();
             ++macrocell)
          {
            fe_eval.reinit(macrocell);
            fe_eval.read_dof_values_plain(tempvec);
            fe_eval.evaluate(false, true);
            for (unsigned int q = 0; q < fe_eval.n_q_points; ++q)
              {
                fe_eval.submit_gradient(-quarter * fe_eval.get_gradient(q), q);
              }
            fe_eval.integrate(false, true);
            fe_eval.distribute_local_to_global(rhs);
          }
      }

    // rhs contribution from electronic charge
    if (d_rhoValuesPtr)
      {
        dealii::FEEvaluation<
          3,
          FEOrderElectro,
          C_num1DQuad<C_rhoNodalPolyOrder<FEOrder, FEOrderElectro>()>()>
          fe_eval_density(*d_matrixFreeDataPtr,
                          d_matrixFreeVectorComponent,
                          d_matrixFreeQuadratureComponentRhsDensity);

        dealii::AlignedVector<dealii::VectorizedArray<double>> rhoQuads(
          fe_eval_density.n_q_points, dealii::make_vectorized_array(0.0));
        for (unsigned int macrocell = 0;
             macrocell < d_matrixFreeDataPtr->n_macro_cells();
             ++macrocell)
          {
            fe_eval_density.reinit(macrocell);

            std::fill(rhoQuads.begin(),
                      rhoQuads.end(),
                      dealii::make_vectorized_array(0.0));
            const unsigned int numSubCells =
              d_matrixFreeDataPtr->n_components_filled(macrocell);
            for (unsigned int iSubCell = 0; iSubCell < numSubCells; ++iSubCell)
              {
                subCellPtr = d_matrixFreeDataPtr->get_cell_iterator(
                  macrocell, iSubCell, d_matrixFreeVectorComponent);
                dealii::CellId             subCellId = subCellPtr->id();
                const std::vector<double> &tempVec =
                  d_rhoValuesPtr->find(subCellId)->second;

                for (unsigned int q = 0; q < fe_eval_density.n_q_points; ++q)
                  rhoQuads[q][iSubCell] = tempVec[q];
              }


            for (unsigned int q = 0; q < fe_eval_density.n_q_points; ++q)
              {
                fe_eval_density.submit_value(rhoQuads[q], q);
              }
            fe_eval_density.integrate(true, false);
            fe_eval_density.distribute_local_to_global(rhs);
          }
      }

    // rhs contribution from atomic charge at fem nodes
    if (d_atomsPtr != NULL)
      for (std::map<dealii::types::global_dof_index, double>::const_iterator
             it = (*d_atomsPtr).begin();
           it != (*d_atomsPtr).end();
           ++it)
        {
          std::vector<dealii::AffineConstraints<double>::size_type>
                                 local_dof_indices_origin(1, it->first); // atomic node
          dealii::Vector<double> cell_rhs_origin(1);
          cell_rhs_origin(0) = -(it->second); // atomic charge

          d_constraintMatrixPtr->distribute_local_to_global(
            cell_rhs_origin, local_dof_indices_origin, rhs);
        }
    else if (d_smearedChargeValuesPtr != NULL && !d_isGradSmearedChargeRhs &&
             !d_isReuseSmearedChargeRhs)
      {
        // const unsigned int   num_quad_points_sc =
        // d_matrixFreeDataPtr->get_quadrature(d_smearedChargeQuadratureId).size();

        dealii::FEEvaluation<3, -1> fe_eval_sc(*d_matrixFreeDataPtr,
                                               d_matrixFreeVectorComponent,
                                               d_smearedChargeQuadratureId);

        const unsigned int numQuadPointsSmearedb = fe_eval_sc.n_q_points;

        dealii::AlignedVector<dealii::VectorizedArray<double>> smearedbQuads(
          numQuadPointsSmearedb, dealii::make_vectorized_array(0.0));
        for (unsigned int macrocell = 0;
             macrocell < d_matrixFreeDataPtr->n_macro_cells();
             ++macrocell)
          {
            std::fill(smearedbQuads.begin(),
                      smearedbQuads.end(),
                      dealii::make_vectorized_array(0.0));
            bool               isMacroCellTrivial = true;
            const unsigned int numSubCells =
              d_matrixFreeDataPtr->n_components_filled(macrocell);
            for (unsigned int iSubCell = 0; iSubCell < numSubCells; ++iSubCell)
              {
                subCellPtr = d_matrixFreeDataPtr->get_cell_iterator(
                  macrocell, iSubCell, d_matrixFreeVectorComponent);
                dealii::CellId             subCellId = subCellPtr->id();
                const std::vector<double> &tempVec =
                  d_smearedChargeValuesPtr->find(subCellId)->second;
                if (tempVec.size() == 0)
                  continue;

                for (unsigned int q = 0; q < numQuadPointsSmearedb; ++q)
                  smearedbQuads[q][iSubCell] = tempVec[q];

                isMacroCellTrivial = false;
              }

            if (!isMacroCellTrivial)
              {
                fe_eval_sc.reinit(macrocell);
                for (unsigned int q = 0; q < fe_eval_sc.n_q_points; ++q)
                  {
                    fe_eval_sc.submit_value(smearedbQuads[q], q);
                  }
                fe_eval_sc.integrate(true, false);

                fe_eval_sc.distribute_local_to_global(rhs);

                if (d_isStoreSmearedChargeRhs)
                  fe_eval_sc.distribute_local_to_global(d_rhsSmearedCharge);
              }
          }
      }
    else if (d_smearedChargeValuesPtr != NULL && d_isGradSmearedChargeRhs)
      {
        dealii::FEEvaluation<3, -1> fe_eval_sc2(*d_matrixFreeDataPtr,
                                                d_matrixFreeVectorComponent,
                                                d_smearedChargeQuadratureId);

        const unsigned int numQuadPointsSmearedb = fe_eval_sc2.n_q_points;

        dealii::Tensor<1, 3, dealii::VectorizedArray<double>> zeroTensor;
        for (unsigned int i = 0; i < 3; i++)
          zeroTensor[i] = dealii::make_vectorized_array(0.0);

        dealii::AlignedVector<
          dealii::Tensor<1, 3, dealii::VectorizedArray<double>>>
          smearedbQuads(numQuadPointsSmearedb, zeroTensor);
        for (unsigned int macrocell = 0;
             macrocell < d_matrixFreeDataPtr->n_macro_cells();
             ++macrocell)
          {
            std::fill(smearedbQuads.begin(),
                      smearedbQuads.end(),
                      dealii::make_vectorized_array(0.0));
            bool               isMacroCellTrivial = true;
            const unsigned int numSubCells =
              d_matrixFreeDataPtr->n_components_filled(macrocell);
            for (unsigned int iSubCell = 0; iSubCell < numSubCells; ++iSubCell)
              {
                subCellPtr = d_matrixFreeDataPtr->get_cell_iterator(
                  macrocell, iSubCell, d_matrixFreeVectorComponent);
                dealii::CellId             subCellId = subCellPtr->id();
                const std::vector<double> &tempVec =
                  d_smearedChargeValuesPtr->find(subCellId)->second;
                if (tempVec.size() == 0)
                  continue;

                for (unsigned int q = 0; q < numQuadPointsSmearedb; ++q)
                  smearedbQuads[q][d_smearedChargeGradientComponentId]
                               [iSubCell] = tempVec[q];

                isMacroCellTrivial = false;
              }

            if (!isMacroCellTrivial)
              {
                fe_eval_sc2.reinit(macrocell);
                for (unsigned int q = 0; q < fe_eval_sc2.n_q_points; ++q)
                  {
                    fe_eval_sc2.submit_gradient(smearedbQuads[q], q);
                  }
                fe_eval_sc2.integrate(false, true);
                fe_eval_sc2.distribute_local_to_global(rhs);
              }
          }
      }

    // MPI operation to sync data
    rhs.compress(dealii::VectorOperation::add);

    if (d_isReuseSmearedChargeRhs)
      rhs += d_rhsSmearedCharge;

    if (d_isStoreSmearedChargeRhs)
      d_rhsSmearedCharge.compress(dealii::VectorOperation::add);

    if (d_isMeanValueConstraintComputed)
      meanValueConstraintDistributeSlaveToMaster(rhs);

    // FIXME: check if this is really required
    d_constraintMatrixPtr->set_zero(rhs);
  }


  // Compute and fill value at mean value constrained dof
  // u_o= -\sum_{i \neq o} a_i * u_i where i runs over all dofs
  // except the mean value constrained dof (o^{th})
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::
    meanValueConstraintDistribute(distributedGPUVec<double> &vec) const
  {
    // -\sum_{i \neq o} a_i * u_i computation which involves summation across
    // MPI tasks
    const double constrainedNodeValue =
      cudaUtils::dot(d_meanValueConstraintGPUVec.begin(),
                     vec.begin(),
                     d_xLenLocalDof,
                     mpi_communicator,
                     *d_cublasHandlePtr);

    if (dealii::Utilities::MPI::this_mpi_process(mpi_communicator) ==
        d_meanValueConstraintProcId)
      cudaUtils::set(vec.begin() + d_meanValueConstraintNodeIdLocal,
                     constrainedNodeValue,
                     1);
  }

  // Distribute value at mean value constrained dof (u_o) to all other dofs
  // u_i+= -a_i * u_o, and subsequently set u_o to 0
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::
    meanValueConstraintDistributeSlaveToMaster(
      distributedGPUVec<double> &vec) const
  {
    double constrainedNodeValue = 0;
    if (dealii::Utilities::MPI::this_mpi_process(mpi_communicator) ==
        d_meanValueConstraintProcId)

      cudaUtils::copyCUDAVecToHostVec<double>(
        vec.begin() + d_meanValueConstraintNodeIdLocal,
        &constrainedNodeValue,
        1);

    // broadcast value at mean value constraint to all other tasks ids
    MPI_Bcast(&constrainedNodeValue,
              1,
              MPI_DOUBLE,
              d_meanValueConstraintProcId,
              mpi_communicator);

    cudaUtils::add(vec.begin(),
                   d_meanValueConstraintGPUVec.begin(),
                   constrainedNodeValue,
                   d_xLenLocalDof,
                   *d_cublasHandlePtr);

    // meanValueConstraintSetZero
    if (d_isMeanValueConstraintComputed)
      if (dealii::Utilities::MPI::this_mpi_process(mpi_communicator) ==
          d_meanValueConstraintProcId)
        cudaUtils::set<double>(vec.begin() + d_meanValueConstraintNodeIdLocal,
                               0,
                               1);
  }

  // Distribute value at mean value constrained dof (u_o) to all other dofs
  // u_i+= -a_i * u_o, and subsequently set u_o to 0
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::
    meanValueConstraintDistributeSlaveToMaster(
      distributedCPUVec<double> &vec) const
  {
    double constrainedNodeValue = 0;
    if (dealii::Utilities::MPI::this_mpi_process(mpi_communicator) ==
        d_meanValueConstraintProcId)
      constrainedNodeValue = vec[d_meanValueConstraintNodeIdLocal];

    // broadcast value at mean value constraint to all other tasks ids
    MPI_Bcast(&constrainedNodeValue,
              1,
              MPI_DOUBLE,
              d_meanValueConstraintProcId,
              mpi_communicator);

    vec.add(constrainedNodeValue, d_meanValueConstraintVec);

    meanValueConstraintSetZero(vec);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::meanValueConstraintSetZero(
    distributedCPUVec<double> &vec) const
  {
    if (d_isMeanValueConstraintComputed)
      if (dealii::Utilities::MPI::this_mpi_process(mpi_communicator) ==
          d_meanValueConstraintProcId)
        vec[d_meanValueConstraintNodeIdLocal] = 0;
  }

  //
  // Compute mean value constraint which is required in case of fully periodic
  // boundary conditions
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder,
                           FEOrderElectro>::computeMeanValueConstraint()
  {
    // allocate parallel distibuted vector to store mean value constraint
    d_meanValueConstraintVec.reinit(*d_xPtr);
    d_meanValueConstraintVec = 0;

    // allocate parallel distibuted gpu vector to store mean value constraint
    d_meanValueConstraintGPUVec.reinit(
      d_meanValueConstraintVec.get_partitioner(), 1);

    const dealii::DoFHandler<3> &dofHandler =
      d_matrixFreeDataPtr->get_dof_handler(d_matrixFreeVectorComponent);

    const dealii::Quadrature<3> &quadrature =
      d_matrixFreeDataPtr->get_quadrature(d_matrixFreeQuadratureComponentAX);
    dealii::FEValues<3>    fe_values(dofHandler.get_fe(),
                                  quadrature,
                                  dealii::update_values |
                                    dealii::update_JxW_values);
    const unsigned int     dofs_per_cell   = dofHandler.get_fe().dofs_per_cell;
    const unsigned int     num_quad_points = quadrature.size();
    dealii::Vector<double> elementalValues(dofs_per_cell);
    std::vector<dealii::types::global_dof_index> local_dof_indices(
      dofs_per_cell);

    // parallel loop over all elements
    typename dealii::DoFHandler<3>::active_cell_iterator
      cell = dofHandler.begin_active(),
      endc = dofHandler.end();
    for (; cell != endc; ++cell)
      if (cell->is_locally_owned())
        {
          fe_values.reinit(cell);

          cell->get_dof_indices(local_dof_indices);

          elementalValues = 0.0;
          for (unsigned int i = 0; i < dofs_per_cell; ++i)
            for (unsigned int q_point = 0; q_point < num_quad_points; ++q_point)
              elementalValues(i) +=
                fe_values.shape_value(i, q_point) * fe_values.JxW(q_point);

          d_constraintMatrixPtr->distribute_local_to_global(
            elementalValues, local_dof_indices, d_meanValueConstraintVec);
        }

    // MPI operation to sync data
    d_meanValueConstraintVec.compress(dealii::VectorOperation::add);

    dealii::IndexSet locallyOwnedElements =
      d_meanValueConstraintVec.locally_owned_elements();

    dealii::IndexSet locallyRelevantElements;
    dealii::DoFTools::extract_locally_relevant_dofs(dofHandler,
                                                    locallyRelevantElements);

    // pick mean value constrained node such that it is not part
    // of periodic and hanging node constraint equations (both slave and master
    // node). This is done for simplicity of implementation.
    dealii::IndexSet allIndicesTouchedByConstraints(
      d_meanValueConstraintVec.size());
    std::vector<dealii::types::global_dof_index> tempSet;
    for (dealii::IndexSet::ElementIterator it = locallyRelevantElements.begin();
         it < locallyRelevantElements.end();
         it++)
      if (d_constraintMatrixPtr->is_constrained(*it))
        {
          const dealii::types::global_dof_index lineDof = *it;
          const std::vector<std::pair<dealii::types::global_dof_index, double>>
            *rowData = d_constraintMatrixPtr->get_constraint_entries(lineDof);
          tempSet.push_back(lineDof);
          for (unsigned int j = 0; j < rowData->size(); ++j)
            tempSet.push_back((*rowData)[j].first);
        }

    if (d_atomsPtr)
      for (std::map<dealii::types::global_dof_index, double>::const_iterator
             it = (*d_atomsPtr).begin();
           it != (*d_atomsPtr).end();
           ++it)
        tempSet.push_back(it->first);

    allIndicesTouchedByConstraints.add_indices(tempSet.begin(), tempSet.end());
    locallyOwnedElements.subtract_set(allIndicesTouchedByConstraints);


    const unsigned int localSizeOfPotentialChoices =
      locallyOwnedElements.n_elements();
    const unsigned int totalProcs =
      dealii::Utilities::MPI::n_mpi_processes(mpi_communicator);
    const unsigned int this_mpi_process =
      dealii::Utilities::MPI::this_mpi_process(mpi_communicator);
    std::vector<unsigned int> localSizesOfPotentialChoices(totalProcs, 0);
    MPI_Allgather(&localSizeOfPotentialChoices,
                  1,
                  MPI_UNSIGNED,
                  &localSizesOfPotentialChoices[0],
                  1,
                  MPI_UNSIGNED,
                  mpi_communicator);

    d_meanValueConstraintProcId = 0;
    for (unsigned int iproc = 0; iproc < totalProcs; iproc++)
      {
        if (localSizesOfPotentialChoices[iproc] > 0)
          {
            d_meanValueConstraintProcId = iproc;
            break;
          }
      }

    double valueAtConstraintNode = 0;
    if (this_mpi_process == d_meanValueConstraintProcId)
      {
        AssertThrow(locallyOwnedElements.size() != 0,
                    dealii::ExcMessage(
                      "DFT-FE Error: please contact developers."));
        d_meanValueConstraintNodeId = *locallyOwnedElements.begin();
        AssertThrow(!d_constraintMatrixPtr->is_constrained(
                      d_meanValueConstraintNodeId),
                    dealii::ExcMessage(
                      "DFT-FE Error: Mean value constraint creation bug."));
        valueAtConstraintNode =
          d_meanValueConstraintVec[d_meanValueConstraintNodeId];
      }

    MPI_Bcast(&valueAtConstraintNode,
              1,
              MPI_DOUBLE,
              d_meanValueConstraintProcId,
              mpi_communicator);

    d_meanValueConstraintVec /= -valueAtConstraintNode;

    if (this_mpi_process == d_meanValueConstraintProcId)
      d_meanValueConstraintVec[d_meanValueConstraintNodeId] = 0;

    d_meanValueConstraintNodeIdLocal =
      d_meanValueConstraintVec.get_partitioner()->global_to_local(
        d_meanValueConstraintNodeId);
    cudaUtils::copyHostVecToCUDAVec<double>(d_meanValueConstraintVec.begin(),
                                            d_meanValueConstraintGPUVec.begin(),
                                            d_xLenLocalDof);
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::computeDiagonalA()
  {
    d_diagonalA.reinit(*d_xPtr);
    d_diagonalA = 0;

    const dealii::DoFHandler<3> &dofHandler =
      d_matrixFreeDataPtr->get_dof_handler(d_matrixFreeVectorComponent);

    const dealii::Quadrature<3> &quadrature =
      d_matrixFreeDataPtr->get_quadrature(d_matrixFreeQuadratureComponentAX);
    dealii::FEValues<3>    fe_values(dofHandler.get_fe(),
                                  quadrature,
                                  dealii::update_values |
                                    dealii::update_gradients |
                                    dealii::update_JxW_values);
    const unsigned int     dofs_per_cell   = dofHandler.get_fe().dofs_per_cell;
    const unsigned int     num_quad_points = quadrature.size();
    dealii::Vector<double> elementalDiagonalA(dofs_per_cell);
    std::vector<dealii::types::global_dof_index> local_dof_indices(
      dofs_per_cell);

    // parallel loop over all elements
    typename dealii::DoFHandler<3>::active_cell_iterator
      cell = dofHandler.begin_active(),
      endc = dofHandler.end();
    for (; cell != endc; ++cell)
      if (cell->is_locally_owned())
        {
          fe_values.reinit(cell);

          cell->get_dof_indices(local_dof_indices);

          elementalDiagonalA = 0.0;
          for (unsigned int i = 0; i < dofs_per_cell; ++i)
            for (unsigned int q_point = 0; q_point < num_quad_points; ++q_point)
              elementalDiagonalA(i) += (1.0 / (4.0 * M_PI)) *
                                       (fe_values.shape_grad(i, q_point) *
                                        fe_values.shape_grad(i, q_point)) *
                                       fe_values.JxW(q_point);

          d_constraintMatrixPtr->distribute_local_to_global(elementalDiagonalA,
                                                            local_dof_indices,
                                                            d_diagonalA);
        }

    // MPI operation to sync data
    d_diagonalA.compress(dealii::VectorOperation::add);

    for (dealii::types::global_dof_index i = 0; i < d_diagonalA.size(); ++i)
      if (d_diagonalA.in_local_range(i))
        if (!d_constraintMatrixPtr->is_constrained(i))
          d_diagonalA(i) = 1.0 / d_diagonalA(i);

    d_diagonalA.compress(dealii::VectorOperation::insert);
    d_diagonalAdevice.reinit(d_diagonalA.get_partitioner(), 1);
    cudaUtils::copyHostVecToCUDAVec<double>(d_diagonalA.begin(),
                                            d_diagonalAdevice.begin(),
                                            d_xLenLocalDof);
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::setX()
  {
    AssertThrow(false, dftUtils::ExcNotImplementedYet());
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  distributedGPUVec<double> &
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::getPreconditioner()
  {
    return d_diagonalAdevice;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::setupMatrixFree()
  {
    constexpr int p   = FEOrderElectro + 1;
    constexpr int dim = 3;

    // shape info helps to obtain reference cell basis function and lex
    // numbering
    const dealii::DoFHandler<3> &dofHandler =
      d_matrixFreeDataPtr->get_dof_handler(d_matrixFreeVectorComponent);
    const int dofs_per_cell = dofHandler.get_fe().dofs_per_cell;

    dealii::internal::MatrixFreeFunctions::ShapeInfo<double> shapeInfo;
    const dealii::Quadrature<3> &                            quadrature =
      d_matrixFreeDataPtr->get_quadrature(d_matrixFreeQuadratureComponentAX);

    int               num_quad_points = std::cbrt(quadrature.size());
    dealii::QGauss<1> quad(num_quad_points);
    shapeInfo.reinit(quad, dofHandler.get_fe());
    std::vector<unsigned int> lexMap3D = shapeInfo.lexicographic_numbering;

    const auto shapeGrad  = shapeInfo.data.front().shape_gradients;
    const auto shapeValue = shapeInfo.data.front().shape_values;

    dealii::FE_Q<1> feCell1D(FEOrderElectro);
    shapeInfo.reinit(quad, feCell1D);
    std::vector<unsigned int> lexMap1D = shapeInfo.lexicographic_numbering;

    std::vector<double> quadWeights(p);
    for (int i = 0; i < p; i++)
      {
        quadWeights[i] = quad.weight(lexMap1D[i]);
      }

    thrust::host_vector<double> spV(p * p), spG(p * p);
    for (int i = 0; i < p; i++)
      {
        for (int j = 0; j < p; j++)
          {
            spV[i + j * p] =
              shapeValue[i * p + lexMap1D[j]] * std::sqrt(quadWeights[j]);
            spG[i + j * p] =
              shapeGrad[i * p + lexMap1D[j]] * std::sqrt(quadWeights[j]);
          }
      }

    dealii::Triangulation<1> reference_cell;
    dealii::GridGenerator::hyper_cube(reference_cell, 0, 1);
    dealii::FEValues<1> fe_values_reference(feCell1D,
                                            quad,
                                            dealii::update_values |
                                              dealii::update_gradients |
                                              dealii::update_JxW_values);
    fe_values_reference.reinit(reference_cell.begin());

    // Map making
    thrust::host_vector<int> map(dofs_per_cell * d_nLocalCells);
    std::vector<dealii::types::global_dof_index> local_dof_globalIndices(
      dofs_per_cell);

    // Lexicographic Map making
    int cellIdx = 0;
    for (const auto &cell : dofHandler.active_cell_iterators())
      {
        if (cell->is_locally_owned())
          {
            cell->get_dof_indices(local_dof_globalIndices);

            for (int dofIdx = 0; dofIdx < dofs_per_cell; dofIdx++)
              {
                dealii::types::global_dof_index globalIdx =
                  local_dof_globalIndices[lexMap3D[dofIdx]];
                int localIdx =
                  d_xPtr->get_partitioner()->global_to_local(globalIdx);
                map[dofIdx + cellIdx * dofs_per_cell] = localIdx;
              }
            cellIdx++;
          }
      }

    // Jacobian
    dealii::QGauss<dim> quadrature_formula(dofHandler.get_fe().degree + 1);
    const int           qPoints = quadrature_formula.size();

    std::vector<dealii::DerivativeForm<1, dim, dim>> inv_jacobians_tensor;
    std::vector<double> detJacobian(d_nLocalCells * qPoints),
      invJac(d_nLocalCells * dim * dim);
    thrust::host_vector<double> jacobianAction(d_nLocalCells * dim * dim);

    dealii::FEValues<dim> fe_values(dofHandler.get_fe(),
                                    quadrature_formula,
                                    dealii::update_inverse_jacobians |
                                      dealii::update_JxW_values |
                                      dealii::update_quadrature_points);

    constexpr double coeffLaplacian = 1.0 / (4.0 * M_PI);

    cellIdx = 0;
    for (const auto &cell : dofHandler.active_cell_iterators())
      {
        if (cell->is_locally_owned())
          {
            fe_values.reinit(cell);
            inv_jacobians_tensor = fe_values.get_inverse_jacobians();

            for (int i = 0; i < dim; i++)
              for (int j = 0; j < dim; j++)
                invJac[j + i * dim + cellIdx * dim * dim] =
                  inv_jacobians_tensor[0][j][i];

            for (int i = 0; i < qPoints; i++)
              detJacobian[i + cellIdx * qPoints] =
                fe_values.JxW(lexMap3D[i]) /
                quadrature_formula.weight(lexMap3D[i]) * coeffLaplacian;

            cellIdx++;
          }
      }

    for (int cellIdx = 0; cellIdx < d_nLocalCells; cellIdx++)
      for (int i = 0; i < dim; i++)
        for (int j = 0; j < dim; j++)
          for (int k = 0; k < dim; k++)
            jacobianAction[j + i * dim + cellIdx * dim * dim] +=
              invJac[i + k * dim + cellIdx * dim * dim] *
              invJac[j + k * dim + cellIdx * dim * dim] *
              detJacobian[cellIdx * qPoints];

    // Construct the device vectors
    d_shapeFunctionValue    = spV;
    d_shapeFunctionGradient = spG;
    d_jacobianAction        = jacobianAction;
    d_map                   = map;

    shapeFunctionValuePtr =
      thrust::raw_pointer_cast(d_shapeFunctionValue.data());
    shapeFunctionGradientPtr =
      thrust::raw_pointer_cast(d_shapeFunctionGradient.data());
    jacobianActionPtr = thrust::raw_pointer_cast(d_jacobianAction.data());
    mapPtr            = thrust::raw_pointer_cast(d_map.data());
  }


  template <typename Type, int M, int N, int K, int vecShared, int dim>
  __global__ void
  computeAXKernel(Type *      V,
                  const Type *U,
                  const Type *P,
                  const Type *D,
                  const Type *J,
                  const int * map,
                  const int   lenU)
  {
    // V = AU
    // gridDim.x = cells;
    // gridDim.y = batch;
    // nVec = vecShared * batch;
    // vecShared -> No of vectors in shared memory
    // sharedT is used to temporarily store sharedX, sharedY or sharedZ

    if (M < 11)
      {
        __shared__ Type sharedX[vecShared * M * N], sharedY[vecShared * M * N],
          sharedZ[vecShared * M * N], sharedT[vecShared * M * N];

        __shared__ Type sharedN[M * K], sharedD[M * K], sharedNT[M * K],
          sharedDT[M * K], sharedJ[dim * dim];

        __shared__ int sharedMap[M * N];

        // Copy Map to shared memory
#pragma unroll
        for (int i = threadIdx.x + threadIdx.y * blockDim.x; i < M * N;
             i += blockDim.x * blockDim.y)
          {
            sharedMap[i] = map[i + blockIdx.x * M * N];
          }

          // Copy Shape Function Values and Gradients to shared memory
#pragma unroll
        for (int i = threadIdx.x + threadIdx.y * blockDim.x; i < M * K;
             i += blockDim.x * blockDim.y)
          {
            sharedN[i]                      = P[i];
            sharedNT[(i / M) + (i % M) * K] = P[i];
            sharedD[i]                      = D[i];
            sharedDT[(i / M) + (i % M) * K] = D[i];
          }

        __syncthreads();

        //////////////////////////////////////////////////////////////
        // First index is the fastest
        // Interpolation combined with Extraction
        // X -> DNNU
        // Y -> NDNU
        // Z -> NNDU

        // 1st GEMM
        // X, Y and Z Directions
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type y[M], x[M], u[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                y[i] = 0.0;
                x[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                u[k] = U[threadIdx.x + sharedMap[j + k * N] * vecShared +
                         blockIdx.y * vecShared * lenU];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    y[i] += sharedDT[i + k * M] * u[k];
                    x[i] += sharedNT[i + k * M] * u[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedY[threadIdx.x + i * vecShared + j * M * vecShared] = y[i];
                sharedX[threadIdx.x + i * vecShared + j * M * vecShared] = x[i];
              }
          }

        __syncthreads();

        // 2nd GEMM
        // Z Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type t[M], y[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                t[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                y[k] = sharedY[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    t[i] += sharedNT[i + k * M] * y[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedT[threadIdx.x + i * vecShared + j * M * vecShared] = t[i];
              }
          }

        __syncthreads();

        // X and Y Directions
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type y[M], z[M], x[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                y[i] = 0.0;
                z[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                x[k] = sharedX[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    y[i] += sharedNT[i + k * M] * x[k];
                    z[i] += sharedDT[i + k * M] * x[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedY[threadIdx.x + i * vecShared + j * M * vecShared] = y[i];
                sharedZ[threadIdx.x + i * vecShared + j * M * vecShared] = z[i];
              }
          }

        __syncthreads();

        // 3rd GEMM
        // X Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type x[M], y[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                x[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                y[k] = sharedY[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    x[i] += sharedDT[i + k * M] * y[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedX[threadIdx.x + i * vecShared + j * M * vecShared] = x[i];
              }
          }

        __syncthreads();

        // Y Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type y[M], z[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                y[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                z[k] = sharedZ[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    y[i] += sharedNT[i + k * M] * z[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedY[threadIdx.x + i * vecShared + j * M * vecShared] = y[i];
              }
          }

        __syncthreads();

        // Z Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type z[M], t[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                z[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                t[k] = sharedT[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    z[i] += sharedNT[i + k * M] * t[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedZ[threadIdx.x + i * vecShared + j * M * vecShared] = z[i];
              }
          }

        __syncthreads();

        //////////////////////////////////////////////////////////////////
        // sharedX, sharedY, sharedZ have the respective gemms of X, Y, Z
        // directions

        // Copy Jacobian Action to shared memory
#pragma unroll
        for (int i = threadIdx.x + threadIdx.y * blockDim.x; i < dim * dim;
             i += blockDim.x * blockDim.y)
          {
            sharedJ[i] = J[i + blockIdx.x * dim * dim];
          }

        __syncthreads();

        // Gemm with Jacobian Action
#pragma unroll
        for (int j = threadIdx.y; j < M * N; j += blockDim.y)
          {
            Type t[3];
            t[0] = sharedX[threadIdx.x + j * vecShared];
            t[1] = sharedY[threadIdx.x + j * vecShared];
            t[2] = sharedZ[threadIdx.x + j * vecShared];

            sharedX[threadIdx.x + j * vecShared] =
              sharedJ[0] * t[0] + sharedJ[1] * t[1] + sharedJ[2] * t[2];
            sharedY[threadIdx.x + j * vecShared] =
              sharedJ[3] * t[0] + sharedJ[4] * t[1] + sharedJ[5] * t[2];
            sharedZ[threadIdx.x + j * vecShared] =
              sharedJ[6] * t[0] + sharedJ[7] * t[1] + sharedJ[8] * t[2];
          }

        __syncthreads();

        // Integration
        // X -> DNNU
        // Y -> NDNU
        // Z -> NNDU

        // 1st GEMM
        // Z Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type t[M], z[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                t[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                z[k] = sharedZ[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    t[i] += sharedD[i + k * M] * z[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedT[threadIdx.x + i * vecShared + j * M * vecShared] = t[i];
              }
          }

        __syncthreads();

        // Y Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type z[M], y[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                z[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                y[k] = sharedY[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    z[i] += sharedN[i + k * M] * y[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedZ[threadIdx.x + i * vecShared + j * M * vecShared] = z[i];
              }
          }

        __syncthreads();

        // X Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type y[M], x[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                y[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                x[k] = sharedX[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    y[i] += sharedN[i + k * M] * x[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedY[threadIdx.x + i * vecShared + j * M * vecShared] = y[i];
              }
          }

        __syncthreads();

        // 2nd GEMM
        // X Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type x[M], y[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                x[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                y[k] = sharedY[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    x[i] += sharedN[i + k * M] * y[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedX[threadIdx.x + i * vecShared + j * M * vecShared] = x[i];
              }
          }

        __syncthreads();

        // Y Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type y[M], z[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                y[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                z[k] = sharedZ[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    y[i] += sharedD[i + k * M] * z[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedY[threadIdx.x + i * vecShared + j * M * vecShared] = y[i];
              }
          }

        __syncthreads();

        // Z Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type y[M], t[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                y[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                t[k] = sharedT[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    y[i] += sharedN[i + k * M] * t[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedY[threadIdx.x + i * vecShared + j * M * vecShared] +=
                  y[i];
              }
          }

        __syncthreads();

        // 3rd GEMM
        // Z Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type t[M], y[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                t[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                y[k] = sharedY[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    t[i] += sharedN[i + k * M] * y[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedT[threadIdx.x + i * vecShared + j * M * vecShared] = t[i];
              }
          }

        __syncthreads();

        // X Direction
        for (int j = threadIdx.y; j < N; j += blockDim.y)
          {
            Type t[M], x[K];

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                t[i] = 0.0;
              }

            for (int k = 0; k < K; k++)
              {
                x[k] = sharedX[threadIdx.x + j * vecShared + k * N * vecShared];

#pragma unroll
                for (int i = 0; i < M; i++)
                  {
                    t[i] += sharedD[i + k * M] * x[k];
                  }
              }

#pragma unroll
            for (int i = 0; i < M; i++)
              {
                sharedT[threadIdx.x + i * vecShared + j * M * vecShared] +=
                  t[i];
              }
          }

        __syncthreads();

#pragma unroll
        for (int j = threadIdx.y; j < M * N; j += blockDim.y)
          {
            atomicAdd(&V[threadIdx.x + sharedMap[j] * vecShared +
                         blockIdx.y * vecShared * lenU],
                      sharedT[threadIdx.x + j * vecShared]);
          }
      }
  }


  // computeAX
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  poissonSolverProblemCUDA<FEOrder, FEOrderElectro>::computeAX(
    distributedGPUVec<double> &Ax,
    distributedGPUVec<double> &x)
  {
    constexpr int d_nVec = 1;
    constexpr int d_dim  = 3;
    constexpr int d_p    = FEOrderElectro + 1;

    constexpr int d_vecShared =
      (d_nVec < 4 ? 1 : FEOrderElectro < 7 ? 4 : FEOrderElectro == 7 ? 5 : 1);
    constexpr int d_yThreads =
      (d_nVec < 4 ? (FEOrderElectro == 8 ? 192 : 128) :
                    FEOrderElectro < 7 ? 96 : FEOrderElectro == 7 ? 128 : 160);
    constexpr int batch = d_nVec / d_vecShared;

    dim3 blocks(d_nLocalCells, batch, 1);
    dim3 threads(d_vecShared, d_yThreads, 1);

    if (d_isMeanValueConstraintComputed)
      meanValueConstraintDistribute(x);

    x.updateGhostValues();

    constraintsTotalPotentialInfo.distribute(x, 1);

    computeAXKernel<double, d_p, d_p * d_p, d_p, d_vecShared, d_dim>
      <<<blocks, threads>>>(Ax.begin(),
                            x.begin(),
                            shapeFunctionValuePtr,
                            shapeFunctionGradientPtr,
                            jacobianActionPtr,
                            mapPtr,
                            d_xLen);

    constraintsTotalPotentialInfo.set_zero(x, 1);

    constraintsTotalPotentialInfo.distribute_slave_to_master(Ax, 1);

    Ax.compressAdd();

    if (d_isMeanValueConstraintComputed)
      meanValueConstraintDistributeSlaveToMaster(Ax);
  }

#include "poissonSolverProblemCUDA.inst.cu"
} // namespace dftfe
