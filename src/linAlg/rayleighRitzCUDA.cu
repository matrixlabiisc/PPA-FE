// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2018 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
// @author Sambit Das



#include<linearAlgebraOperationsInternalCUDA.h>
#include<linearAlgebraOperationsCUDA.h>
#include<dftParameters.h>
#include<dftUtils.h>

namespace dftfe
{
  namespace linearAlgebraOperationsCUDA
  {
 

    void rayleighRitzSpectrumSplitDirect(operatorDFTCUDAClass & operatorMatrix,
		      const double* X,
                      double* XFrac,
                      cudaVectorType & Xb,
                      cudaVectorType & HXb,
                      cudaVectorType & projectorKetTimesVector,
		      const unsigned int M,
                      const unsigned int N,
                      const unsigned int Noc,
                      const bool isElpaStep1,
                      const bool isElpaStep2,
		      const MPI_Comm &mpiComm,
		      double* eigenValues,
                      hipblasHandle_t & handle,
                      dealii::ScaLAPACKMatrix<double> & projHamPar,
                      const std::shared_ptr< const dealii::Utilities::MPI::ProcessGrid> & processGrid,
                      const bool useMixedPrecOverall)
    {

          int this_process;
          MPI_Comm_rank(mpiComm, &this_process);

          const unsigned int Nfr = N-Noc;

          double gpu_time;

          if (!isElpaStep2)
          {
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime();
                      }

		      if (processGrid->is_process_active())
			  std::fill(&projHamPar.local_el(0,0),
				    &projHamPar.local_el(0,0)+projHamPar.local_m()*projHamPar.local_n(),
				    0.0);

                      if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
			      operatorMatrix.XtHXMixedPrec(X,
						 Xb,
						 HXb,
						 projectorKetTimesVector,
						 M,
						 N,
                                                 Noc,
						 handle,
						 processGrid,
						 projHamPar);
                      else
                      {
                              if (dftParameters::overlapComputeCommunXtHX)
				      operatorMatrix.XtHXOverlapComputeCommun(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                              else
				      operatorMatrix.XtHX(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                      }

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
                        {
                          if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
                            std::cout<<"Time for Blocked XtHX Mixed Prec, RR step: "<<gpu_time<<std::endl;
                          else
  			     std::cout<<"Time for Blocked XtHX, RR step: "<<gpu_time<<std::endl;
                        }
		      }
          }
               
          if (isElpaStep1)
               return;

          if (!isElpaStep2)
          {
		      //
		      //compute eigendecomposition of ProjHam
		      //
		      std::vector<double> eigenValuesStdVec(Nfr,0.0);
		      
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                         hipDeviceSynchronize();
                         MPI_Barrier(MPI_COMM_WORLD);
			 gpu_time = MPI_Wtime();
                      }

		      eigenValuesStdVec=projHamPar.eigenpairs_symmetric_by_index_MRRR(std::make_pair(Noc,N-1),true);
		      std::copy(eigenValuesStdVec.begin(),eigenValuesStdVec.end(),eigenValues);

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
			  std::cout<<"Time for ScaLAPACK eigen decomp, RR step: "<<gpu_time<<std::endl;
		      }
          }
             
	  //
	  //rotate the basis in the subspace Xfr = Xfr*Q, implemented as Xfr^{T}=Q^{T}*Xfr^{T} with Xfr^{T}
	  //stored in the column major format
	  //
          if (dftParameters::gpuFineGrainedTimings)
          {
              hipDeviceSynchronize();
              MPI_Barrier(MPI_COMM_WORLD);
	      gpu_time = MPI_Wtime();
          }

          subspaceRotationSpectrumSplitScalapack(X,
                    XFrac,
		    M,
		    N,
                    Nfr,
		    handle,
		    processGrid,
		    mpiComm,
		    projHamPar,
		    true);

          if (dftParameters::gpuFineGrainedTimings)
          {
	      hipDeviceSynchronize();
              MPI_Barrier(MPI_COMM_WORLD);
	      gpu_time = MPI_Wtime() - gpu_time;

	      if (this_process==0)
	        std::cout<<"Time for Blocked subspace rotation, RR step: "<<gpu_time<<std::endl;
          }
    }


    void rayleighRitz(operatorDFTCUDAClass & operatorMatrix,
		      double* X,
                      cudaVectorType & Xb,
                      cudaVectorType & HXb,
                      cudaVectorType & projectorKetTimesVector,
		      const unsigned int M,
                      const unsigned int N,
                      const bool isElpaStep1,
                      const bool isElpaStep2,
		      const MPI_Comm &mpiComm,
                      const MPI_Comm &interBandGroupComm,
		      double* eigenValues,
                      hipblasHandle_t & handle,
                      dealii::ScaLAPACKMatrix<double> & projHamPar,
                      const std::shared_ptr< const dealii::Utilities::MPI::ProcessGrid> & processGrid,
                      const bool useMixedPrecOverall)
    {

              int this_process;
              MPI_Comm_rank(mpiComm, &this_process);

              double gpu_time;

              if (!isElpaStep2)
              {
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime();
                      }

		      if (processGrid->is_process_active())
			  std::fill(&projHamPar.local_el(0,0),
				    &projHamPar.local_el(0,0)+projHamPar.local_m()*projHamPar.local_n(),
				    0.0);

                      if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
			      operatorMatrix.XtHXMixedPrec(X,
						 Xb,
						 HXb,
						 projectorKetTimesVector,
						 M,
						 N,
                                                 N-dftParameters::mixedPrecXtHXFracStates,
						 handle,
						 processGrid,
						 projHamPar);
                      else
                      {
                              if (dftParameters::overlapComputeCommunXtHX)
				      operatorMatrix.XtHXOverlapComputeCommun(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                              else
				      operatorMatrix.XtHX(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                      }

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
                        {
                          if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
                            std::cout<<"Time for Blocked XtHX Mixed Prec, RR step: "<<gpu_time<<std::endl;
                          else
  			     std::cout<<"Time for Blocked XtHX, RR step: "<<gpu_time<<std::endl;
                        }
		      }
              }
               
              if (isElpaStep1)
                 return;

              if (!isElpaStep2)
              {
		      //
		      //compute eigendecomposition of ProjHam
		      //
		      const unsigned int numberEigenValues = N;
		      std::vector<double> eigenValuesStdVec(numberEigenValues,0.0);
		      
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                         hipDeviceSynchronize();
                         MPI_Barrier(MPI_COMM_WORLD);
			 gpu_time = MPI_Wtime();
                      }

		      eigenValuesStdVec=projHamPar.eigenpairs_symmetric_by_index_MRRR(std::make_pair(0,numberEigenValues-1),true);
		      std::copy(eigenValuesStdVec.begin(),eigenValuesStdVec.end(),eigenValues);

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
			  std::cout<<"Time for ScaLAPACK eigen decomp, RR step: "<<gpu_time<<std::endl;
		      }
              }
             
	      //
	      //rotate the basis in the subspace X = X*Q, implemented as X^{T}=Q^{T}*X^{T} with X^{T}
	      //stored in the column major format
	      //
              if (dftParameters::gpuFineGrainedTimings)
              {
                 hipDeviceSynchronize();
                 MPI_Barrier(MPI_COMM_WORLD);
	         gpu_time = MPI_Wtime();
              }

              if (useMixedPrecOverall && dftParameters::useMixedPrecSubspaceRotRR)
                 subspaceRotationRRMixedPrecScalapack(X,
                            M,
                            N,
                            handle,
                            processGrid,
                            mpiComm,
                            interBandGroupComm,
                            projHamPar,
                            true);
              else
                 subspaceRotationScalapack(X,
                            M,
                            N,
                            handle,
                            processGrid,
                            mpiComm,
                            interBandGroupComm,
                            projHamPar,
                            true);

              if (dftParameters::gpuFineGrainedTimings)
              {
                 hipDeviceSynchronize();
                 MPI_Barrier(MPI_COMM_WORLD);                 
                 gpu_time = MPI_Wtime() - gpu_time;

                 if (this_process==0)
                  if (useMixedPrecOverall && dftParameters::useMixedPrecSubspaceRotRR)
                     std::cout<<"Time for Blocked subspace rotation Mixed Prec, RR step: "<<gpu_time<<std::endl;
                  else
                     std::cout<<"Time for Blocked subspace rotation, RR step: "<<gpu_time<<std::endl;
              }

    }

    void rayleighRitzGEP(operatorDFTCUDAClass & operatorMatrix,
		      double* X,
                      cudaVectorType & Xb,
                      cudaVectorType & HXb,
                      cudaVectorType & projectorKetTimesVector,
		      const unsigned int M,
                      const unsigned int N,
                      const bool isElpaStep1,
                      const bool isElpaStep2,
		      const MPI_Comm &mpiComm,
                      const MPI_Comm &interBandGroupComm,
		      double* eigenValues,
                      hipblasHandle_t & handle,
                      dealii::ScaLAPACKMatrix<double> & projHamPar,
                      dealii::ScaLAPACKMatrix<double> & overlapMatPar,
                      const std::shared_ptr< const dealii::Utilities::MPI::ProcessGrid> & processGrid,
                      const bool useMixedPrecOverall)
    {

              int this_process;
              MPI_Comm_rank(MPI_COMM_WORLD, &this_process);

              double gpu_time;

	      const unsigned int rowsBlockSize=operatorMatrix.getScalapackBlockSize();

              if (!isElpaStep2)
              {
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                         hipDeviceSynchronize();
                         MPI_Barrier(MPI_COMM_WORLD);
		         gpu_time = MPI_Wtime();
                      }

		      //S=X*X^{T}. Implemented as S=X^{T}*X with X^{T} stored in the column major format
		      if (dftParameters::useMixedPrecPGS_O && useMixedPrecOverall)
				     linearAlgebraOperationsCUDA::
					     fillParallelOverlapMatMixedPrecScalapack
							      (X,
							       M,
							       N,
							       handle,
							       mpiComm,
                                                               interBandGroupComm,
							       processGrid,
							       overlapMatPar);

		      else
				     linearAlgebraOperationsCUDA::
					     fillParallelOverlapMatScalapack
							      (X,
							       M,
							       N,
							       handle,
							       mpiComm,
                                                               interBandGroupComm,
							       processGrid,
							       overlapMatPar); 
			    
		      if (dftParameters::gpuFineGrainedTimings)
		      { 
                            hipDeviceSynchronize();
                            MPI_Barrier(MPI_COMM_WORLD);
			    gpu_time = MPI_Wtime() - gpu_time;
			    if (this_process==0)
			    {
			      if (dftParameters::useMixedPrecPGS_O && useMixedPrecOverall)
				  std::cout<<"Time for X^{T}X Mixed Prec, RR GEP step: "<<gpu_time<<std::endl;
			      else
				  std::cout<<"Time for X^{T}X, RR GEP step: "<<gpu_time<<std::endl;
			    }
		      }

		      if (dftParameters::gpuFineGrainedTimings)
                      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime();
                      }

		      if (processGrid->is_process_active())
			  std::fill(&projHamPar.local_el(0,0),
				    &projHamPar.local_el(0,0)+projHamPar.local_m()*projHamPar.local_n(),
				    0.0);

                      if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
			      operatorMatrix.XtHXMixedPrec(X,
						 Xb,
						 HXb,
						 projectorKetTimesVector,
						 M,
						 N,
                                                 N-dftParameters::mixedPrecXtHXFracStates,
						 handle,
						 processGrid,
						 projHamPar);
                      else
                      {
                              if (dftParameters::overlapComputeCommunXtHX)
				      operatorMatrix.XtHXOverlapComputeCommun(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                              else
				      operatorMatrix.XtHX(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                      }

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
                        {
                          if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
                            std::cout<<"Time for X^{T}HX Mixed Prec, RR GEP step: "<<gpu_time<<std::endl;
                          else
  			     std::cout<<"Time for X^{T}HX, RR GEP step: "<<gpu_time<<std::endl;
                        }
		      }
              }
               
              if (isElpaStep1)
                 return;

              if (!isElpaStep2)
              {

                      
		      //S=L*L^{T}
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                         hipDeviceSynchronize();
                         MPI_Barrier(MPI_COMM_WORLD);
			 gpu_time = MPI_Wtime();
                      }

                      overlapMatPar.compute_cholesky_factorization();
                      dealii::LAPACKSupport::Property overlapMatPropertyPostCholesky=overlapMatPar.get_property();

		      AssertThrow(overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular
				  ||overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::upper_triangular
				   ,dealii::ExcMessage("DFT-FE Error: overlap matrix property after cholesky factorization incorrect"));
		      dealii::ScaLAPACKMatrix<double> LMatPar(N,
							 processGrid,
							 rowsBlockSize,
							 overlapMatPropertyPostCholesky); 

		      //copy triangular part of projHamPar into LMatPar
		      if (processGrid->is_process_active())
			 for (unsigned int i = 0; i < overlapMatPar.local_n(); ++i)
			   {
			     const unsigned int glob_i = overlapMatPar.global_column(i);
			     for (unsigned int j = 0; j < overlapMatPar.local_m(); ++j)
			       {
				 const unsigned int glob_j = overlapMatPar.global_row(j);
				 if (overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular)
				 {
				     if (glob_i <= glob_j)
					LMatPar.local_el(j, i)=overlapMatPar.local_el(j, i);
				     else
					LMatPar.local_el(j, i)=0;
				 }
				 else
				 {
				     if (glob_j <= glob_i)
					LMatPar.local_el(j, i)=overlapMatPar.local_el(j, i);
				     else
					LMatPar.local_el(j, i)=0;
				 }
			       }
			   }

                      // invert triangular matrix
                      LMatPar.invert();

		      //
		      //compute projected Hamiltonian
		      //
		      dealii::ScaLAPACKMatrix<double> projHamParTrans(N,
							    processGrid,
							    rowsBlockSize);

		      if (processGrid->is_process_active())
			  std::fill(&projHamParTrans.local_el(0,0),
				    &projHamParTrans.local_el(0,0)+projHamParTrans.local_m()*projHamParTrans.local_n(),
				    0.0);


		      projHamParTrans.copy_transposed(projHamPar);
		      projHamPar.add(projHamParTrans,1.0,1.0);

		      if (processGrid->is_process_active())
			 for (unsigned int i = 0; i < projHamPar.local_n(); ++i)
			   {
			     const unsigned int glob_i = projHamPar.global_column(i);
			     for (unsigned int j = 0; j < projHamPar.local_m(); ++j)
			       {
				 const unsigned int glob_j = projHamPar.global_row(j);
				 if (glob_i==glob_j)
				    projHamPar.local_el(j, i)*=0.5;
			       }
			   }

		      dealii::ScaLAPACKMatrix<double> projHamParCopy(N,
							    processGrid,
							    rowsBlockSize);

		      if (overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular)
		      {
			  LMatPar.mmult(projHamParCopy,projHamPar);
			  projHamParCopy.mTmult(projHamPar,LMatPar);
		      }
		      else
		      {
			  LMatPar.Tmmult(projHamParCopy,projHamPar);
			  projHamParCopy.mmult(projHamPar,LMatPar);
		      }

		      //
		      //compute eigendecomposition of ProjHam
		      //
		      const unsigned int numberEigenValues = N;
		      std::vector<double> eigenValuesStdVec(numberEigenValues,0.0);
		      
		      eigenValuesStdVec=projHamPar.eigenpairs_symmetric_by_index_MRRR(std::make_pair(0,numberEigenValues-1),true);
		      std::copy(eigenValuesStdVec.begin(),eigenValuesStdVec.end(),eigenValues);

		      projHamPar.copy_to(projHamParCopy);
		      if (overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular)
			LMatPar.Tmmult(projHamPar,projHamParCopy);
		      else
			LMatPar.mmult(projHamPar,projHamParCopy);

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
			  std::cout<<"Time for ScaLAPACK GEP eigen decomp, RR GEP step: "<<gpu_time<<std::endl;
		      }
              }
             
	      //
	      //rotate the basis in the subspace X = X*L_{inv}^{T}*Q implemented as X^{T}=Q^{T}*L^{-1}*X^{T}
	      //stored in the column major format
	      //
              if (dftParameters::gpuFineGrainedTimings)
              {
                 hipDeviceSynchronize();
                 MPI_Barrier(MPI_COMM_WORLD);
	         gpu_time = MPI_Wtime();
              }

              if (useMixedPrecOverall && dftParameters::useMixedPrecSubspaceRotRR)
                 subspaceRotationRRMixedPrecScalapack(X,
                            M,
                            N,
                            handle,
                            processGrid,
                            mpiComm,
                            interBandGroupComm,
                            projHamPar,
                            true);
              else
                 subspaceRotationScalapack(X,
                            M,
                            N,
                            handle,
                            processGrid,
                            mpiComm,
                            interBandGroupComm,
                            projHamPar,
                            true);

              if (dftParameters::gpuFineGrainedTimings)
              {
                 hipDeviceSynchronize();
                 gpu_time = MPI_Wtime() - gpu_time;

                 if (this_process==0)
                  if (useMixedPrecOverall && dftParameters::useMixedPrecSubspaceRotRR)
                     std::cout<<"Time for X = X*L_{inv}^{T}*Q mixed prec, RR GEP step: "<<gpu_time<<std::endl;
                  else
                     std::cout<<"Time for X = X*L_{inv}^{T}*Q, RR GEP step: "<<gpu_time<<std::endl;
              }

    }

    void rayleighRitzGEPSpectrumSplitDirect(operatorDFTCUDAClass & operatorMatrix,
		      double* X,
                      double* XFrac,
                      cudaVectorType & Xb,
                      cudaVectorType & HXb,
                      cudaVectorType & projectorKetTimesVector,
		      const unsigned int M,
                      const unsigned int N,
                      const unsigned int Noc,
                      const bool isElpaStep1,
                      const bool isElpaStep2,
		      const MPI_Comm &mpiComm,
                      const MPI_Comm &interBandGroupComm,
		      double* eigenValues,
                      hipblasHandle_t & handle,
                      dealii::ScaLAPACKMatrix<double> & projHamPar,
                      dealii::ScaLAPACKMatrix<double> & overlapMatPar,
                      const std::shared_ptr< const dealii::Utilities::MPI::ProcessGrid> & processGrid,
                      const bool useMixedPrecOverall)
    {

              int this_process;
              MPI_Comm_rank(MPI_COMM_WORLD, &this_process);

              const unsigned int Nfr=N-Noc;

              double gpu_time;

	      const unsigned int rowsBlockSize=operatorMatrix.getScalapackBlockSize();

              if (!isElpaStep2)
              {
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                         hipDeviceSynchronize();
                         MPI_Barrier(MPI_COMM_WORLD);
		         gpu_time = MPI_Wtime();
                      }

		      //S=X*X^{T}. Implemented as S=X^{T}*X with X^{T} stored in the column major format
		      if (dftParameters::useMixedPrecPGS_O && useMixedPrecOverall)
				     linearAlgebraOperationsCUDA::
					     fillParallelOverlapMatMixedPrecScalapack
							      (X,
							       M,
							       N,
							       handle,
							       mpiComm,
                                                               interBandGroupComm,
							       processGrid,
							       overlapMatPar);

		      else
				     linearAlgebraOperationsCUDA::
					     fillParallelOverlapMatScalapack
							      (X,
							       M,
							       N,
							       handle,
							       mpiComm,
                                                               interBandGroupComm,
							       processGrid,
							       overlapMatPar); 
			    
		      if (dftParameters::gpuFineGrainedTimings)
		      { 
                            hipDeviceSynchronize();
                            MPI_Barrier(MPI_COMM_WORLD);
			    gpu_time = MPI_Wtime() - gpu_time;
			    if (this_process==0)
			    {
			      if (dftParameters::useMixedPrecPGS_O && useMixedPrecOverall)
				  std::cout<<"Time for X^{T}X Mixed Prec, RR GEP step: "<<gpu_time<<std::endl;
			      else
				  std::cout<<"Time for X^{T}X, RR GEP step: "<<gpu_time<<std::endl;
			    }
		      }

		      if (dftParameters::gpuFineGrainedTimings)
                      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime();
                      }

		      if (processGrid->is_process_active())
			  std::fill(&projHamPar.local_el(0,0),
				    &projHamPar.local_el(0,0)+projHamPar.local_m()*projHamPar.local_n(),
				    0.0);

                      if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
			      operatorMatrix.XtHXMixedPrec(X,
						 Xb,
						 HXb,
						 projectorKetTimesVector,
						 M,
						 N,
                                                 Noc,
						 handle,
						 processGrid,
						 projHamPar);
                      else
                      {
                              if (dftParameters::overlapComputeCommunXtHX)
				      operatorMatrix.XtHXOverlapComputeCommun(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                              else
				      operatorMatrix.XtHX(X,
							 Xb,
							 HXb,
							 projectorKetTimesVector,
							 M,
							 N,
							 handle,
							 processGrid,
							 projHamPar);
                      }

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
                        {
                          if (useMixedPrecOverall && dftParameters::useMixedPrecXTHXSpectrumSplit)
                            std::cout<<"Time for X^{T}HX Mixed Prec, RR GEP step: "<<gpu_time<<std::endl;
                          else
  			     std::cout<<"Time for X^{T}HX, RR GEP step: "<<gpu_time<<std::endl;
                        }
		      }
              }
               
              if (isElpaStep1)
                 return;

              
	      dealii::ScaLAPACKMatrix<double> LMatPar(N,
						 processGrid,
						 rowsBlockSize);
              overlapMatPar.copy_to(LMatPar);
              dealii::LAPACKSupport::Property overlapMatPropertyPostCholesky=overlapMatPar.get_property(); 
              if (!isElpaStep2)
              {

                      
		      //
		      //compute eigendecomposition
		      //
		      if (dftParameters::gpuFineGrainedTimings)
                      {
                         hipDeviceSynchronize();
                         MPI_Barrier(MPI_COMM_WORLD);
			 gpu_time = MPI_Wtime();
                      }

                      overlapMatPar.compute_cholesky_factorization();
                      overlapMatPropertyPostCholesky=overlapMatPar.get_property();

		      AssertThrow(overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular
				  ||overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::upper_triangular
				   ,dealii::ExcMessage("DFT-FE Error: overlap matrix property after cholesky factorization incorrect"));
		      LMatPar.set_property(overlapMatPropertyPostCholesky); 

		      //copy triangular part of projHamPar into LMatPar
		      if (processGrid->is_process_active())
			 for (unsigned int i = 0; i < overlapMatPar.local_n(); ++i)
			   {
			     const unsigned int glob_i = overlapMatPar.global_column(i);
			     for (unsigned int j = 0; j < overlapMatPar.local_m(); ++j)
			       {
				 const unsigned int glob_j = overlapMatPar.global_row(j);
				 if (overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular)
				 {
				     if (glob_i <= glob_j)
					LMatPar.local_el(j, i)=overlapMatPar.local_el(j, i);
				     else
					LMatPar.local_el(j, i)=0;
				 }
				 else
				 {
				     if (glob_j <= glob_i)
					LMatPar.local_el(j, i)=overlapMatPar.local_el(j, i);
				     else
					LMatPar.local_el(j, i)=0;
				 }
			       }
			   }
                      LMatPar.invert();

		      dealii::ScaLAPACKMatrix<double> projHamParTrans(N,
							    processGrid,
							    rowsBlockSize);

		      if (processGrid->is_process_active())
			  std::fill(&projHamParTrans.local_el(0,0),
				    &projHamParTrans.local_el(0,0)+projHamParTrans.local_m()*projHamParTrans.local_n(),
				    0.0);


		      projHamParTrans.copy_transposed(projHamPar);
		      projHamPar.add(projHamParTrans,1.0,1.0);

		      if (processGrid->is_process_active())
			 for (unsigned int i = 0; i < projHamPar.local_n(); ++i)
			   {
			     const unsigned int glob_i = projHamPar.global_column(i);
			     for (unsigned int j = 0; j < projHamPar.local_m(); ++j)
			       {
				 const unsigned int glob_j = projHamPar.global_row(j);
				 if (glob_i==glob_j)
				    projHamPar.local_el(j, i)*=0.5;
			       }
			   }

		      dealii::ScaLAPACKMatrix<double> projHamParCopy(N,
							    processGrid,
							    rowsBlockSize);

		      if (overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular)
		      {
			  LMatPar.mmult(projHamParCopy,projHamPar);
			  projHamParCopy.mTmult(projHamPar,LMatPar);
		      }
		      else
		      {
			  LMatPar.Tmmult(projHamParCopy,projHamPar);
			  projHamParCopy.mmult(projHamPar,LMatPar);
		      }

		      std::vector<double> eigenValuesStdVec(Nfr,0.0);
		      
		      eigenValuesStdVec=projHamPar.eigenpairs_symmetric_by_index_MRRR(std::make_pair(Noc,N-1),true);
		      std::copy(eigenValuesStdVec.begin(),eigenValuesStdVec.end(),eigenValues);

		      projHamPar.copy_to(projHamParCopy);
		      if (overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::lower_triangular)
			LMatPar.Tmmult(projHamPar,projHamParCopy);
		      else
			LMatPar.mmult(projHamPar,projHamParCopy);

		      if (dftParameters::gpuFineGrainedTimings)
		      {
                        hipDeviceSynchronize();
                        MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;
			if (this_process==0)
			  std::cout<<"Time for ScaLAPACK eigen decomp, RR GEP step: "<<gpu_time<<std::endl;
		      }
              }
             

              //
              //rotate the basis in the subspace X_{fr}=X*(L^{-1}^{T}*Q_{fr}
              //
	      if (dftParameters::gpuFineGrainedTimings)
	      {
		 hipDeviceSynchronize();
		 MPI_Barrier(MPI_COMM_WORLD);
		 gpu_time = MPI_Wtime();
	      }

              subspaceRotationSpectrumSplitScalapack(X,
			    XFrac,
			    M,
			    N,
			    Nfr,
			    handle,
			    processGrid,
			    mpiComm,
			    projHamPar,
			    true);

	      if (dftParameters::gpuFineGrainedTimings)
	      {
		 hipDeviceSynchronize();
		 MPI_Barrier(MPI_COMM_WORLD);
		 gpu_time = MPI_Wtime() - gpu_time;

		 if (this_process==0)
		      std::cout<<"Time for X_{fr}=X*(L^{-1}^{T}*Q_{fr}), RR GEP step: "<<gpu_time<<std::endl;
	      }

              //
              //X=X*L^{-1}^{T} implemented as X^{T}=L^{-1}*X^{T} with X^{T} stored in the column major format
              //
              if (dftParameters::gpuFineGrainedTimings)
              {
                 hipDeviceSynchronize();
                 MPI_Barrier(MPI_COMM_WORLD);
	         gpu_time = MPI_Wtime();
              }

              if (useMixedPrecOverall && dftParameters::useMixedPrecPGS_SR)
	         subspaceRotationPGSMixedPrecScalapack(X,
			    M,
			    N,
			    handle,
			    processGrid,
			    mpiComm,
			    interBandGroupComm,
			    LMatPar,
			    overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::upper_triangular?true:false);
              else
	         subspaceRotationScalapack(X,
			    M,
			    N,
			    handle,
			    processGrid,
			    mpiComm,
			    interBandGroupComm,
			    LMatPar,
			    overlapMatPropertyPostCholesky==dealii::LAPACKSupport::Property::upper_triangular?true:false,
			    dftParameters::triMatPGSOpt?true:false);

              if (dftParameters::gpuFineGrainedTimings)
              {
                 hipDeviceSynchronize();
                 gpu_time = MPI_Wtime() - gpu_time;

                 if (this_process==0)
                  if (useMixedPrecOverall && dftParameters::useMixedPrecSubspaceRotRR)
                     std::cout<<"Time for X=X*L^{-1}^{T} mixed prec, RR GEP step: "<<gpu_time<<std::endl;
                  else
                     std::cout<<"Time for X=X*L^{-1}^{T}, RR GEP step: "<<gpu_time<<std::endl;
              }



    }


  }
}
