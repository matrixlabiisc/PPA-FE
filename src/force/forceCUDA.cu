#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2020 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das
//

//source file for force related computations

#include <forceCUDA.h>
#include <dftParameters.h>
#include <dftUtils.h>
#include <constants.h>
#include <vectorUtilities.h>

namespace dftfe
{
	namespace forceCUDA
	{

		namespace
		{

			__global__
				void stridedCopyToBlockKernel(const unsigned int BVec,
						const double *xVec,
						const unsigned int M,
						const unsigned int N,
						double * yVec,
						const unsigned int startingXVecId)
				{

					const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
					const unsigned int numberEntries = M*BVec;

					for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
					{
						unsigned int blockIndex = index/BVec;
						unsigned int intraBlockIndex=index-blockIndex*BVec;
						yVec[index]
							=xVec[blockIndex*N+startingXVecId+intraBlockIndex];
					}

				}


			__global__
				void copyCUDAKernel(const unsigned int contiguousBlockSize,
						const unsigned int numContiguousBlocks,
						const double *copyFromVec,
						double *copyToVec,
						const dealii::types::global_dof_index *copyFromVecStartingContiguousBlockIds)
				{

					const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
					const unsigned int numberEntries = numContiguousBlocks*contiguousBlockSize;

					for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
					{
						unsigned int blockIndex = index/contiguousBlockSize;
						unsigned int intraBlockIndex=index-blockIndex*contiguousBlockSize;
						copyToVec[index]
							=copyFromVec[copyFromVecStartingContiguousBlockIds[blockIndex]+intraBlockIndex];
					}

				}


			__global__
				void computeELocWfcEshelbyTensorContributions(const unsigned int contiguousBlockSize,
						const unsigned int numContiguousBlocks,
						const unsigned int numQuads,
						const double * psiQuadValues,
						const double * gradPsiQuadValuesX,
						const double * gradPsiQuadValuesY,
						const double * gradPsiQuadValuesZ,
						const double * eigenValues,
						const double * partialOccupancies,
						double *eshelbyTensor)
				{

					const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
					const unsigned int numberEntries = numContiguousBlocks*contiguousBlockSize;

					for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
					{
						const unsigned int blockIndex = index/contiguousBlockSize;
						const unsigned int intraBlockIndex=index-blockIndex*contiguousBlockSize;
						const unsigned int blockIndex2=blockIndex/6;
						const unsigned int eshelbyIndex=blockIndex-6*blockIndex2;
						const unsigned int cellIndex=blockIndex2/numQuads;
						const unsigned int quadId=blockIndex2-cellIndex*numQuads;
						const unsigned int tempIndex=(cellIndex)*numQuads*contiguousBlockSize+quadId*contiguousBlockSize+intraBlockIndex;
						const double psi=psiQuadValues[tempIndex];
						const double gradPsiX=gradPsiQuadValuesX[tempIndex];
						const double gradPsiY=gradPsiQuadValuesY[tempIndex];
						const double gradPsiZ=gradPsiQuadValuesZ[tempIndex];
						const double eigenValue=eigenValues[intraBlockIndex];
						const double partOcc=partialOccupancies[intraBlockIndex];

						const double identityFactor=partOcc*(gradPsiX*gradPsiX+gradPsiY*gradPsiY+gradPsiZ*gradPsiZ)-2.0*partOcc*eigenValue*psi*psi;

						if (eshelbyIndex==0)
							eshelbyTensor[index]=-2.0*partOcc*gradPsiX*gradPsiX+identityFactor;
						else if (eshelbyIndex==1)
							eshelbyTensor[index]=-2.0*partOcc*gradPsiY*gradPsiX;
						else if (eshelbyIndex==2)
							eshelbyTensor[index]=-2.0*partOcc*gradPsiY*gradPsiY+identityFactor;
						else if (eshelbyIndex==3)
							eshelbyTensor[index]=-2.0*partOcc*gradPsiZ*gradPsiX;
						else if (eshelbyIndex==4)
							eshelbyTensor[index]=-2.0*partOcc*gradPsiZ*gradPsiY;
						else if (eshelbyIndex==5)
							eshelbyTensor[index]=-2.0*partOcc*gradPsiZ*gradPsiZ+identityFactor;
					}

				}


			__global__
				void nlpPsiContractionCUDAKernel(const unsigned int numPsi,
						const unsigned int numQuadsNLP, 
						const unsigned int totalNonTrivialPseudoWfcs,
						const unsigned int startingId,
						const double * projectorKetTimesVectorPar,
						const double * psiQuadValuesNLP,
						const double * partialOccupancies,
						const unsigned int * nonTrivialIdToElemIdMap,
						const unsigned int * projecterKetTimesFlattenedVectorLocalIds,
						double *nlpContractionContribution)
				{

					const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
					const unsigned int numberEntries = totalNonTrivialPseudoWfcs*numQuadsNLP*numPsi;

					for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
					{
						const unsigned int blockIndex = index/numPsi;
						const unsigned int wfcId=index-blockIndex*numPsi;
						unsigned int pseudoWfcId=blockIndex/numQuadsNLP;
						const unsigned int quadId=blockIndex-pseudoWfcId*numQuadsNLP;
						pseudoWfcId+=startingId;
						nlpContractionContribution[index]=partialOccupancies[wfcId]*psiQuadValuesNLP[nonTrivialIdToElemIdMap[pseudoWfcId]*numQuadsNLP*numPsi+quadId*numPsi+wfcId]*projectorKetTimesVectorPar[projecterKetTimesFlattenedVectorLocalIds[pseudoWfcId]*numPsi+wfcId];
					}

				}

		}


		void computeNonLocalProjectorKetTimesPsiTimesVH(operatorDFTCUDAClass & operatorMatrix,
				const double * X,
				const unsigned int startingVecId,
				const unsigned int BVec,
				const unsigned int N,
				double * projectorKetTimesPsiTimesVH)
		{

			distributedGPUVec<double> cudaFlattenedArrayBlock;
			vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
					BVec,
					cudaFlattenedArrayBlock);


			distributedGPUVec<double> projectorKetTimesVector;
			vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
					BVec,
					projectorKetTimesVector);


			const unsigned int M=operatorMatrix.getMatrixFreeData()->get_vector_partitioner()->local_size();
			stridedCopyToBlockKernel<<<(BVec+255)/256*M, 256>>>(BVec,
					X,
					M,
					N,
					cudaFlattenedArrayBlock.begin(),
					startingVecId);
			cudaFlattenedArrayBlock.update_ghost_values();

			(operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
					BVec);

			operatorMatrix.computeNonLocalProjectorKetTimesXTimesV(cudaFlattenedArrayBlock.begin(),
					projectorKetTimesVector,
					BVec);


			const unsigned int totalSize=projectorKetTimesVector.get_partitioner()->n_ghost_indices()+projectorKetTimesVector.local_size();

			hipMemcpy(projectorKetTimesPsiTimesVH,
					projectorKetTimesVector.begin(),
					totalSize*sizeof(double),
					hipMemcpyDeviceToHost);  
		}


		void interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD(operatorDFTCUDAClass & operatorMatrix,
				distributedGPUVec<double> & Xb,
				const unsigned int BVec,
				const unsigned int numCells,
				const unsigned int numQuads,
				const unsigned int numQuadsNLP,
				const unsigned int numNodesPerElement,
				const thrust::device_vector<double> & eigenValuesD,
				const thrust::device_vector<double> & partialOccupanciesD,
				const thrust::device_vector<double> & onesVecD,
				const unsigned int cellsBlockSize,
				thrust::device_vector<double> & psiQuadsFlatD,
				thrust::device_vector<double> & gradPsiQuadsXFlatD,
				thrust::device_vector<double> & gradPsiQuadsYFlatD,
				thrust::device_vector<double> & gradPsiQuadsZFlatD,
				thrust::device_vector<double> & gradPsiQuadsNLPFlatD,
				thrust::device_vector<double> & eshelbyTensorContributionsD,
				thrust::device_vector<double> & eshelbyTensorQuadValuesD)
		{
			thrust::device_vector<double> & cellWaveFunctionMatrix = operatorMatrix.getCellWaveFunctionMatrix();

			copyCUDAKernel<<<(BVec+255)/256*numCells*numNodesPerElement,256>>>
				(BVec,
				 numCells*numNodesPerElement,
				 Xb.begin(),
				 thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				 thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));

			const int blockSize=cellsBlockSize;
			const int numberBlocks=numCells/blockSize;
			const int remBlockSize=numCells-numberBlocks*blockSize;

			thrust::device_vector<double> shapeFunctionGradientValuesNLPReferenceD(blockSize*numQuadsNLP*3*numNodesPerElement,0.0);
			thrust::device_vector<double> shapeFunctionGradientValuesNLPD(blockSize*numQuadsNLP*3*numNodesPerElement,0.0);

      for (unsigned int i=0; i<blockSize;i++)
        thrust::copy(operatorMatrix.getShapeFunctionGradientValuesNLPInverted().begin(), operatorMatrix.getShapeFunctionGradientValuesNLPInverted().end(), shapeFunctionGradientValuesNLPReferenceD.begin()+i*numQuadsNLP*3*numNodesPerElement);
			
      for (int iblock=0; iblock<(numberBlocks+1); iblock++)
			{
				const int currentBlockSize= (iblock==numberBlocks)?remBlockSize:blockSize;
				const int startingId=iblock*blockSize;

				if (currentBlockSize>0)
				{
          double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
          int strideA = BVec*numNodesPerElement;
          int strideB = 0;
          int strideC = BVec*numQuads;

          hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              BVec,
              numQuads,
              numNodesPerElement,
              &scalarCoeffAlpha,
              thrust::raw_pointer_cast(&cellWaveFunctionMatrix[startingId*numNodesPerElement*BVec]),
              BVec,
              strideA,
              thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted())[0]),
              numNodesPerElement,
              strideB,
              &scalarCoeffBeta,
              thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
              BVec,
              strideC,
              currentBlockSize);

          strideB=numNodesPerElement*numQuads;

          hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              BVec,
              numQuads,
              numNodesPerElement,
              &scalarCoeffAlpha,
              thrust::raw_pointer_cast(&cellWaveFunctionMatrix[startingId*numNodesPerElement*BVec]),
              BVec,
              strideA,
              thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted())[startingId*numQuads*numNodesPerElement]),
              numNodesPerElement,
              strideB,
              &scalarCoeffBeta,
              thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
              BVec,
              strideC,
              currentBlockSize);


          hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              BVec,
              numQuads,
              numNodesPerElement,
              &scalarCoeffAlpha,
              thrust::raw_pointer_cast(&cellWaveFunctionMatrix[startingId*numNodesPerElement*BVec]),
              BVec,
              strideA,
              thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted())[startingId*numQuads*numNodesPerElement]),
              numNodesPerElement,
              strideB,
              &scalarCoeffBeta,
              thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
              BVec,
              strideC,
              currentBlockSize);

          hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              BVec,
              numQuads,
              numNodesPerElement,
              &scalarCoeffAlpha,
              thrust::raw_pointer_cast(&cellWaveFunctionMatrix[startingId*numNodesPerElement*BVec]),
              BVec,
              strideA,
              thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted())[startingId*numQuads*numNodesPerElement]),
              numNodesPerElement,
              strideB,
              &scalarCoeffBeta,
              thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
              BVec,
              strideC,
              currentBlockSize);

          //shapeGradRef^T*invJacobian^T
          hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              numNodesPerElement,
              3,
              3,
              &scalarCoeffAlpha,
              thrust::raw_pointer_cast(&shapeFunctionGradientValuesNLPReferenceD[0]),
              numNodesPerElement,
              numNodesPerElement*3,
              thrust::raw_pointer_cast(&(operatorMatrix.getInverseJacobiansNLP())[startingId*numQuadsNLP*3*3]),
              3,
              3*3,
              &scalarCoeffBeta,
              thrust::raw_pointer_cast(&shapeFunctionGradientValuesNLPD[0]),
              numNodesPerElement,
              numNodesPerElement*3,
              currentBlockSize*numQuadsNLP);  
          

          const int strideCNLPGrad = BVec*3*numQuadsNLP;
          const int strideBNLPGrad=numNodesPerElement*3*numQuadsNLP;
          
          hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              BVec,
              3*numQuadsNLP,
              numNodesPerElement,
              &scalarCoeffAlpha,
              thrust::raw_pointer_cast(&cellWaveFunctionMatrix[startingId*numNodesPerElement*BVec]),
              BVec,
              strideA,
              thrust::raw_pointer_cast(&shapeFunctionGradientValuesNLPD[0]),
              numNodesPerElement,
              strideBNLPGrad,
              &scalarCoeffBeta,
              thrust::raw_pointer_cast(&gradPsiQuadsNLPFlatD[startingId*numQuadsNLP*3*BVec]),
              BVec,
              strideCNLPGrad,
              currentBlockSize);
          
					computeELocWfcEshelbyTensorContributions<<<(BVec+255)/256*currentBlockSize*numQuads*6,256>>>
						(BVec,
						 currentBlockSize*numQuads*6,
						 numQuads,
						 thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
						 thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
						 thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
						 thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
						 thrust::raw_pointer_cast(&eigenValuesD[0]),
						 thrust::raw_pointer_cast(&partialOccupanciesD[0]),
						 thrust::raw_pointer_cast(&eshelbyTensorContributionsD[0]));

					scalarCoeffAlpha = 1.0;
					scalarCoeffBeta = 1.0;



					hipblasDgemm(operatorMatrix.getCublasHandle(),
							HIPBLAS_OP_N,
							HIPBLAS_OP_N,
							1,
							currentBlockSize*numQuads*6,
							BVec,
							&scalarCoeffAlpha,
							thrust::raw_pointer_cast(&onesVecD[0]),
							1,
							thrust::raw_pointer_cast(&eshelbyTensorContributionsD[0]),
							BVec,
							&scalarCoeffBeta,
							thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD[startingId*numQuads*6]),
							1);

				}
			}
		}

		void nlpPsiContractionD(operatorDFTCUDAClass & operatorMatrix,
				const thrust::device_vector<double> & gradPsiQuadsNLPD,
				const thrust::device_vector<double> & partialOccupanciesD,
				const thrust::device_vector<double> & onesVecD,
				const double * projectorKetTimesVectorParFlattenedD,
				const thrust::device_vector<unsigned int> & nonTrivialIdToElemIdMapD,
				const thrust::device_vector<unsigned int> & projecterKetTimesFlattenedVectorLocalIdsD,
				const unsigned int numCells, 
				const unsigned int numQuadsNLP,
				const unsigned int numPsi,
				const unsigned int totalNonTrivialPseudoWfcs,
				const unsigned int innerBlockSizeEnlp,
				thrust::device_vector<double> & nlpContractionContributionD,
				thrust::device_vector<double> & projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD)
		{
			const int blockSizeNlp=innerBlockSizeEnlp;
			const int numberBlocksNlp=totalNonTrivialPseudoWfcs/blockSizeNlp;
			const int remBlockSizeNlp=totalNonTrivialPseudoWfcs-numberBlocksNlp*blockSizeNlp;

			double scalarCoeffAlphaNlp = 1.0,scalarCoeffBetaNlp = 1.0;      

			for (int iblocknlp=0; iblocknlp<(numberBlocksNlp+1); iblocknlp++)
			{
				const int currentBlockSizeNlp= (iblocknlp==numberBlocksNlp)?remBlockSizeNlp:blockSizeNlp;
				const int startingIdNlp=iblocknlp*blockSizeNlp;
				if (currentBlockSizeNlp>0)
				{
					nlpPsiContractionCUDAKernel<<<(numPsi+255)/256*numQuadsNLP*3*currentBlockSizeNlp,256>>>
						(numPsi,
						 numQuadsNLP*3,
						 currentBlockSizeNlp,
						 startingIdNlp,
						 projectorKetTimesVectorParFlattenedD,
						 thrust::raw_pointer_cast(&gradPsiQuadsNLPD[0]),
						 thrust::raw_pointer_cast(&partialOccupanciesD[0]),
						 thrust::raw_pointer_cast(&nonTrivialIdToElemIdMapD[0]),
						 thrust::raw_pointer_cast(&projecterKetTimesFlattenedVectorLocalIdsD[0]),
						 thrust::raw_pointer_cast(&nlpContractionContributionD[0]));

					hipblasDgemm(operatorMatrix.getCublasHandle(),
							HIPBLAS_OP_N,
							HIPBLAS_OP_N,
							1,
							currentBlockSizeNlp*numQuadsNLP*3,
							numPsi,
							&scalarCoeffAlphaNlp,
							thrust::raw_pointer_cast(&onesVecD[0]),
							1,
							thrust::raw_pointer_cast(&nlpContractionContributionD[0]),
							numPsi,
							&scalarCoeffBetaNlp,
							thrust::raw_pointer_cast(&projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD[startingIdNlp*numQuadsNLP*3]),
							1);
				}
			}
		}


		void gpuPortedForceKernelsAllD(operatorDFTCUDAClass & operatorMatrix,
				distributedGPUVec<double> & cudaFlattenedArrayBlock,
				distributedGPUVec<double> & projectorKetTimesVectorD,
				const double * X,
				const thrust::device_vector<double> & eigenValuesD,
				const thrust::device_vector<double> & partialOccupanciesD,
				const thrust::device_vector<double> & onesVecD,
				const thrust::device_vector<unsigned int> & nonTrivialIdToElemIdMapD,
				const thrust::device_vector<unsigned int> & projecterKetTimesFlattenedVectorLocalIdsD, 
				const unsigned int startingVecId,
				const unsigned int N,
				const unsigned int numPsi,
				const unsigned int numCells,
				const unsigned int numQuads,
				const unsigned int numQuadsNLP,
				const unsigned int numNodesPerElement,
				const unsigned int totalNonTrivialPseudoWfcs,
				thrust::device_vector<double> & psiQuadsFlatD,
				thrust::device_vector<double> & gradPsiQuadsXFlatD,
				thrust::device_vector<double> & gradPsiQuadsYFlatD,
				thrust::device_vector<double> & gradPsiQuadsZFlatD,
				thrust::device_vector<double> & gradPsiQuadsNLPFlatD,
				thrust::device_vector<double> & eshelbyTensorContributionsD,
				thrust::device_vector<double> & eshelbyTensorQuadValuesD,
				thrust::device_vector<double> & nlpContractionContributionD,
				thrust::device_vector<double> & projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD,
				const unsigned int cellsBlockSize,
				const unsigned int innerBlockSizeEnlp,
				const bool isPsp)
				{

					int this_process;
					MPI_Comm_rank(MPI_COMM_WORLD, &this_process);

					const unsigned int M=operatorMatrix.getMatrixFreeData()->get_vector_partitioner()->local_size();
					stridedCopyToBlockKernel<<<(numPsi+255)/256*M, 256>>>(numPsi,
							X,
							M,
							N,
							cudaFlattenedArrayBlock.begin(),
							startingVecId);
					cudaFlattenedArrayBlock.update_ghost_values();

					(operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
							numPsi);


					//hipDeviceSynchronize();
					//MPI_Barrier(MPI_COMM_WORLD);
					//double kernel1_time = MPI_Wtime();

					interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD(operatorMatrix,
							cudaFlattenedArrayBlock,
							numPsi,
							numCells,
							numQuads,
							numQuadsNLP,
							numNodesPerElement,
							eigenValuesD,
							partialOccupanciesD,
							onesVecD,
							cellsBlockSize,
							psiQuadsFlatD,
							gradPsiQuadsXFlatD,
							gradPsiQuadsYFlatD,
							gradPsiQuadsZFlatD,
				      gradPsiQuadsNLPFlatD,
							eshelbyTensorContributionsD,
							eshelbyTensorQuadValuesD);

					//hipDeviceSynchronize();
					//MPI_Barrier(MPI_COMM_WORLD);
					//kernel1_time = MPI_Wtime() - kernel1_time;

					//if (this_process==0 && dftParameters::verbosity>=5)
					//	 std::cout<<"Time for interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD inside blocked loop: "<<kernel1_time<<std::endl;

					if (isPsp)
					{
						//hipDeviceSynchronize();
						//MPI_Barrier(MPI_COMM_WORLD);
						//double kernel2_time = MPI_Wtime();

						operatorMatrix.computeNonLocalProjectorKetTimesXTimesV(cudaFlattenedArrayBlock.begin(),
								projectorKetTimesVectorD,
								numPsi);

						//hipDeviceSynchronize();
						//MPI_Barrier(MPI_COMM_WORLD);
						//kernel2_time = MPI_Wtime() - kernel2_time;

						//if (this_process==0 && dftParameters::verbosity>=5)
						//  std::cout<<"Time for computeNonLocalProjectorKetTimesXTimesV inside blocked loop: "<<kernel2_time<<std::endl;

						//hipDeviceSynchronize();
						//MPI_Barrier(MPI_COMM_WORLD);
						//double kernel3_time = MPI_Wtime();

						if (totalNonTrivialPseudoWfcs>0)
						{
							nlpPsiContractionD(operatorMatrix,
									gradPsiQuadsNLPFlatD,
									partialOccupanciesD,
									onesVecD,
									projectorKetTimesVectorD.begin(),
									nonTrivialIdToElemIdMapD,
									projecterKetTimesFlattenedVectorLocalIdsD,
									numCells, 
									numQuadsNLP,
									numPsi,
									totalNonTrivialPseudoWfcs,
									innerBlockSizeEnlp,
									nlpContractionContributionD,
									projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD);
						}

						//hipDeviceSynchronize();
						//MPI_Barrier(MPI_COMM_WORLD);
						//kernel3_time = MPI_Wtime() - kernel3_time;

						//if (this_process==0 && dftParameters::verbosity>=5)
						//	 std::cout<<"Time for nlpPsiContractionD inside blocked loop: "<<kernel3_time<<std::endl;
					}
				}

		void gpuPortedForceKernelsAllH(operatorDFTCUDAClass & operatorMatrix,
				const double * X,
				const double * eigenValuesH,
				const double  fermiEnergy,
				const unsigned int * nonTrivialIdToElemIdMapH,
				const unsigned int * projecterKetTimesFlattenedVectorLocalIdsH, 
				const unsigned int N,
				const unsigned int numCells,
				const unsigned int numQuads,
				const unsigned int numQuadsNLP,
				const unsigned int numNodesPerElement,
				const unsigned int totalNonTrivialPseudoWfcs,
				double * eshelbyTensorQuadValuesH,
				double * projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedH,
				const MPI_Comm & interBandGroupComm,
				const bool isPsp)
		{
			//band group parallelization data structures
			const unsigned int numberBandGroups=
				dealii::Utilities::MPI::n_mpi_processes(interBandGroupComm);
			const unsigned int bandGroupTaskId = dealii::Utilities::MPI::this_mpi_process(interBandGroupComm);
			std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
			dftUtils::createBandParallelizationIndices(interBandGroupComm,
					N,
					bandGroupLowHighPlusOneIndices);

			const unsigned int blockSize=std::min(dftParameters::chebyWfcBlockSize,
					bandGroupLowHighPlusOneIndices[1]);

			int this_process;
			MPI_Comm_rank(MPI_COMM_WORLD, &this_process);
			hipDeviceSynchronize();
			MPI_Barrier(MPI_COMM_WORLD);
			double gpu_time=MPI_Wtime();

			distributedGPUVec<double> cudaFlattenedArrayBlock;
			distributedGPUVec<double> projectorKetTimesVectorD;
			vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
					blockSize,
					cudaFlattenedArrayBlock);

      if (isPsp)
        vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
            blockSize,
            projectorKetTimesVectorD);

			hipDeviceSynchronize();
			MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;

			if (this_process==0 && dftParameters::verbosity>=2)
				std::cout<<"Time for creating cuda parallel vectors for force computation: "<<gpu_time<<std::endl;

			gpu_time = MPI_Wtime();

			thrust::device_vector<double> eigenValuesD(blockSize,0.0);
			thrust::device_vector<double> partialOccupanciesD(blockSize,0.0);
			thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD(numCells*numQuads*6,0.0);
				
			thrust::device_vector<double> onesVecD(blockSize,1.0);

			const unsigned int cellsBlockSize=std::min((unsigned int)10,numCells);

			thrust::device_vector<double> psiQuadsFlatD(cellsBlockSize*numQuads*blockSize,0.0);
			thrust::device_vector<double> gradPsiQuadsXFlatD(cellsBlockSize*numQuads*blockSize,0.0);
			thrust::device_vector<double> gradPsiQuadsYFlatD(cellsBlockSize*numQuads*blockSize,0.0);
			thrust::device_vector<double> gradPsiQuadsZFlatD(cellsBlockSize*numQuads*blockSize,0.0);
			thrust::device_vector<double> gradPsiQuadsNLPFlatD(numCells*numQuadsNLP*3*blockSize,0.0);

			thrust::device_vector<double> eshelbyTensorContributionsD(cellsBlockSize*numQuads*blockSize*6,0.0);

		  const unsigned int innerBlockSizeEnlp=std::min((unsigned int)10,totalNonTrivialPseudoWfcs);
			thrust::device_vector<double> nlpContractionContributionD(innerBlockSizeEnlp*numQuadsNLP*3*blockSize,0.0);
			thrust::device_vector<double> projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD;
			thrust::device_vector<unsigned int> projecterKetTimesFlattenedVectorLocalIdsD;
			thrust::device_vector<unsigned int> nonTrivialIdToElemIdMapD;
			if (totalNonTrivialPseudoWfcs>0)
			{
				projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD.resize(totalNonTrivialPseudoWfcs*numQuadsNLP*3,0.0);
				projecterKetTimesFlattenedVectorLocalIdsD.resize(totalNonTrivialPseudoWfcs,0.0);
				nonTrivialIdToElemIdMapD.resize(totalNonTrivialPseudoWfcs,0.0);

				hipMemcpy(thrust::raw_pointer_cast(&nonTrivialIdToElemIdMapD[0]),
						nonTrivialIdToElemIdMapH,
						totalNonTrivialPseudoWfcs*sizeof(unsigned int),
						hipMemcpyHostToDevice);


				hipMemcpy(thrust::raw_pointer_cast(&projecterKetTimesFlattenedVectorLocalIdsD[0]),
						projecterKetTimesFlattenedVectorLocalIdsH,
						totalNonTrivialPseudoWfcs*sizeof(unsigned int),
						hipMemcpyHostToDevice);
			}


			for(unsigned int ivec = 0; ivec < N; ivec+=blockSize)
			{
				if((ivec+blockSize)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
						(ivec+blockSize)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
				{
					std::vector<double> blockedEigenValues(blockSize,0.0);
					std::vector<double> blockedPartialOccupancies(blockSize,0.0);
					for (unsigned int iWave=0; iWave<blockSize;++iWave)
					{
						blockedEigenValues[iWave]=eigenValuesH[ivec+iWave];
						blockedPartialOccupancies[iWave]
							=dftUtils::getPartialOccupancy(blockedEigenValues[iWave],
									fermiEnergy,
									C_kb,
									dftParameters::TVal);

					}



					hipMemcpy(thrust::raw_pointer_cast(&eigenValuesD[0]),
							&blockedEigenValues[0],
							blockSize*sizeof(double),
							hipMemcpyHostToDevice);

					hipMemcpy(thrust::raw_pointer_cast(&partialOccupanciesD[0]),
							&blockedPartialOccupancies[0],
							blockSize*sizeof(double),
							hipMemcpyHostToDevice);

					//hipDeviceSynchronize();
					//MPI_Barrier(MPI_COMM_WORLD);
					//double kernel_time = MPI_Wtime();

					gpuPortedForceKernelsAllD(operatorMatrix,
							cudaFlattenedArrayBlock,
							projectorKetTimesVectorD,
							X,
							eigenValuesD,
							partialOccupanciesD,
							onesVecD,
							nonTrivialIdToElemIdMapD,
							projecterKetTimesFlattenedVectorLocalIdsD,
							ivec,
							N,
							blockSize,
							numCells,
							numQuads,
							numQuadsNLP,
							numNodesPerElement,
							totalNonTrivialPseudoWfcs,
							psiQuadsFlatD,
							gradPsiQuadsXFlatD,
							gradPsiQuadsYFlatD,
							gradPsiQuadsZFlatD,
							gradPsiQuadsNLPFlatD,
							eshelbyTensorContributionsD,
							elocWfcEshelbyTensorQuadValuesD,
							nlpContractionContributionD,
							projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD,
							cellsBlockSize,
							innerBlockSizeEnlp,
							isPsp);

					//hipDeviceSynchronize();
					//MPI_Barrier(MPI_COMM_WORLD);
					//kernel_time = MPI_Wtime() - kernel_time;

					//if (this_process==0 && dftParameters::verbosity>=5)
					//   std::cout<<"Time for force kernels all insided block loop: "<<kernel_time<<std::endl;
				}//band parallelization
			}//ivec loop

			hipMemcpy(eshelbyTensorQuadValuesH,
					thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD[0]),
					numCells*numQuads*6*sizeof(double),
					hipMemcpyDeviceToHost);  


			if (totalNonTrivialPseudoWfcs>0)
      {
				hipMemcpy(projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedH,
						thrust::raw_pointer_cast(&projectorKetTimesPsiTimesVTimesPartOccContractionGradPsiQuadsFlattenedD[0]),
						totalNonTrivialPseudoWfcs*numQuadsNLP*3*sizeof(double),
						hipMemcpyDeviceToHost);
      }
			hipDeviceSynchronize();
			MPI_Barrier(MPI_COMM_WORLD);
			gpu_time = MPI_Wtime() - gpu_time;

			if (this_process==0 && dftParameters::verbosity>=1)
				std::cout<<"Time taken for all gpu kernels force computation: "<<gpu_time<<std::endl;
		}

	}//forceCUDA namespace
}//dftfe namespace
