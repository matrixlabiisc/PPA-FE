#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2020 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das
//

//source file for force related computations

#include <forceCUDA.h>
#include <dftParameters.h>
#include <dftUtils.h>
#include <constants.h>
#include <vectorUtilities.h>

namespace dftfe
{
   namespace forceCUDA
   {

       namespace
       {

         __global__
         void stridedCopyToBlockKernel(const unsigned int BVec,
                            const double *xVec,
                            const unsigned int M,
                            const unsigned int N,
                            double * yVec,
                            const unsigned int startingXVecId)
         {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = M*BVec;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      unsigned int blockIndex = index/BVec;
		      unsigned int intraBlockIndex=index-blockIndex*BVec;
		      yVec[index]
			      =xVec[blockIndex*N+startingXVecId+intraBlockIndex];
		   }

          }


          __global__
          void copyCUDAKernel(const unsigned int contiguousBlockSize,
                            const unsigned int numContiguousBlocks,
                            const double *copyFromVec,
                            double *copyToVec,
                            const dealii::types::global_dof_index *copyFromVecStartingContiguousBlockIds)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = numContiguousBlocks*contiguousBlockSize;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      unsigned int blockIndex = index/contiguousBlockSize;
		      unsigned int intraBlockIndex=index-blockIndex*contiguousBlockSize;
		      copyToVec[index]
			      =copyFromVec[copyFromVecStartingContiguousBlockIds[blockIndex]+intraBlockIndex];
		   }

          }


          __global__
          void computeELocWfcEshelbyTensorContributions(const unsigned int contiguousBlockSize,
                                                        const unsigned int numContiguousBlocks,
                                                        const unsigned int startingCellId,
                                                        const unsigned int numQuads,
					                const double * psiQuadValues,
					                const double * gradPsiQuadValuesX,
					                const double * gradPsiQuadValuesY,
					                const double * gradPsiQuadValuesZ,
					                const double * eigenValues,
					                const double * partialOccupancies,
                                                        double *eshelbyTensor00,
                                                        double *eshelbyTensor10,
                                                        double *eshelbyTensor11,
                                                        double *eshelbyTensor20,
                                                        double *eshelbyTensor21,
                                                        double *eshelbyTensor22)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = numContiguousBlocks*contiguousBlockSize;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      const unsigned int blockIndex = index/contiguousBlockSize;
		      const unsigned int intraBlockIndex=index-blockIndex*contiguousBlockSize;
                      const unsigned int cellIndex=blockIndex/numQuads;
                      const unsigned int quadId=blockIndex-cellIndex*numQuads;
                      const unsigned int tempIndex=(startingCellId+cellIndex)*numQuads*contiguousBlockSize+quadId*contiguousBlockSize+intraBlockIndex;
                      const double psi=psiQuadValues[tempIndex];
                      const double gradPsiX=gradPsiQuadValuesX[tempIndex];
                      const double gradPsiY=gradPsiQuadValuesY[tempIndex];
                      const double gradPsiZ=gradPsiQuadValuesZ[tempIndex];
                      const double eigenValue=eigenValues[intraBlockIndex];
                      const double partOcc=partialOccupancies[intraBlockIndex];

                      const double identityFactor=partOcc*(gradPsiX*gradPsiX+gradPsiY*gradPsiY+gradPsiZ*gradPsiZ)-2.0*partOcc*eigenValue*psi*psi;
		      eshelbyTensor00[index]=-2.0*partOcc*gradPsiX*gradPsiX+identityFactor;
                      eshelbyTensor10[index]=-2.0*partOcc*gradPsiY*gradPsiX;
                      eshelbyTensor11[index]=-2.0*partOcc*gradPsiY*gradPsiY+identityFactor;
                      eshelbyTensor20[index]=-2.0*partOcc*gradPsiZ*gradPsiX;
                      eshelbyTensor21[index]=-2.0*partOcc*gradPsiZ*gradPsiY;
                      eshelbyTensor22[index]=-2.0*partOcc*gradPsiZ*gradPsiZ+identityFactor;
		   }

          }


          __global__
          void nlpPsiContractionCUDAKernel(const unsigned int numPsi,
                                           const unsigned int numQuadsNLP, 
                                           const unsigned int totalNonTrivialPseudoWfcs,
                                           const unsigned int startingId,
					   const double * projectorKetTimesVectorPar,
					   const double * psiQuadValuesNLP,
                                           const double * partialOccupancies,
					   const unsigned int * nonTrivialIdToElemIdMap,
					   const unsigned int * projecterKetTimesFlattenedVectorLocalIds,
					   double *nlpContractionContribution)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = totalNonTrivialPseudoWfcs*numQuadsNLP*numPsi;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      const unsigned int blockIndex = index/numPsi;
		      const unsigned int wfcId=index-blockIndex*numPsi;
                      unsigned int pseudoWfcId=blockIndex/numQuadsNLP;
                      const unsigned int quadId=blockIndex-pseudoWfcId*numQuadsNLP;
                      pseudoWfcId+=startingId;
                      nlpContractionContribution[index]=partialOccupancies[wfcId]*psiQuadValuesNLP[nonTrivialIdToElemIdMap[pseudoWfcId]*numQuadsNLP*numPsi+quadId*numPsi+wfcId]*projectorKetTimesVectorPar[projecterKetTimesFlattenedVectorLocalIds[pseudoWfcId]*numPsi+wfcId];
		   }

          }


          __global__
          void computeRhoGradRhoFromInterpolatedValues(const unsigned int numberEntries,
			    double *rhoCellsWfcContributions,
                            double *gradRhoCellsWfcContributionsX,
                            double *gradRhoCellsWfcContributionsY,
                            double *gradRhoCellsWfcContributionsZ,
                            const bool isEvaluateGradRho)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
                      const double psi=rhoCellsWfcContributions[index];
                      rhoCellsWfcContributions[index]=psi*psi;

                      if (isEvaluateGradRho)
                      {
                        const double gradPsiX=gradRhoCellsWfcContributionsX[index];
                        gradRhoCellsWfcContributionsX[index]=2.0*psi*gradPsiX;
                         
                        const double gradPsiY=gradRhoCellsWfcContributionsY[index];
                        gradRhoCellsWfcContributionsY[index]=2.0*psi*gradPsiY;

                        const double gradPsiZ=gradRhoCellsWfcContributionsZ[index];
                        gradRhoCellsWfcContributionsZ[index]=2.0*psi*gradPsiZ;
                         
                      }
		   }

          }


      }

      void computeELocWfcEshelbyTensorNonPeriodicH(operatorDFTCUDAClass & operatorMatrix,
                                                 const double * psiQuadValuesH,
                                                 const double * gradPsiQuadValuesXH,
                                                 const double * gradPsiQuadValuesYH,
                                                 const double * gradPsiQuadValuesZH,
                                                 const double * eigenValuesH,
                                                 const double * partialOccupanciesH,
                                                 const unsigned int numCells,
                                                 const unsigned int numQuads,
                                                 const unsigned int numPsi,
                                                 double * eshelbyTensorQuadValuesH00,
                                                 double * eshelbyTensorQuadValuesH10,
                                                 double * eshelbyTensorQuadValuesH11,
                                                 double * eshelbyTensorQuadValuesH20,
                                                 double * eshelbyTensorQuadValuesH21,
                                                 double * eshelbyTensorQuadValuesH22)
      { 
           hipDeviceSynchronize(); 
           thrust::device_vector<double> psiQuadValuesD(numCells*numQuads*numPsi,0.0);
           thrust::device_vector<double> gradPsiQuadValuesXD(numCells*numQuads*numPsi,0.0);
           thrust::device_vector<double> gradPsiQuadValuesYD(numCells*numQuads*numPsi,0.0);
           thrust::device_vector<double> gradPsiQuadValuesZD(numCells*numQuads*numPsi,0.0);
           thrust::device_vector<double> eigenValuesD(numPsi,0.0);
           thrust::device_vector<double> partialOccupanciesD(numPsi,0.0);
           thrust::device_vector<double> eshelbyTensorQuadValuesD00(numCells*numQuads,0.0);
           thrust::device_vector<double> eshelbyTensorQuadValuesD10(numCells*numQuads,0.0);
           thrust::device_vector<double> eshelbyTensorQuadValuesD11(numCells*numQuads,0.0);
           thrust::device_vector<double> eshelbyTensorQuadValuesD20(numCells*numQuads,0.0);
           thrust::device_vector<double> eshelbyTensorQuadValuesD21(numCells*numQuads,0.0);
           thrust::device_vector<double> eshelbyTensorQuadValuesD22(numCells*numQuads,0.0);


           hipMemcpy(thrust::raw_pointer_cast(&psiQuadValuesD[0]),
		      psiQuadValuesH,
		      numCells*numQuads*numPsi*sizeof(double),
		      hipMemcpyHostToDevice);

           hipMemcpy(thrust::raw_pointer_cast(&gradPsiQuadValuesXD[0]),
		      gradPsiQuadValuesXH,
		      numCells*numQuads*numPsi*sizeof(double),
		      hipMemcpyHostToDevice);

           hipMemcpy(thrust::raw_pointer_cast(&gradPsiQuadValuesYD[0]),
		      gradPsiQuadValuesYH,
		      numCells*numQuads*numPsi*sizeof(double),
		      hipMemcpyHostToDevice);

           hipMemcpy(thrust::raw_pointer_cast(&gradPsiQuadValuesZD[0]),
		      gradPsiQuadValuesZH,
		      numCells*numQuads*numPsi*sizeof(double),
		      hipMemcpyHostToDevice);

           hipMemcpy(thrust::raw_pointer_cast(&eigenValuesD[0]),
		      eigenValuesH,
		      numPsi*sizeof(double),
		      hipMemcpyHostToDevice);

           hipMemcpy(thrust::raw_pointer_cast(&partialOccupanciesD[0]),
		      partialOccupanciesH,
		      numPsi*sizeof(double),
		      hipMemcpyHostToDevice);

           hipMemcpy(thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD00[0]),
		      eshelbyTensorQuadValuesH00,
		      numCells*numQuads*sizeof(double),
		      hipMemcpyHostToDevice); 

           hipMemcpy(thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD10[0]),
		      eshelbyTensorQuadValuesH10,
		      numCells*numQuads*sizeof(double),
		      hipMemcpyHostToDevice); 

           hipMemcpy(thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD11[0]),
		      eshelbyTensorQuadValuesH11,
		      numCells*numQuads*sizeof(double),
		      hipMemcpyHostToDevice); 

           hipMemcpy(thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD20[0]),
		      eshelbyTensorQuadValuesH20,
		      numCells*numQuads*sizeof(double),
		      hipMemcpyHostToDevice); 

           hipMemcpy(thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD21[0]),
		      eshelbyTensorQuadValuesH21,
		      numCells*numQuads*sizeof(double),
		      hipMemcpyHostToDevice); 

           hipMemcpy(thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD22[0]),
		      eshelbyTensorQuadValuesH22,
		      numCells*numQuads*sizeof(double),
		      hipMemcpyHostToDevice); 
           
           computeELocWfcEshelbyTensorNonPeriodicD(operatorMatrix,
                                                  psiQuadValuesD,
                                                  gradPsiQuadValuesXD,
                                                  gradPsiQuadValuesYD,
                                                  gradPsiQuadValuesZD,
                                                  eigenValuesD,
                                                  partialOccupanciesD,
                                                  numCells,
                                                  numQuads,
                                                  numPsi,
                                                  eshelbyTensorQuadValuesD00,
                                                  eshelbyTensorQuadValuesD10,
                                                  eshelbyTensorQuadValuesD11,
                                                  eshelbyTensorQuadValuesD20,
                                                  eshelbyTensorQuadValuesD21,
                                                  eshelbyTensorQuadValuesD22); 
           
           hipMemcpy(eshelbyTensorQuadValuesH00,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD00[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);  

           hipMemcpy(eshelbyTensorQuadValuesH10,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD10[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
           hipMemcpy(eshelbyTensorQuadValuesH11,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD11[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
           hipMemcpy(eshelbyTensorQuadValuesH20,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD20[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
           hipMemcpy(eshelbyTensorQuadValuesH21,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD21[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
           hipMemcpy(eshelbyTensorQuadValuesH22,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD22[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);    
           hipDeviceSynchronize();
      }

      void computeELocWfcEshelbyTensorNonPeriodicD(operatorDFTCUDAClass & operatorMatrix,
                                                 const thrust::device_vector<double> & psiQuadValuesD,
                                                 const thrust::device_vector<double> & gradPsiQuadValuesXD,
                                                 const thrust::device_vector<double> & gradPsiQuadValuesYD,
                                                 const thrust::device_vector<double> & gradPsiQuadValuesZD,
                                                 const thrust::device_vector<double> & eigenValuesD,
                                                 const thrust::device_vector<double> & partialOccupanciesD,
                                                 const unsigned int numCells, 
                                                 const unsigned int numQuads,
                                                 const unsigned int numPsi,
                                                 thrust::device_vector<double> & eshelbyTensorQuadValuesD00,
                                                 thrust::device_vector<double> & eshelbyTensorQuadValuesD10,
                                                 thrust::device_vector<double> & eshelbyTensorQuadValuesD11,
                                                 thrust::device_vector<double> & eshelbyTensorQuadValuesD20,
                                                 thrust::device_vector<double> & eshelbyTensorQuadValuesD21,
                                                 thrust::device_vector<double> & eshelbyTensorQuadValuesD22)
      { 
   
           thrust::device_vector<double> eshelbyTensorContributionsD00(numCells*numQuads*numPsi,0.0);
           thrust::device_vector<double> eshelbyTensorContributionsD10(numCells*numQuads*numPsi,0.0); 
           thrust::device_vector<double> eshelbyTensorContributionsD11(numCells*numQuads*numPsi,0.0); 
           thrust::device_vector<double> eshelbyTensorContributionsD20(numCells*numQuads*numPsi,0.0); 
           thrust::device_vector<double> eshelbyTensorContributionsD21(numCells*numQuads*numPsi,0.0); 
           thrust::device_vector<double> eshelbyTensorContributionsD22(numCells*numQuads*numPsi,0.0); 
           

           thrust::device_vector<double> onesVectorD(numPsi,1.0);
           
	   computeELocWfcEshelbyTensorContributions<<<(numPsi+255)/256*numCells*numQuads,256>>>
							  (numPsi,
							   numCells*numQuads,
                                                           0,
                                                           numQuads,
                                                           thrust::raw_pointer_cast(&psiQuadValuesD[0]),
                                                           thrust::raw_pointer_cast(&gradPsiQuadValuesXD[0]),
                                                           thrust::raw_pointer_cast(&gradPsiQuadValuesYD[0]),
                                                           thrust::raw_pointer_cast(&gradPsiQuadValuesZD[0]),
							   thrust::raw_pointer_cast(&eigenValuesD[0]),
                                                           thrust::raw_pointer_cast(&partialOccupanciesD[0]),
							   thrust::raw_pointer_cast(&eshelbyTensorContributionsD00[0]),
                                                           thrust::raw_pointer_cast(&eshelbyTensorContributionsD10[0]),
                                                           thrust::raw_pointer_cast(&eshelbyTensorContributionsD11[0]),
                                                           thrust::raw_pointer_cast(&eshelbyTensorContributionsD20[0]),
                                                           thrust::raw_pointer_cast(&eshelbyTensorContributionsD21[0]),
                                                           thrust::raw_pointer_cast(&eshelbyTensorContributionsD22[0]));

          


	   double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 1.0;


	  
	   hipblasDgemm(operatorMatrix.getCublasHandle(),
		      HIPBLAS_OP_N,
		      HIPBLAS_OP_N,
		      1,
		      numCells*numQuads,
		      numPsi,
		      &scalarCoeffAlpha,
		      thrust::raw_pointer_cast(&onesVectorD[0]),
		      1,
		      thrust::raw_pointer_cast(&eshelbyTensorContributionsD00[0]),
		      numPsi,
		      &scalarCoeffBeta,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD00[0]),
		      1);

	   hipblasDgemm(operatorMatrix.getCublasHandle(),
		      HIPBLAS_OP_N,
		      HIPBLAS_OP_N,
		      1,
		      numCells*numQuads,
		      numPsi,
		      &scalarCoeffAlpha,
		      thrust::raw_pointer_cast(&onesVectorD[0]),
		      1,
		      thrust::raw_pointer_cast(&eshelbyTensorContributionsD10[0]),
		      numPsi,
		      &scalarCoeffBeta,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD10[0]),
		      1); 

	   hipblasDgemm(operatorMatrix.getCublasHandle(),
		      HIPBLAS_OP_N,
		      HIPBLAS_OP_N,
		      1,
		      numCells*numQuads,
		      numPsi,
		      &scalarCoeffAlpha,
		      thrust::raw_pointer_cast(&onesVectorD[0]),
		      1,
		      thrust::raw_pointer_cast(&eshelbyTensorContributionsD11[0]),
		      numPsi,
		      &scalarCoeffBeta,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD11[0]),
		      1); 

	   hipblasDgemm(operatorMatrix.getCublasHandle(),
		      HIPBLAS_OP_N,
		      HIPBLAS_OP_N,
		      1,
		      numCells*numQuads,
		      numPsi,
		      &scalarCoeffAlpha,
		      thrust::raw_pointer_cast(&onesVectorD[0]),
		      1,
		      thrust::raw_pointer_cast(&eshelbyTensorContributionsD20[0]),
		      numPsi,
		      &scalarCoeffBeta,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD20[0]),
		      1); 

	   hipblasDgemm(operatorMatrix.getCublasHandle(),
		      HIPBLAS_OP_N,
		      HIPBLAS_OP_N,
		      1,
		      numCells*numQuads,
		      numPsi,
		      &scalarCoeffAlpha,
		      thrust::raw_pointer_cast(&onesVectorD[0]),
		      1,
		      thrust::raw_pointer_cast(&eshelbyTensorContributionsD21[0]),
		      numPsi,
		      &scalarCoeffBeta,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD21[0]),
		      1); 

	   hipblasDgemm(operatorMatrix.getCublasHandle(),
		      HIPBLAS_OP_N,
		      HIPBLAS_OP_N,
		      1,
		      numCells*numQuads,
		      numPsi,
		      &scalarCoeffAlpha,
		      thrust::raw_pointer_cast(&onesVectorD[0]),
		      1,
		      thrust::raw_pointer_cast(&eshelbyTensorContributionsD22[0]),
		      numPsi,
		      &scalarCoeffBeta,
		      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD22[0]),
		      1);       
      }

      void computeNonLocalProjectorKetTimesPsiTimesVH(operatorDFTCUDAClass & operatorMatrix,
                                                      const double * X,
                                                      const unsigned int startingVecId,
                                                      const unsigned int BVec,
                                                      const unsigned int N,
                                                      double * projectorKetTimesPsiTimesVH)
      {

	    cudaVectorType cudaFlattenedArrayBlock;
	    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
					    BVec,
					    cudaFlattenedArrayBlock);


	    cudaVectorType projectorKetTimesVector;
	    vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
					    BVec,
					    projectorKetTimesVector);


            const unsigned int M=operatorMatrix.getMatrixFreeData()->get_vector_partitioner()->local_size();
            stridedCopyToBlockKernel<<<(BVec+255)/256*M, 256>>>(BVec,
								X,
								M,
								N,
								cudaFlattenedArrayBlock.begin(),
								startingVecId);
            cudaFlattenedArrayBlock.update_ghost_values();
  
            (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
								         BVec);

            operatorMatrix.computeNonLocalProjectorKetTimesXTimesV(cudaFlattenedArrayBlock.begin(),
						                   projectorKetTimesVector,
							           BVec);


            const unsigned int totalSize=projectorKetTimesVector.get_partitioner()->n_ghost_indices()+projectorKetTimesVector.local_size();

            hipMemcpy(projectorKetTimesPsiTimesVH,
		       projectorKetTimesVector.begin(),
		       totalSize*sizeof(double),
		       hipMemcpyDeviceToHost);  
      }


     void interpolatePsiH(operatorDFTCUDAClass & operatorMatrix,
                          const double * X,
                          const unsigned int startingVecId,
                          const unsigned int BVec,
                          const unsigned int N,
                          const unsigned int numCells,
                          const unsigned int numQuads,
                          const unsigned int numQuadsNLP,
                          const unsigned int numNodesPerElement,
                          double * psiQuadsFlatH,
                          double * psiQuadsNLPFlatH,
                          double * gradPsiQuadsXFlatH,
                          double * gradPsiQuadsYFlatH,
                          double * gradPsiQuadsZFlatH,
                          const bool interpolateForNLPQuad)
     {
            thrust::device_vector<double> psiQuadsFlatD(numCells*numQuads*BVec,0.0);
            thrust::device_vector<double> psiQuadsNLPFlatD(numCells*numQuadsNLP*BVec,0.0);
            thrust::device_vector<double> gradPsiQuadsXFlatD(numCells*numQuads*BVec,0.0);
            thrust::device_vector<double> gradPsiQuadsYFlatD(numCells*numQuads*BVec,0.0);
            thrust::device_vector<double> gradPsiQuadsZFlatD(numCells*numQuads*BVec,0.0);


	    cudaVectorType cudaFlattenedArrayBlock;
	    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
					    BVec,
					    cudaFlattenedArrayBlock);


            const unsigned int M=operatorMatrix.getMatrixFreeData()->get_vector_partitioner()->local_size();
            stridedCopyToBlockKernel<<<(BVec+255)/256*M, 256>>>(BVec,
								X,
								M,
								N,
								cudaFlattenedArrayBlock.begin(),
								startingVecId);
            cudaFlattenedArrayBlock.update_ghost_values();
  
            (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
								         BVec);

	    interpolatePsiD(operatorMatrix,
			    cudaFlattenedArrayBlock,
			    BVec,
			    N,
			    numCells,
			    numQuads,
			    numQuadsNLP,
			    numNodesPerElement,
			    psiQuadsFlatD,
			    psiQuadsNLPFlatD,
			    gradPsiQuadsXFlatD,
			    gradPsiQuadsYFlatD,
			    gradPsiQuadsZFlatD,
			    interpolateForNLPQuad);

            hipMemcpy(psiQuadsFlatH,
		      thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
		      numCells*numQuads*BVec*sizeof(double),
		      hipMemcpyDeviceToHost);

            if (interpolateForNLPQuad)
		    hipMemcpy(psiQuadsNLPFlatH,
			      thrust::raw_pointer_cast(&psiQuadsNLPFlatD[0]),
			      numCells*numQuadsNLP*BVec*sizeof(double),
			      hipMemcpyDeviceToHost); 

            hipMemcpy(gradPsiQuadsXFlatH,
		      thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
		      numCells*numQuads*BVec*sizeof(double),
		      hipMemcpyDeviceToHost);

            hipMemcpy(gradPsiQuadsYFlatH,
		      thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
		      numCells*numQuads*BVec*sizeof(double),
		      hipMemcpyDeviceToHost);

            hipMemcpy(gradPsiQuadsZFlatH,
		      thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
		      numCells*numQuads*BVec*sizeof(double),
		      hipMemcpyDeviceToHost); 

     }

     void interpolatePsiD(operatorDFTCUDAClass & operatorMatrix,
                          cudaVectorType & Xb,
                          const unsigned int BVec,
                          const unsigned int N,
                          const unsigned int numCells,
                          const unsigned int numQuads,
                          const unsigned int numQuadsNLP,
                          const unsigned int numNodesPerElement,
                          thrust::device_vector<double> & psiQuadsFlatD,
                          thrust::device_vector<double> & psiQuadsNLPFlatD,
                          thrust::device_vector<double> & gradPsiQuadsXFlatD,
                          thrust::device_vector<double> & gradPsiQuadsYFlatD,
                          thrust::device_vector<double> & gradPsiQuadsZFlatD,
                          const bool interpolateForNLPQuad)
     {
            thrust::device_vector<double> & cellWaveFunctionMatrix = operatorMatrix.getCellWaveFunctionMatrix();

	    copyCUDAKernel<<<(BVec+255)/256*numCells*numNodesPerElement,256>>>
							  (BVec,
							   numCells*numNodesPerElement,
							   Xb.begin(),
							   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));

	    double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
	    int strideA = BVec*numNodesPerElement;
	    int strideB = 0;
	    int strideC = BVec*numQuads;

	  
	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
				    BVec,
				    strideC,
				    numCells);

            if (interpolateForNLPQuad)
            {
		    int strideCNLP = BVec*numQuadsNLP;
		    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    BVec,
					    numQuadsNLP,
					    numNodesPerElement,
					    &scalarCoeffAlpha,
					    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
					    BVec,
					    strideA,
					    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesNLPInverted())[0]),
					    numNodesPerElement,
					    strideB,
					    &scalarCoeffBeta,
					    thrust::raw_pointer_cast(&psiQuadsNLPFlatD[0]),
					    BVec,
					    strideCNLP,
					    numCells);
            }

	    strideB=numNodesPerElement*numQuads;

	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
				    BVec,
				    strideC,
				    numCells);


	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
				    BVec,
				    strideC,
				    numCells);

	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
				    BVec,
				    strideC,
				    numCells);
     }

     void interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD(operatorDFTCUDAClass & operatorMatrix,
						  cudaVectorType & Xb,
						  const unsigned int BVec,
						  const unsigned int numCells,
						  const unsigned int numQuads,
						  const unsigned int numNodesPerElement,
                                                  const thrust::device_vector<double> & eigenValuesD,
                                                  const thrust::device_vector<double> & partialOccupanciesD,
                                                  const unsigned int innerBlockSizeEloc,
                                                  thrust::device_vector<double> & psiQuadsFlatD,
                                                  thrust::device_vector<double> & gradPsiQuadsXFlatD,
                                                  thrust::device_vector<double> & gradPsiQuadsYFlatD,
                                                  thrust::device_vector<double> & gradPsiQuadsZFlatD,
				                  thrust::device_vector<double> & eshelbyTensorContributionsD00,
			                     	  thrust::device_vector<double> & eshelbyTensorContributionsD10,
					          thrust::device_vector<double> & eshelbyTensorContributionsD11,
					          thrust::device_vector<double> & eshelbyTensorContributionsD20,
					          thrust::device_vector<double> & eshelbyTensorContributionsD21,
					          thrust::device_vector<double> & eshelbyTensorContributionsD22,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD00,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD10,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD11,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD20,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD21,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD22)
     {
            //thrust::device_vector<double> gradPsiQuadsXFlatD(numCells*numQuads*BVec,0.0);
            //thrust::device_vector<double> gradPsiQuadsYFlatD(numCells*numQuads*BVec,0.0);
            //thrust::device_vector<double> gradPsiQuadsZFlatD(numCells*numQuads*BVec,0.0);

            thrust::device_vector<double> & cellWaveFunctionMatrix = operatorMatrix.getCellWaveFunctionMatrix();

	    copyCUDAKernel<<<(BVec+255)/256*numCells*numNodesPerElement,256>>>
							  (BVec,
							   numCells*numNodesPerElement,
							   Xb.begin(),
							   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));
            
	    double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
	    int strideA = BVec*numNodesPerElement;
	    int strideB = 0;
	    int strideC = BVec*numQuads;

	  
	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
				    BVec,
				    strideC,
				    numCells);

	    strideB=numNodesPerElement*numQuads;

	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
				    BVec,
				    strideC,
				    numCells);


	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
				    BVec,
				    strideC,
				    numCells);

	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
				    BVec,
				    strideC,
				    numCells);
           
           const int blockSize=innerBlockSizeEloc;
           const int numberBlocks=numCells/blockSize;
           const int remBlockSize=numCells-numberBlocks*blockSize;
           //thrust::device_vector<double> eshelbyTensorContributionsD00(blockSize*numQuads*BVec,0.0);
           //thrust::device_vector<double> eshelbyTensorContributionsD10(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD11(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD20(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD21(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD22(blockSize*numQuads*BVec,0.0); 
           thrust::device_vector<double> onesVectorD(BVec,1.0);
           
           for (int iblock=0; iblock<(numberBlocks+1); iblock++)
	   {
                   const int currentBlockSize= (iblock==numberBlocks)?remBlockSize:blockSize;
                   const int startingId=iblock*blockSize;
                  
                   if (currentBlockSize>0)
	           {
                           
			   computeELocWfcEshelbyTensorContributions<<<(BVec+255)/256*currentBlockSize*numQuads,256>>>
									  (BVec,
									   currentBlockSize*numQuads,
									   startingId,
									   numQuads,
									   thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
									   thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
									   thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
									   thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
									   thrust::raw_pointer_cast(&eigenValuesD[0]),
									   thrust::raw_pointer_cast(&partialOccupanciesD[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD00[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD10[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD11[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD20[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD21[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD22[0]));
			  
			   scalarCoeffAlpha = 1.0;
			   scalarCoeffBeta = 1.0;

			  
			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVectorD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD00[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD00[startingId*numQuads]),
				      1);

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVectorD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD10[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD10[startingId*numQuads]),
				      1); 

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVectorD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD11[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD11[startingId*numQuads]),
				      1); 

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVectorD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD20[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD20[startingId*numQuads]),
				      1); 

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVectorD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD21[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD21[startingId*numQuads]),
				      1); 

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVectorD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD22[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD22[startingId*numQuads]),
				      1); 
		   }
	   }
     }

     void interpolatePsiNLPD(operatorDFTCUDAClass & operatorMatrix,
                          cudaVectorType & Xb,
                          const unsigned int BVec,
                          const unsigned int N,
                          const unsigned int numCells,
                          const unsigned int numQuadsNLP,
                          const unsigned int numNodesPerElement,
                          thrust::device_vector<double> & psiQuadsNLPFlatD)
     {
            thrust::device_vector<double> & cellWaveFunctionMatrix = operatorMatrix.getCellWaveFunctionMatrix();

	    copyCUDAKernel<<<(BVec+255)/256*numCells*numNodesPerElement,256>>>
							  (BVec,
							   numCells*numNodesPerElement,
							   Xb.begin(),
							   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));

	    double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
	    int strideA = BVec*numNodesPerElement;
	    int strideB = 0;

	    int strideCNLP = BVec*numQuadsNLP;
	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuadsNLP,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesNLPInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&psiQuadsNLPFlatD[0]),
				    BVec,
				    strideCNLP,
				    numCells);
     }


     void nlpPsiContractionD(operatorDFTCUDAClass & operatorMatrix,
			    const thrust::device_vector<double> & psiQuadValuesNLPD,
                            const thrust::device_vector<double> & partialOccupanciesD,
                            const double * projectorKetTimesVectorParFlattenedD,
                            const thrust::device_vector<unsigned int> & nonTrivialIdToElemIdMapD,
                            const thrust::device_vector<unsigned int> & projecterKetTimesFlattenedVectorLocalIdsD,
                            const unsigned int numCells, 
			    const unsigned int numQuadsNLP,
			    const unsigned int numPsi,
                            const unsigned int totalNonTrivialPseudoWfcs,
                            const unsigned int innerBlockSizeEnlp,
                            thrust::device_vector<double> & nlpContractionContributionD,
                            thrust::device_vector<double> & projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD)
     {
            const int blockSize=innerBlockSizeEnlp;
            const int numberBlocks=totalNonTrivialPseudoWfcs/blockSize;
            const int remBlockSize=totalNonTrivialPseudoWfcs-numberBlocks*blockSize;
            //thrust::device_vector<double> nlpContractionContributionD(blockSize*numQuadsNLP*numPsi,0.0);
	    thrust::device_vector<double> onesMatD(numPsi,1.0);

            for (int iblock=0; iblock<(numberBlocks+1); iblock++)
            {
                    const int currentBlockSize= (iblock==numberBlocks)?remBlockSize:blockSize;
                    const int startingId=iblock*blockSize;
                    if (currentBlockSize>0)
                    {
			    nlpPsiContractionCUDAKernel<<<(numPsi+255)/256*numQuadsNLP*currentBlockSize,256>>>
									  (numPsi,
									   numQuadsNLP,
									   currentBlockSize,
                                                                           startingId,
									   projectorKetTimesVectorParFlattenedD,
									   thrust::raw_pointer_cast(&psiQuadValuesNLPD[0]),
									   thrust::raw_pointer_cast(&partialOccupanciesD[0]),
									   thrust::raw_pointer_cast(&nonTrivialIdToElemIdMapD[0]),
									   thrust::raw_pointer_cast(&projecterKetTimesFlattenedVectorLocalIdsD[0]),
									   thrust::raw_pointer_cast(&nlpContractionContributionD[0]));
			    double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 1.0;

			  
			    hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuadsNLP,
				      numPsi,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesMatD[0]),
				      1,
				      thrust::raw_pointer_cast(&nlpContractionContributionD[0]),
				      numPsi,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD[startingId*numQuadsNLP]),
				      1);
                    }
            }
     }


     void gpuPortedForceKernelsAllD(operatorDFTCUDAClass & operatorMatrix,
                             cudaVectorType & cudaFlattenedArrayBlock,
                             cudaVectorType & projectorKetTimesVectorD,
                             const double * X,
		             const thrust::device_vector<double> & eigenValuesD,
			     const thrust::device_vector<double> & partialOccupanciesD,
			     const thrust::device_vector<unsigned int> & nonTrivialIdToElemIdMapD,
			     const thrust::device_vector<unsigned int> & projecterKetTimesFlattenedVectorLocalIdsD, 
			     const unsigned int startingVecId,
			     const unsigned int N,
                             const unsigned int numPsi,
			     const unsigned int numCells,
			     const unsigned int numQuads,
			     const unsigned int numQuadsNLP,
			     const unsigned int numNodesPerElement,
			     const unsigned int totalNonTrivialPseudoWfcs,
		  	     thrust::device_vector<double> & psiQuadsFlatD,
			     thrust::device_vector<double> & gradPsiQuadsXFlatD,
			     thrust::device_vector<double> & gradPsiQuadsYFlatD,
			     thrust::device_vector<double> & gradPsiQuadsZFlatD,
	                     thrust::device_vector<double> & eshelbyTensorContributionsD00,
			     thrust::device_vector<double> & eshelbyTensorContributionsD10,
			     thrust::device_vector<double> & eshelbyTensorContributionsD11,
			     thrust::device_vector<double> & eshelbyTensorContributionsD20,
			     thrust::device_vector<double> & eshelbyTensorContributionsD21,
			     thrust::device_vector<double> & eshelbyTensorContributionsD22,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD00,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD10,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD11,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD20,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD21,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD22,
                             thrust::device_vector<double> & nlpContractionContributionD,
			     thrust::device_vector<double> & projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD,
                             const unsigned int innerBlockSizeEloc,
                             const unsigned int innerBlockSizeEnlp,
                             const bool isPsp,
			     const bool interpolateForNLPQuad)
     {

            int this_process;
            MPI_Comm_rank(MPI_COMM_WORLD, &this_process);

            const unsigned int M=operatorMatrix.getMatrixFreeData()->get_vector_partitioner()->local_size();
            stridedCopyToBlockKernel<<<(numPsi+255)/256*M, 256>>>(numPsi,
								X,
								M,
								N,
								cudaFlattenedArrayBlock.begin(),
								startingVecId);
            cudaFlattenedArrayBlock.update_ghost_values();
  
            (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
								         numPsi);


            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
            double kernel1_time = MPI_Wtime();

           interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD(operatorMatrix,
						   cudaFlattenedArrayBlock,
						   numPsi,
						   numCells,
						   numQuads,
						   numNodesPerElement,
                                                   eigenValuesD,
                                                   partialOccupanciesD,
                                                   innerBlockSizeEloc,
                                                   psiQuadsFlatD,
                                                   gradPsiQuadsXFlatD,
                                                   gradPsiQuadsYFlatD,
                                                   gradPsiQuadsZFlatD,
	                                           eshelbyTensorContributionsD00,
			                           eshelbyTensorContributionsD10,
			                           eshelbyTensorContributionsD11,
			                           eshelbyTensorContributionsD20,
			                           eshelbyTensorContributionsD21,
			                           eshelbyTensorContributionsD22,
                                                   eshelbyTensorQuadValuesD00,
                                                   eshelbyTensorQuadValuesD10,
                                                   eshelbyTensorQuadValuesD11,
                                                   eshelbyTensorQuadValuesD20,
                                                   eshelbyTensorQuadValuesD21,
                                                   eshelbyTensorQuadValuesD22);

	   hipDeviceSynchronize();
	   MPI_Barrier(MPI_COMM_WORLD);
	   kernel1_time = MPI_Wtime() - kernel1_time;
	    
	   if (this_process==0 && dftParameters::verbosity>=2)
		 std::cout<<"Time for interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD inside blocked loop: "<<kernel1_time<<std::endl;

           if (isPsp)
           {
                   thrust::device_vector<double> psiQuadsNLPFlatD;
                   if (interpolateForNLPQuad)
                   {
                           psiQuadsFlatD.clear();
			   psiQuadsNLPFlatD.resize(numCells*numQuadsNLP*numPsi,0.0);
			   interpolatePsiNLPD(operatorMatrix,
				    cudaFlattenedArrayBlock,
				    numPsi,
				    N,
				    numCells,
				    numQuadsNLP,
				    numNodesPerElement,
				    psiQuadsNLPFlatD);
                   }

		   hipDeviceSynchronize();
		   MPI_Barrier(MPI_COMM_WORLD);
		   double kernel2_time = MPI_Wtime();

		   operatorMatrix.computeNonLocalProjectorKetTimesXTimesV(cudaFlattenedArrayBlock.begin(),
									   projectorKetTimesVectorD,
									   numPsi);

		   hipDeviceSynchronize();
		   MPI_Barrier(MPI_COMM_WORLD);
		   kernel2_time = MPI_Wtime() - kernel2_time;
		    
		   if (this_process==0 && dftParameters::verbosity>=2)
			 std::cout<<"Time for computeNonLocalProjectorKetTimesXTimesV inside blocked loop: "<<kernel2_time<<std::endl;

		   hipDeviceSynchronize();
		   MPI_Barrier(MPI_COMM_WORLD);
		   double kernel3_time = MPI_Wtime();

		   if (totalNonTrivialPseudoWfcs>0)
		   {
			   nlpPsiContractionD(operatorMatrix,
					      interpolateForNLPQuad?psiQuadsNLPFlatD:psiQuadsFlatD,
					      partialOccupanciesD,
					      projectorKetTimesVectorD.begin(),
					      nonTrivialIdToElemIdMapD,
					      projecterKetTimesFlattenedVectorLocalIdsD,
					      numCells, 
					      numQuadsNLP,
					      numPsi,
					      totalNonTrivialPseudoWfcs,
                                              innerBlockSizeEnlp,
                                              nlpContractionContributionD,
					      projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD);
		   }

		   hipDeviceSynchronize();
		   MPI_Barrier(MPI_COMM_WORLD);
		   kernel3_time = MPI_Wtime() - kernel3_time;
		    
		   if (this_process==0 && dftParameters::verbosity>=2)
			 std::cout<<"Time for nlpPsiContractionD inside blocked loop: "<<kernel3_time<<std::endl;
	   }
     }

     void gpuPortedForceKernelsAllH(operatorDFTCUDAClass & operatorMatrix,
                             const double * X,
		             const double * eigenValuesH,
                             const double  fermiEnergy,
			     const unsigned int * nonTrivialIdToElemIdMapH,
			     const unsigned int * projecterKetTimesFlattenedVectorLocalIdsH, 
			     const unsigned int N,
			     const unsigned int numCells,
			     const unsigned int numQuads,
			     const unsigned int numQuadsNLP,
			     const unsigned int numNodesPerElement,
			     const unsigned int totalNonTrivialPseudoWfcs,
			     double * eshelbyTensorQuadValuesH00,
			     double * eshelbyTensorQuadValuesH10,
			     double * eshelbyTensorQuadValuesH11,
			     double * eshelbyTensorQuadValuesH20,
			     double * eshelbyTensorQuadValuesH21,
			     double * eshelbyTensorQuadValuesH22,
			     double * projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedH,
                             const MPI_Comm & interBandGroupComm,
                             const bool isPsp,
			     const bool interpolateForNLPQuad)
     {
	    //band group parallelization data structures
	    const unsigned int numberBandGroups=
		dealii::Utilities::MPI::n_mpi_processes(interBandGroupComm);
	    const unsigned int bandGroupTaskId = dealii::Utilities::MPI::this_mpi_process(interBandGroupComm);
	    std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
	    dftUtils::createBandParallelizationIndices(interBandGroupComm,
						       N,
						       bandGroupLowHighPlusOneIndices);

	    const unsigned int blockSize=std::min(dftParameters::chebyWfcBlockSize,
						bandGroupLowHighPlusOneIndices[1]);

            int this_process;
            MPI_Comm_rank(MPI_COMM_WORLD, &this_process);
            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
            double gpu_time=MPI_Wtime();

            cudaVectorType cudaFlattenedArrayBlock;
            cudaVectorType projectorKetTimesVectorD;
	    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
					   blockSize,
					   cudaFlattenedArrayBlock);
	    vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
					    blockSize,
					    projectorKetTimesVectorD);

            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
            gpu_time = MPI_Wtime() - gpu_time;
            
            if (this_process==0 && dftParameters::verbosity>=2)
              std::cout<<"Time for creating cuda parallel vectors for force computation: "<<gpu_time<<std::endl;

            gpu_time = MPI_Wtime();

            thrust::device_vector<double> eigenValuesD(blockSize,0.0);
            thrust::device_vector<double> partialOccupanciesD(blockSize,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD00(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD10(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD11(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD20(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD21(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD22(numCells*numQuads,0.0);


            thrust::device_vector<double> psiQuadsFlatD(numCells*numQuads*blockSize,0.0);
            thrust::device_vector<double> gradPsiQuadsXFlatD(numCells*numQuads*blockSize,0.0);
            thrust::device_vector<double> gradPsiQuadsYFlatD(numCells*numQuads*blockSize,0.0);
            thrust::device_vector<double> gradPsiQuadsZFlatD(numCells*numQuads*blockSize,0.0);

            const unsigned int innerBlockSizeEloc=50;
            thrust::device_vector<double> eshelbyTensorContributionsD00(innerBlockSizeEloc*numQuads*blockSize,0.0);
            thrust::device_vector<double> eshelbyTensorContributionsD10(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD11(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD20(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD21(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD22(innerBlockSizeEloc*numQuads*blockSize,0.0); 

            const unsigned int innerBlockSizeEnlp=200;
            thrust::device_vector<double> nlpContractionContributionD(innerBlockSizeEnlp*numQuadsNLP*blockSize,0.0);
            thrust::device_vector<double> projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD;
	    thrust::device_vector<unsigned int> projecterKetTimesFlattenedVectorLocalIdsD;
	    thrust::device_vector<unsigned int> nonTrivialIdToElemIdMapD;
            if (totalNonTrivialPseudoWfcs>0)
            {
		    projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD.resize(totalNonTrivialPseudoWfcs*numQuadsNLP,0.0);
		    projecterKetTimesFlattenedVectorLocalIdsD.resize(totalNonTrivialPseudoWfcs,0.0);
		    nonTrivialIdToElemIdMapD.resize(totalNonTrivialPseudoWfcs,0.0);

		    hipMemcpy(thrust::raw_pointer_cast(&nonTrivialIdToElemIdMapD[0]),
			      nonTrivialIdToElemIdMapH,
			      totalNonTrivialPseudoWfcs*sizeof(unsigned int),
			      hipMemcpyHostToDevice);


		    hipMemcpy(thrust::raw_pointer_cast(&projecterKetTimesFlattenedVectorLocalIdsD[0]),
			      projecterKetTimesFlattenedVectorLocalIdsH,
			      totalNonTrivialPseudoWfcs*sizeof(unsigned int),
			      hipMemcpyHostToDevice);
            }


	    for(unsigned int ivec = 0; ivec < N; ivec+=blockSize)
	    {
	         if((ivec+blockSize)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
		    (ivec+blockSize)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
	         {
		      std::vector<double> blockedEigenValues(blockSize,0.0);
		      std::vector<double> blockedPartialOccupancies(blockSize,0.0);
		      for (unsigned int iWave=0; iWave<blockSize;++iWave)
		      {
			 blockedEigenValues[iWave]=eigenValuesH[ivec+iWave];
			 blockedPartialOccupancies[iWave]
			     =dftUtils::getPartialOccupancy(blockedEigenValues[iWave],
                                                            fermiEnergy,
							    C_kb,
                                                            dftParameters::TVal);
                                                            
		      }



		      hipMemcpy(thrust::raw_pointer_cast(&eigenValuesD[0]),
			      &blockedEigenValues[0],
			      blockSize*sizeof(double),
			      hipMemcpyHostToDevice);

		      hipMemcpy(thrust::raw_pointer_cast(&partialOccupanciesD[0]),
			      &blockedPartialOccupancies[0],
			      blockSize*sizeof(double),
			      hipMemcpyHostToDevice);
                      
                      hipDeviceSynchronize();
                      MPI_Barrier(MPI_COMM_WORLD);
                      double kernel_time = MPI_Wtime();

		      gpuPortedForceKernelsAllD(operatorMatrix,
                                               cudaFlattenedArrayBlock,
                                               projectorKetTimesVectorD,
				               X,
					       eigenValuesD,
					       partialOccupanciesD,
					       nonTrivialIdToElemIdMapD,
					       projecterKetTimesFlattenedVectorLocalIdsD,
					       ivec,
					       N,
					       blockSize,
			                       numCells,
			                       numQuads,
			                       numQuadsNLP,
			                       numNodesPerElement,
					       totalNonTrivialPseudoWfcs,
                                               psiQuadsFlatD,
                                               gradPsiQuadsXFlatD,
                                               gradPsiQuadsYFlatD,
                                               gradPsiQuadsZFlatD,
	                                       eshelbyTensorContributionsD00,
			                       eshelbyTensorContributionsD10,
			                       eshelbyTensorContributionsD11,
			                       eshelbyTensorContributionsD20,
			                       eshelbyTensorContributionsD21,
			                       eshelbyTensorContributionsD22,
					       elocWfcEshelbyTensorQuadValuesD00,
					       elocWfcEshelbyTensorQuadValuesD10,
					       elocWfcEshelbyTensorQuadValuesD11,
					       elocWfcEshelbyTensorQuadValuesD20,
					       elocWfcEshelbyTensorQuadValuesD21,
					       elocWfcEshelbyTensorQuadValuesD22,
                                               nlpContractionContributionD,
					       projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD,
                                               innerBlockSizeEloc,
                                               innerBlockSizeEnlp,
                                               isPsp,
			                       interpolateForNLPQuad);

		      hipDeviceSynchronize();
		      MPI_Barrier(MPI_COMM_WORLD);
		      kernel_time = MPI_Wtime() - kernel_time;
		    
		      if (this_process==0 && dftParameters::verbosity>=2)
		         std::cout<<"Time for force kernels all insided block loop: "<<kernel_time<<std::endl;
                 }//band parallelization
            }//ivec loop

            hipMemcpy(eshelbyTensorQuadValuesH00,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD00[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);  

            hipMemcpy(eshelbyTensorQuadValuesH10,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD10[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH11,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD11[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH20,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD20[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH21,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD21[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH22,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD22[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);  

            if (totalNonTrivialPseudoWfcs>0)
			   hipMemcpy(projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedH,
				      thrust::raw_pointer_cast(&projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD[0]),
				      totalNonTrivialPseudoWfcs*numQuadsNLP*sizeof(double),
				      hipMemcpyDeviceToHost); 
            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
            gpu_time = MPI_Wtime() - gpu_time;
            
            if (this_process==0 && dftParameters::verbosity>=1)
              std::cout<<"Time taken for all gpu kernels force computation: "<<gpu_time<<std::endl;
     }

   }//forceCUDA namespace
}//dftfe namespace
