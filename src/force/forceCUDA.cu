#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2020 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das
//

//source file for force related computations

#include <forceCUDA.h>
#include <dftParameters.h>
#include <dftUtils.h>
#include <constants.h>
#include <vectorUtilities.h>

namespace dftfe
{
   namespace forceCUDA
   {

       namespace
       {

         __global__
         void stridedCopyToBlockKernel(const unsigned int BVec,
                            const double *xVec,
                            const unsigned int M,
                            const unsigned int N,
                            double * yVec,
                            const unsigned int startingXVecId)
         {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = M*BVec;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      unsigned int blockIndex = index/BVec;
		      unsigned int intraBlockIndex=index-blockIndex*BVec;
		      yVec[index]
			      =xVec[blockIndex*N+startingXVecId+intraBlockIndex];
		   }

          }


          __global__
          void copyCUDAKernel(const unsigned int contiguousBlockSize,
                            const unsigned int numContiguousBlocks,
                            const double *copyFromVec,
                            double *copyToVec,
                            const dealii::types::global_dof_index *copyFromVecStartingContiguousBlockIds)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = numContiguousBlocks*contiguousBlockSize;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      unsigned int blockIndex = index/contiguousBlockSize;
		      unsigned int intraBlockIndex=index-blockIndex*contiguousBlockSize;
		      copyToVec[index]
			      =copyFromVec[copyFromVecStartingContiguousBlockIds[blockIndex]+intraBlockIndex];
		   }

          }


          __global__
          void computeELocWfcEshelbyTensorContributions(const unsigned int contiguousBlockSize,
                                                        const unsigned int numContiguousBlocks,
                                                        const unsigned int startingCellId,
                                                        const unsigned int numQuads,
					                const double * psiQuadValues,
					                const double * gradPsiQuadValuesX,
					                const double * gradPsiQuadValuesY,
					                const double * gradPsiQuadValuesZ,
					                const double * eigenValues,
					                const double * partialOccupancies,
                                                        double *eshelbyTensor00,
                                                        double *eshelbyTensor10,
                                                        double *eshelbyTensor11,
                                                        double *eshelbyTensor20,
                                                        double *eshelbyTensor21,
                                                        double *eshelbyTensor22)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = numContiguousBlocks*contiguousBlockSize;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      const unsigned int blockIndex = index/contiguousBlockSize;
		      const unsigned int intraBlockIndex=index-blockIndex*contiguousBlockSize;
                      const unsigned int cellIndex=blockIndex/numQuads;
                      const unsigned int quadId=blockIndex-cellIndex*numQuads;
                      const unsigned int tempIndex=(startingCellId+cellIndex)*numQuads*contiguousBlockSize+quadId*contiguousBlockSize+intraBlockIndex;
                      const double psi=psiQuadValues[tempIndex];
                      const double gradPsiX=gradPsiQuadValuesX[tempIndex];
                      const double gradPsiY=gradPsiQuadValuesY[tempIndex];
                      const double gradPsiZ=gradPsiQuadValuesZ[tempIndex];
                      const double eigenValue=eigenValues[intraBlockIndex];
                      const double partOcc=partialOccupancies[intraBlockIndex];

                      const double identityFactor=partOcc*(gradPsiX*gradPsiX+gradPsiY*gradPsiY+gradPsiZ*gradPsiZ)-2.0*partOcc*eigenValue*psi*psi;
		      eshelbyTensor00[index]=-2.0*partOcc*gradPsiX*gradPsiX+identityFactor;
                      eshelbyTensor10[index]=-2.0*partOcc*gradPsiY*gradPsiX;
                      eshelbyTensor11[index]=-2.0*partOcc*gradPsiY*gradPsiY+identityFactor;
                      eshelbyTensor20[index]=-2.0*partOcc*gradPsiZ*gradPsiX;
                      eshelbyTensor21[index]=-2.0*partOcc*gradPsiZ*gradPsiY;
                      eshelbyTensor22[index]=-2.0*partOcc*gradPsiZ*gradPsiZ+identityFactor;
		   }

          }


          __global__
          void nlpPsiContractionCUDAKernel(const unsigned int numPsi,
                                           const unsigned int numQuadsNLP, 
                                           const unsigned int totalNonTrivialPseudoWfcs,
                                           const unsigned int startingId,
					   const double * projectorKetTimesVectorPar,
					   const double * psiQuadValuesNLP,
                                           const double * partialOccupancies,
					   const unsigned int * nonTrivialIdToElemIdMap,
					   const unsigned int * projecterKetTimesFlattenedVectorLocalIds,
					   double *nlpContractionContribution)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = totalNonTrivialPseudoWfcs*numQuadsNLP*numPsi;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      const unsigned int blockIndex = index/numPsi;
		      const unsigned int wfcId=index-blockIndex*numPsi;
                      unsigned int pseudoWfcId=blockIndex/numQuadsNLP;
                      const unsigned int quadId=blockIndex-pseudoWfcId*numQuadsNLP;
                      pseudoWfcId+=startingId;
                      nlpContractionContribution[index]=partialOccupancies[wfcId]*psiQuadValuesNLP[nonTrivialIdToElemIdMap[pseudoWfcId]*numQuadsNLP*numPsi+quadId*numPsi+wfcId]*projectorKetTimesVectorPar[projecterKetTimesFlattenedVectorLocalIds[pseudoWfcId]*numPsi+wfcId];
		   }

          }

      }


      void computeNonLocalProjectorKetTimesPsiTimesVH(operatorDFTCUDAClass & operatorMatrix,
                                                      const double * X,
                                                      const unsigned int startingVecId,
                                                      const unsigned int BVec,
                                                      const unsigned int N,
                                                      double * projectorKetTimesPsiTimesVH)
      {

	    cudaVectorType cudaFlattenedArrayBlock;
	    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
					    BVec,
					    cudaFlattenedArrayBlock);


	    cudaVectorType projectorKetTimesVector;
	    vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
					    BVec,
					    projectorKetTimesVector);


            const unsigned int M=operatorMatrix.getMatrixFreeData()->get_vector_partitioner()->local_size();
            stridedCopyToBlockKernel<<<(BVec+255)/256*M, 256>>>(BVec,
								X,
								M,
								N,
								cudaFlattenedArrayBlock.begin(),
								startingVecId);
            cudaFlattenedArrayBlock.update_ghost_values();
  
            (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
								         BVec);

            operatorMatrix.computeNonLocalProjectorKetTimesXTimesV(cudaFlattenedArrayBlock.begin(),
						                   projectorKetTimesVector,
							           BVec);


            const unsigned int totalSize=projectorKetTimesVector.get_partitioner()->n_ghost_indices()+projectorKetTimesVector.local_size();

            hipMemcpy(projectorKetTimesPsiTimesVH,
		       projectorKetTimesVector.begin(),
		       totalSize*sizeof(double),
		       hipMemcpyDeviceToHost);  
      }


     void interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD(operatorDFTCUDAClass & operatorMatrix,
						  cudaVectorType & Xb,
						  const unsigned int BVec,
						  const unsigned int numCells,
						  const unsigned int numQuads,
                                                  const unsigned int numQuadsNLP,
						  const unsigned int numNodesPerElement,
                                                  const thrust::device_vector<double> & eigenValuesD,
                                                  const thrust::device_vector<double> & partialOccupanciesD,
                                                  const thrust::device_vector<double> & onesVecD,
                                                  const unsigned int innerBlockSizeEloc,
                                                  thrust::device_vector<double> & psiQuadsFlatD,
                                                  thrust::device_vector<double> & psiQuadsNLPFlatD,
                                                  thrust::device_vector<double> & gradPsiQuadsXFlatD,
                                                  thrust::device_vector<double> & gradPsiQuadsYFlatD,
                                                  thrust::device_vector<double> & gradPsiQuadsZFlatD,
				                  thrust::device_vector<double> & eshelbyTensorContributionsD00,
			                     	  thrust::device_vector<double> & eshelbyTensorContributionsD10,
					          thrust::device_vector<double> & eshelbyTensorContributionsD11,
					          thrust::device_vector<double> & eshelbyTensorContributionsD20,
					          thrust::device_vector<double> & eshelbyTensorContributionsD21,
					          thrust::device_vector<double> & eshelbyTensorContributionsD22,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD00,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD10,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD11,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD20,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD21,
                                                  thrust::device_vector<double> & eshelbyTensorQuadValuesD22,
                                                  const bool interpolateForNLPQuad)
     {
            //thrust::device_vector<double> gradPsiQuadsXFlatD(numCells*numQuads*BVec,0.0);
            //thrust::device_vector<double> gradPsiQuadsYFlatD(numCells*numQuads*BVec,0.0);
            //thrust::device_vector<double> gradPsiQuadsZFlatD(numCells*numQuads*BVec,0.0);

            thrust::device_vector<double> & cellWaveFunctionMatrix = operatorMatrix.getCellWaveFunctionMatrix();

	    copyCUDAKernel<<<(BVec+255)/256*numCells*numNodesPerElement,256>>>
							  (BVec,
							   numCells*numNodesPerElement,
							   Xb.begin(),
							   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));
            
	    double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
	    int strideA = BVec*numNodesPerElement;
	    int strideB = 0;
	    int strideC = BVec*numQuads;

	  
	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
				    BVec,
				    strideC,
				    numCells);

            if (interpolateForNLPQuad)
            {
		    int strideCNLP = BVec*numQuadsNLP;
		    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    BVec,
					    numQuadsNLP,
					    numNodesPerElement,
					    &scalarCoeffAlpha,
					    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
					    BVec,
					    strideA,
					    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesNLPInverted())[0]),
					    numNodesPerElement,
					    strideB,
					    &scalarCoeffBeta,
					    thrust::raw_pointer_cast(&psiQuadsNLPFlatD[0]),
					    BVec,
					    strideCNLP,
					    numCells);
            }

	    strideB=numNodesPerElement*numQuads;

	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
				    BVec,
				    strideC,
				    numCells);


	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
				    BVec,
				    strideC,
				    numCells);

	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuads,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
				    BVec,
				    strideC,
				    numCells);
           
           const int blockSize=innerBlockSizeEloc;
           const int numberBlocks=numCells/blockSize;
           const int remBlockSize=numCells-numberBlocks*blockSize;
           //thrust::device_vector<double> eshelbyTensorContributionsD00(blockSize*numQuads*BVec,0.0);
           //thrust::device_vector<double> eshelbyTensorContributionsD10(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD11(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD20(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD21(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> eshelbyTensorContributionsD22(blockSize*numQuads*BVec,0.0); 
           //thrust::device_vector<double> onesVectorD(BVec,1.0);
          
	   //hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
	   //hipStreamCreate(&stream1);
	   //hipStreamCreate(&stream2);
	   //hipStreamCreate(&stream3);
	   //hipStreamCreate(&stream4);
	   //hipStreamCreate(&stream5);
	   //hipStreamCreate(&stream6);
 
           for (int iblock=0; iblock<(numberBlocks+1); iblock++)
	   {
                   const int currentBlockSize= (iblock==numberBlocks)?remBlockSize:blockSize;
                   const int startingId=iblock*blockSize;
                  
                   if (currentBlockSize>0)
	           {
                           
			   computeELocWfcEshelbyTensorContributions<<<(BVec+255)/256*currentBlockSize*numQuads,256>>>
									  (BVec,
									   currentBlockSize*numQuads,
									   startingId,
									   numQuads,
									   thrust::raw_pointer_cast(&psiQuadsFlatD[0]),
									   thrust::raw_pointer_cast(&gradPsiQuadsXFlatD[0]),
									   thrust::raw_pointer_cast(&gradPsiQuadsYFlatD[0]),
									   thrust::raw_pointer_cast(&gradPsiQuadsZFlatD[0]),
									   thrust::raw_pointer_cast(&eigenValuesD[0]),
									   thrust::raw_pointer_cast(&partialOccupanciesD[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD00[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD10[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD11[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD20[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD21[0]),
									   thrust::raw_pointer_cast(&eshelbyTensorContributionsD22[0]));
			  
			   scalarCoeffAlpha = 1.0;
			   scalarCoeffBeta = 1.0;


                           //hipblasSetStream(operatorMatrix.getCublasHandle(),stream1);  			 
 
			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVecD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD00[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD00[startingId*numQuads]),
				      1);

                           //hipblasSetStream(operatorMatrix.getCublasHandle(),stream2);  

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVecD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD10[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD10[startingId*numQuads]),
				      1); 

                           //hipblasSetStream(operatorMatrix.getCublasHandle(),stream3);  

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVecD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD11[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD11[startingId*numQuads]),
				      1); 


                           //hipblasSetStream(operatorMatrix.getCublasHandle(),stream4); 

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVecD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD20[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD20[startingId*numQuads]),
				      1); 

                           //hipblasSetStream(operatorMatrix.getCublasHandle(),stream5); 

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVecD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD21[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD21[startingId*numQuads]),
				      1); 

                           //hipblasSetStream(operatorMatrix.getCublasHandle(),stream6); 

			   hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuads,
				      BVec,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVecD[0]),
				      1,
				      thrust::raw_pointer_cast(&eshelbyTensorContributionsD22[0]),
				      BVec,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&eshelbyTensorQuadValuesD22[startingId*numQuads]),
				      1); 
		   }
	   }
           //hipblasSetStream(operatorMatrix.getCublasHandle(),NULL);
     }

     void interpolatePsiNLPD(operatorDFTCUDAClass & operatorMatrix,
                          cudaVectorType & Xb,
                          const unsigned int BVec,
                          const unsigned int N,
                          const unsigned int numCells,
                          const unsigned int numQuadsNLP,
                          const unsigned int numNodesPerElement,
                          thrust::device_vector<double> & psiQuadsNLPFlatD)
     {
            thrust::device_vector<double> & cellWaveFunctionMatrix = operatorMatrix.getCellWaveFunctionMatrix();

	    copyCUDAKernel<<<(BVec+255)/256*numCells*numNodesPerElement,256>>>
							  (BVec,
							   numCells*numNodesPerElement,
							   Xb.begin(),
							   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));

	    double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
	    int strideA = BVec*numNodesPerElement;
	    int strideB = 0;

	    int strideCNLP = BVec*numQuadsNLP;
	    hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    BVec,
				    numQuadsNLP,
				    numNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
				    BVec,
				    strideA,
				    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesNLPInverted())[0]),
				    numNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&psiQuadsNLPFlatD[0]),
				    BVec,
				    strideCNLP,
				    numCells);
     }


     void nlpPsiContractionD(operatorDFTCUDAClass & operatorMatrix,
			    const thrust::device_vector<double> & psiQuadValuesNLPD,
                            const thrust::device_vector<double> & partialOccupanciesD,
                            const thrust::device_vector<double> & onesVecD,
                            const double * projectorKetTimesVectorParFlattenedD,
                            const thrust::device_vector<unsigned int> & nonTrivialIdToElemIdMapD,
                            const thrust::device_vector<unsigned int> & projecterKetTimesFlattenedVectorLocalIdsD,
                            const unsigned int numCells, 
			    const unsigned int numQuadsNLP,
			    const unsigned int numPsi,
                            const unsigned int totalNonTrivialPseudoWfcs,
                            const unsigned int innerBlockSizeEnlp,
                            thrust::device_vector<double> & nlpContractionContributionD,
                            thrust::device_vector<double> & projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD)
     {
            const int blockSize=innerBlockSizeEnlp;
            const int numberBlocks=totalNonTrivialPseudoWfcs/blockSize;
            const int remBlockSize=totalNonTrivialPseudoWfcs-numberBlocks*blockSize;
            //thrust::device_vector<double> nlpContractionContributionD(blockSize*numQuadsNLP*numPsi,0.0);
	    //thrust::device_vector<double> onesMatD(numPsi,1.0);

            for (int iblock=0; iblock<(numberBlocks+1); iblock++)
            {
                    const int currentBlockSize= (iblock==numberBlocks)?remBlockSize:blockSize;
                    const int startingId=iblock*blockSize;
                    if (currentBlockSize>0)
                    {
			    nlpPsiContractionCUDAKernel<<<(numPsi+255)/256*numQuadsNLP*currentBlockSize,256>>>
									  (numPsi,
									   numQuadsNLP,
									   currentBlockSize,
                                                                           startingId,
									   projectorKetTimesVectorParFlattenedD,
									   thrust::raw_pointer_cast(&psiQuadValuesNLPD[0]),
									   thrust::raw_pointer_cast(&partialOccupanciesD[0]),
									   thrust::raw_pointer_cast(&nonTrivialIdToElemIdMapD[0]),
									   thrust::raw_pointer_cast(&projecterKetTimesFlattenedVectorLocalIdsD[0]),
									   thrust::raw_pointer_cast(&nlpContractionContributionD[0]));
			    double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 1.0;

			  
			    hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      currentBlockSize*numQuadsNLP,
				      numPsi,
				      &scalarCoeffAlpha,
				      thrust::raw_pointer_cast(&onesVecD[0]),
				      1,
				      thrust::raw_pointer_cast(&nlpContractionContributionD[0]),
				      numPsi,
				      &scalarCoeffBeta,
				      thrust::raw_pointer_cast(&projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD[startingId*numQuadsNLP]),
				      1);
                    }
            }
     }


     void gpuPortedForceKernelsAllD(operatorDFTCUDAClass & operatorMatrix,
                             cudaVectorType & cudaFlattenedArrayBlock,
                             cudaVectorType & projectorKetTimesVectorD,
                             const double * X,
		             const thrust::device_vector<double> & eigenValuesD,
			     const thrust::device_vector<double> & partialOccupanciesD,
                             const thrust::device_vector<double> & onesVecD,
			     const thrust::device_vector<unsigned int> & nonTrivialIdToElemIdMapD,
			     const thrust::device_vector<unsigned int> & projecterKetTimesFlattenedVectorLocalIdsD, 
			     const unsigned int startingVecId,
			     const unsigned int N,
                             const unsigned int numPsi,
			     const unsigned int numCells,
			     const unsigned int numQuads,
			     const unsigned int numQuadsNLP,
			     const unsigned int numNodesPerElement,
			     const unsigned int totalNonTrivialPseudoWfcs,
		  	     thrust::device_vector<double> & psiQuadsFlatD,
                             thrust::device_vector<double> & psiQuadsNLPFlatD,
			     thrust::device_vector<double> & gradPsiQuadsXFlatD,
			     thrust::device_vector<double> & gradPsiQuadsYFlatD,
			     thrust::device_vector<double> & gradPsiQuadsZFlatD,
	                     thrust::device_vector<double> & eshelbyTensorContributionsD00,
			     thrust::device_vector<double> & eshelbyTensorContributionsD10,
			     thrust::device_vector<double> & eshelbyTensorContributionsD11,
			     thrust::device_vector<double> & eshelbyTensorContributionsD20,
			     thrust::device_vector<double> & eshelbyTensorContributionsD21,
			     thrust::device_vector<double> & eshelbyTensorContributionsD22,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD00,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD10,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD11,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD20,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD21,
			     thrust::device_vector<double> & eshelbyTensorQuadValuesD22,
                             thrust::device_vector<double> & nlpContractionContributionD,
			     thrust::device_vector<double> & projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD,
                             const unsigned int innerBlockSizeEloc,
                             const unsigned int innerBlockSizeEnlp,
                             const bool isPsp,
			     const bool interpolateForNLPQuad)
     {

            int this_process;
            MPI_Comm_rank(MPI_COMM_WORLD, &this_process);

            const unsigned int M=operatorMatrix.getMatrixFreeData()->get_vector_partitioner()->local_size();
            stridedCopyToBlockKernel<<<(numPsi+255)/256*M, 256>>>(numPsi,
								X,
								M,
								N,
								cudaFlattenedArrayBlock.begin(),
								startingVecId);
            cudaFlattenedArrayBlock.update_ghost_values();
  
            (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
								         numPsi);


            //hipDeviceSynchronize();
            //MPI_Barrier(MPI_COMM_WORLD);
            //double kernel1_time = MPI_Wtime();

           interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD(operatorMatrix,
						   cudaFlattenedArrayBlock,
						   numPsi,
						   numCells,
						   numQuads,
                                                   numQuadsNLP,
						   numNodesPerElement,
                                                   eigenValuesD,
                                                   partialOccupanciesD,
                                                   onesVecD,
                                                   innerBlockSizeEloc,
                                                   psiQuadsFlatD,
                                                   psiQuadsNLPFlatD,
                                                   gradPsiQuadsXFlatD,
                                                   gradPsiQuadsYFlatD,
                                                   gradPsiQuadsZFlatD,
	                                           eshelbyTensorContributionsD00,
			                           eshelbyTensorContributionsD10,
			                           eshelbyTensorContributionsD11,
			                           eshelbyTensorContributionsD20,
			                           eshelbyTensorContributionsD21,
			                           eshelbyTensorContributionsD22,
                                                   eshelbyTensorQuadValuesD00,
                                                   eshelbyTensorQuadValuesD10,
                                                   eshelbyTensorQuadValuesD11,
                                                   eshelbyTensorQuadValuesD20,
                                                   eshelbyTensorQuadValuesD21,
                                                   eshelbyTensorQuadValuesD22,
                                                   interpolateForNLPQuad);

	   //hipDeviceSynchronize();
	   //MPI_Barrier(MPI_COMM_WORLD);
	   //kernel1_time = MPI_Wtime() - kernel1_time;
	    
	   //if (this_process==0 && dftParameters::verbosity>=5)
	   //	 std::cout<<"Time for interpolatePsiComputeELocWfcEshelbyTensorNonPeriodicD inside blocked loop: "<<kernel1_time<<std::endl;

           if (isPsp)
           {
		   //hipDeviceSynchronize();
		   //MPI_Barrier(MPI_COMM_WORLD);
		   //double kernel2_time = MPI_Wtime();

		   operatorMatrix.computeNonLocalProjectorKetTimesXTimesV(cudaFlattenedArrayBlock.begin(),
									   projectorKetTimesVectorD,
									   numPsi);

		   //hipDeviceSynchronize();
		   //MPI_Barrier(MPI_COMM_WORLD);
		   //kernel2_time = MPI_Wtime() - kernel2_time;
		    
		   //if (this_process==0 && dftParameters::verbosity>=5)
	  	   //  std::cout<<"Time for computeNonLocalProjectorKetTimesXTimesV inside blocked loop: "<<kernel2_time<<std::endl;

		   //hipDeviceSynchronize();
		   //MPI_Barrier(MPI_COMM_WORLD);
		   //double kernel3_time = MPI_Wtime();

		   if (totalNonTrivialPseudoWfcs>0)
		   {
			   nlpPsiContractionD(operatorMatrix,
					      interpolateForNLPQuad?psiQuadsNLPFlatD:psiQuadsFlatD,
					      partialOccupanciesD,
                                              onesVecD,
					      projectorKetTimesVectorD.begin(),
					      nonTrivialIdToElemIdMapD,
					      projecterKetTimesFlattenedVectorLocalIdsD,
					      numCells, 
					      numQuadsNLP,
					      numPsi,
					      totalNonTrivialPseudoWfcs,
                                              innerBlockSizeEnlp,
                                              nlpContractionContributionD,
					      projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD);
		   }

		   //hipDeviceSynchronize();
		   //MPI_Barrier(MPI_COMM_WORLD);
		   //kernel3_time = MPI_Wtime() - kernel3_time;
		    
		   //if (this_process==0 && dftParameters::verbosity>=5)
		   //	 std::cout<<"Time for nlpPsiContractionD inside blocked loop: "<<kernel3_time<<std::endl;
	   }
     }

     void gpuPortedForceKernelsAllH(operatorDFTCUDAClass & operatorMatrix,
                             const double * X,
		             const double * eigenValuesH,
                             const double  fermiEnergy,
			     const unsigned int * nonTrivialIdToElemIdMapH,
			     const unsigned int * projecterKetTimesFlattenedVectorLocalIdsH, 
			     const unsigned int N,
			     const unsigned int numCells,
			     const unsigned int numQuads,
			     const unsigned int numQuadsNLP,
			     const unsigned int numNodesPerElement,
			     const unsigned int totalNonTrivialPseudoWfcs,
			     double * eshelbyTensorQuadValuesH00,
			     double * eshelbyTensorQuadValuesH10,
			     double * eshelbyTensorQuadValuesH11,
			     double * eshelbyTensorQuadValuesH20,
			     double * eshelbyTensorQuadValuesH21,
			     double * eshelbyTensorQuadValuesH22,
			     double * projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedH,
                             const MPI_Comm & interBandGroupComm,
                             const bool isPsp,
			     const bool interpolateForNLPQuad)
     {
	    //band group parallelization data structures
	    const unsigned int numberBandGroups=
		dealii::Utilities::MPI::n_mpi_processes(interBandGroupComm);
	    const unsigned int bandGroupTaskId = dealii::Utilities::MPI::this_mpi_process(interBandGroupComm);
	    std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
	    dftUtils::createBandParallelizationIndices(interBandGroupComm,
						       N,
						       bandGroupLowHighPlusOneIndices);

	    const unsigned int blockSize=std::min(dftParameters::chebyWfcBlockSize,
						bandGroupLowHighPlusOneIndices[1]);

            int this_process;
            MPI_Comm_rank(MPI_COMM_WORLD, &this_process);
            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
            double gpu_time=MPI_Wtime();

            cudaVectorType cudaFlattenedArrayBlock;
            cudaVectorType projectorKetTimesVectorD;
	    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
					   blockSize,
					   cudaFlattenedArrayBlock);
	    vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
					    blockSize,
					    projectorKetTimesVectorD);

            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
            gpu_time = MPI_Wtime() - gpu_time;
            
            if (this_process==0 && dftParameters::verbosity>=2)
              std::cout<<"Time for creating cuda parallel vectors for force computation: "<<gpu_time<<std::endl;

            gpu_time = MPI_Wtime();

            thrust::device_vector<double> eigenValuesD(blockSize,0.0);
            thrust::device_vector<double> partialOccupanciesD(blockSize,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD00(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD10(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD11(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD20(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD21(numCells*numQuads,0.0);
            thrust::device_vector<double> elocWfcEshelbyTensorQuadValuesD22(numCells*numQuads,0.0);


            thrust::device_vector<double> psiQuadsFlatD(numCells*numQuads*blockSize,0.0);
            thrust::device_vector<double> psiQuadsNLPFlatD;
            if (interpolateForNLPQuad)                
                 psiQuadsNLPFlatD.resize(numCells*numQuadsNLP*blockSize,0.0);

            thrust::device_vector<double> gradPsiQuadsXFlatD(numCells*numQuads*blockSize,0.0);
            thrust::device_vector<double> gradPsiQuadsYFlatD(numCells*numQuads*blockSize,0.0);
            thrust::device_vector<double> gradPsiQuadsZFlatD(numCells*numQuads*blockSize,0.0);
            thrust::device_vector<double> onesVecD(blockSize,1.0);

            const unsigned int innerBlockSizeEloc=50;
            thrust::device_vector<double> eshelbyTensorContributionsD00(innerBlockSizeEloc*numQuads*blockSize,0.0);
            thrust::device_vector<double> eshelbyTensorContributionsD10(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD11(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD20(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD21(innerBlockSizeEloc*numQuads*blockSize,0.0); 
            thrust::device_vector<double> eshelbyTensorContributionsD22(innerBlockSizeEloc*numQuads*blockSize,0.0); 

            const unsigned int innerBlockSizeEnlp=400;
            thrust::device_vector<double> nlpContractionContributionD(innerBlockSizeEnlp*numQuadsNLP*blockSize,0.0);
            thrust::device_vector<double> projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD;
	    thrust::device_vector<unsigned int> projecterKetTimesFlattenedVectorLocalIdsD;
	    thrust::device_vector<unsigned int> nonTrivialIdToElemIdMapD;
            if (totalNonTrivialPseudoWfcs>0)
            {
		    projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD.resize(totalNonTrivialPseudoWfcs*numQuadsNLP,0.0);
		    projecterKetTimesFlattenedVectorLocalIdsD.resize(totalNonTrivialPseudoWfcs,0.0);
		    nonTrivialIdToElemIdMapD.resize(totalNonTrivialPseudoWfcs,0.0);

		    hipMemcpy(thrust::raw_pointer_cast(&nonTrivialIdToElemIdMapD[0]),
			      nonTrivialIdToElemIdMapH,
			      totalNonTrivialPseudoWfcs*sizeof(unsigned int),
			      hipMemcpyHostToDevice);


		    hipMemcpy(thrust::raw_pointer_cast(&projecterKetTimesFlattenedVectorLocalIdsD[0]),
			      projecterKetTimesFlattenedVectorLocalIdsH,
			      totalNonTrivialPseudoWfcs*sizeof(unsigned int),
			      hipMemcpyHostToDevice);
            }


	    for(unsigned int ivec = 0; ivec < N; ivec+=blockSize)
	    {
	         if((ivec+blockSize)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
		    (ivec+blockSize)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
	         {
		      std::vector<double> blockedEigenValues(blockSize,0.0);
		      std::vector<double> blockedPartialOccupancies(blockSize,0.0);
		      for (unsigned int iWave=0; iWave<blockSize;++iWave)
		      {
			 blockedEigenValues[iWave]=eigenValuesH[ivec+iWave];
			 blockedPartialOccupancies[iWave]
			     =dftUtils::getPartialOccupancy(blockedEigenValues[iWave],
                                                            fermiEnergy,
							    C_kb,
                                                            dftParameters::TVal);
                                                            
		      }



		      hipMemcpy(thrust::raw_pointer_cast(&eigenValuesD[0]),
			      &blockedEigenValues[0],
			      blockSize*sizeof(double),
			      hipMemcpyHostToDevice);

		      hipMemcpy(thrust::raw_pointer_cast(&partialOccupanciesD[0]),
			      &blockedPartialOccupancies[0],
			      blockSize*sizeof(double),
			      hipMemcpyHostToDevice);
                      
                      //hipDeviceSynchronize();
                      //MPI_Barrier(MPI_COMM_WORLD);
                      //double kernel_time = MPI_Wtime();

		      gpuPortedForceKernelsAllD(operatorMatrix,
                                               cudaFlattenedArrayBlock,
                                               projectorKetTimesVectorD,
				               X,
					       eigenValuesD,
					       partialOccupanciesD,
                                               onesVecD,
					       nonTrivialIdToElemIdMapD,
					       projecterKetTimesFlattenedVectorLocalIdsD,
					       ivec,
					       N,
					       blockSize,
			                       numCells,
			                       numQuads,
			                       numQuadsNLP,
			                       numNodesPerElement,
					       totalNonTrivialPseudoWfcs,
                                               psiQuadsFlatD,
                                               psiQuadsNLPFlatD,
                                               gradPsiQuadsXFlatD,
                                               gradPsiQuadsYFlatD,
                                               gradPsiQuadsZFlatD,
	                                       eshelbyTensorContributionsD00,
			                       eshelbyTensorContributionsD10,
			                       eshelbyTensorContributionsD11,
			                       eshelbyTensorContributionsD20,
			                       eshelbyTensorContributionsD21,
			                       eshelbyTensorContributionsD22,
					       elocWfcEshelbyTensorQuadValuesD00,
					       elocWfcEshelbyTensorQuadValuesD10,
					       elocWfcEshelbyTensorQuadValuesD11,
					       elocWfcEshelbyTensorQuadValuesD20,
					       elocWfcEshelbyTensorQuadValuesD21,
					       elocWfcEshelbyTensorQuadValuesD22,
                                               nlpContractionContributionD,
					       projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD,
                                               innerBlockSizeEloc,
                                               innerBlockSizeEnlp,
                                               isPsp,
			                       interpolateForNLPQuad);

		      //hipDeviceSynchronize();
		      //MPI_Barrier(MPI_COMM_WORLD);
		      //kernel_time = MPI_Wtime() - kernel_time;
		    
		      //if (this_process==0 && dftParameters::verbosity>=5)
		      //   std::cout<<"Time for force kernels all insided block loop: "<<kernel_time<<std::endl;
                 }//band parallelization
            }//ivec loop

            hipMemcpy(eshelbyTensorQuadValuesH00,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD00[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);  

            hipMemcpy(eshelbyTensorQuadValuesH10,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD10[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH11,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD11[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH20,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD20[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH21,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD21[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);   
            hipMemcpy(eshelbyTensorQuadValuesH22,
		      thrust::raw_pointer_cast(&elocWfcEshelbyTensorQuadValuesD22[0]),
		      numCells*numQuads*sizeof(double),
		      hipMemcpyDeviceToHost);  

            if (totalNonTrivialPseudoWfcs>0)
			   hipMemcpy(projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedH,
				      thrust::raw_pointer_cast(&projectorKetTimesPsiTimesVTimesPartOccContractionPsiQuadsFlattenedD[0]),
				      totalNonTrivialPseudoWfcs*numQuadsNLP*sizeof(double),
				      hipMemcpyDeviceToHost); 
            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
            gpu_time = MPI_Wtime() - gpu_time;
            
            if (this_process==0 && dftParameters::verbosity>=1)
              std::cout<<"Time taken for all gpu kernels force computation: "<<gpu_time<<std::endl;
     }

   }//forceCUDA namespace
}//dftfe namespace
