#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2018 The Regents of the University of Michigan and DFT-FE
// authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das
//

// source file for electron density related computations
#include <constants.h>
#include <densityCalculatorCUDA.h>
#include <dftParameters.h>
#include <dftUtils.h>
#include <vectorUtilities.h>

namespace dftfe
{
  namespace CUDA
  {
    namespace
    {
      void
      sumRhoData(
        const dealii::DoFHandler<3> &                  dofHandler,
        std::map<dealii::CellId, std::vector<double>> *rhoValues,
        std::map<dealii::CellId, std::vector<double>> *gradRhoValues,
        std::map<dealii::CellId, std::vector<double>> *rhoValuesSpinPolarized,
        std::map<dealii::CellId, std::vector<double>>
          *             gradRhoValuesSpinPolarized,
        const bool      isGradRhoDataPresent,
        const MPI_Comm &interComm)
      {
        typename dealii::DoFHandler<3>::active_cell_iterator
          cell = dofHandler.begin_active(),
          endc = dofHandler.end();

        // gather density from inter communicator
        if (dealii::Utilities::MPI::n_mpi_processes(interComm) > 1)
          for (; cell != endc; ++cell)
            if (cell->is_locally_owned())
              {
                const dealii::CellId cellId = cell->id();

                dealii::Utilities::MPI::sum((*rhoValues)[cellId],
                                            interComm,
                                            (*rhoValues)[cellId]);
                if (isGradRhoDataPresent)
                  dealii::Utilities::MPI::sum((*gradRhoValues)[cellId],
                                              interComm,
                                              (*gradRhoValues)[cellId]);

                if (dftParameters::spinPolarized == 1)
                  {
                    dealii::Utilities::MPI::sum(
                      (*rhoValuesSpinPolarized)[cellId],
                      interComm,
                      (*rhoValuesSpinPolarized)[cellId]);
                    if (isGradRhoDataPresent)
                      dealii::Utilities::MPI::sum(
                        (*gradRhoValuesSpinPolarized)[cellId],
                        interComm,
                        (*gradRhoValuesSpinPolarized)[cellId]);
                  }
              }
      }

      __global__ void
      stridedCopyToBlockKernel(const unsigned int BVec,
                               const double *     xVec,
                               const unsigned int M,
                               const unsigned int N,
                               double *           yVec,
                               const unsigned int startingXVecId)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int numberEntries = M * BVec;

        for (unsigned int index = globalThreadId; index < numberEntries;
             index += blockDim.x * gridDim.x)
          {
            unsigned int blockIndex      = index / BVec;
            unsigned int intraBlockIndex = index - blockIndex * BVec;
            yVec[index] =
              xVec[blockIndex * N + startingXVecId + intraBlockIndex];
          }
      }


      __global__ void
      copyCUDAKernel(const unsigned int contiguousBlockSize,
                     const unsigned int numContiguousBlocks,
                     const double *     copyFromVec,
                     double *           copyToVec,
                     const dealii::types::global_dof_index
                       *copyFromVecStartingContiguousBlockIds)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int numberEntries =
          numContiguousBlocks * contiguousBlockSize;

        for (unsigned int index = globalThreadId; index < numberEntries;
             index += blockDim.x * gridDim.x)
          {
            unsigned int blockIndex = index / contiguousBlockSize;
            unsigned int intraBlockIndex =
              index - blockIndex * contiguousBlockSize;
            copyToVec[index] =
              copyFromVec[copyFromVecStartingContiguousBlockIds[blockIndex] +
                          intraBlockIndex];
          }
      }



      __global__ void
      computeRhoGradRhoFromInterpolatedValues(
        const unsigned int numberEntries,
        double *           rhoCellsWfcContributions,
        double *           gradRhoCellsWfcContributionsX,
        double *           gradRhoCellsWfcContributionsY,
        double *           gradRhoCellsWfcContributionsZ,
        const bool         isEvaluateGradRho)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;

        for (unsigned int index = globalThreadId; index < numberEntries;
             index += blockDim.x * gridDim.x)
          {
            const double psi                = rhoCellsWfcContributions[index];
            rhoCellsWfcContributions[index] = psi * psi;

            if (isEvaluateGradRho)
              {
                const double gradPsiX = gradRhoCellsWfcContributionsX[index];
                gradRhoCellsWfcContributionsX[index] = 2.0 * psi * gradPsiX;

                const double gradPsiY = gradRhoCellsWfcContributionsY[index];
                gradRhoCellsWfcContributionsY[index] = 2.0 * psi * gradPsiY;

                const double gradPsiZ = gradRhoCellsWfcContributionsZ[index];
                gradRhoCellsWfcContributionsZ[index] = 2.0 * psi * gradPsiZ;
              }
          }
      }


    } // namespace

    void
    computeRhoFromPSI(
      const double *                                 X,
      const double *                                 XFrac,
      const unsigned int                             totalNumWaveFunctions,
      const unsigned int                             Nfr,
      const unsigned int                             numLocalDofs,
      const std::vector<std::vector<double>> &       eigenValues,
      const double                                   fermiEnergy,
      const double                                   fermiEnergyUp,
      const double                                   fermiEnergyDown,
      operatorDFTCUDAClass &                         operatorMatrix,
      const unsigned int                             matrixFreeDofhandlerIndex,
      const dealii::DoFHandler<3> &                  dofHandler,
      const unsigned int                             totalLocallyOwnedCells,
      const unsigned int                             numNodesPerElement,
      const unsigned int                             numQuadPoints,
      const std::vector<double> &                    kPointWeights,
      std::map<dealii::CellId, std::vector<double>> *rhoValues,
      std::map<dealii::CellId, std::vector<double>> *gradRhoValues,
      std::map<dealii::CellId, std::vector<double>> *rhoValuesSpinPolarized,
      std::map<dealii::CellId, std::vector<double>> *gradRhoValuesSpinPolarized,
      const bool                                     isEvaluateGradRho,
      const MPI_Comm &                               interpoolcomm,
      const MPI_Comm &                               interBandGroupComm,
      const bool                                     spectrumSplit,
      const bool                                     use2pPlusOneGLQuad)
    {
#ifdef USE_COMPLEX
      AssertThrow(false, dftUtils::ExcNotImplementedYet());
#else
      if (use2pPlusOneGLQuad)
        AssertThrow(!isEvaluateGradRho,
                  dftUtils::ExcNotImplementedYet());

      int this_process;
      MPI_Comm_rank(MPI_COMM_WORLD, &this_process);
      hipDeviceSynchronize();
      MPI_Barrier(MPI_COMM_WORLD);
      double             gpu_time   = MPI_Wtime();
      const unsigned int numKPoints = kPointWeights.size();

      // band group parallelization data structures
      const unsigned int numberBandGroups =
        dealii::Utilities::MPI::n_mpi_processes(interBandGroupComm);
      const unsigned int bandGroupTaskId =
        dealii::Utilities::MPI::this_mpi_process(interBandGroupComm);
      std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
      dftUtils::createBandParallelizationIndices(
        interBandGroupComm,
        totalNumWaveFunctions,
        bandGroupLowHighPlusOneIndices);

      const unsigned int BVec =
        std::min(dftParameters::chebyWfcBlockSize, totalNumWaveFunctions);

      const double scalarCoeffAlphaRho =
        (dftParameters::spinPolarized == 1) ? 1.0 : 2.0;
      const double scalarCoeffBetaRho = 1.0;
      const double scalarCoeffAlphaGradRho =
        (dftParameters::spinPolarized == 1) ? 1.0 : 2.0;
      const double scalarCoeffBetaGradRho = 1.0;

      const unsigned int cellsBlockSize = 50;
      const unsigned int numCellBlocks =
        totalLocallyOwnedCells / cellsBlockSize;
      const unsigned int remCellBlockSize =
        totalLocallyOwnedCells - numCellBlocks * cellsBlockSize;

      thrust::device_vector<double> rhoDevice(totalLocallyOwnedCells *
                                                numQuadPoints,
                                              0.0);
      thrust::device_vector<double> rhoWfcContributionsDevice(
        cellsBlockSize * numQuadPoints * BVec, 0.0);

      thrust::device_vector<double> gradRhoDeviceX(
        isEvaluateGradRho ? (totalLocallyOwnedCells * numQuadPoints) : 1, 0.0);
      thrust::device_vector<double> gradRhoDeviceY(
        isEvaluateGradRho ? (totalLocallyOwnedCells * numQuadPoints) : 1, 0.0);
      thrust::device_vector<double> gradRhoDeviceZ(
        isEvaluateGradRho ? (totalLocallyOwnedCells * numQuadPoints) : 1, 0.0);
      thrust::device_vector<double> gradRhoWfcContributionsDeviceX(
        isEvaluateGradRho ? (cellsBlockSize * numQuadPoints * BVec) : 1, 0.0);
      thrust::device_vector<double> gradRhoWfcContributionsDeviceY(
        isEvaluateGradRho ? (cellsBlockSize * numQuadPoints * BVec) : 1, 0.0);
      thrust::device_vector<double> gradRhoWfcContributionsDeviceZ(
        isEvaluateGradRho ? (cellsBlockSize * numQuadPoints * BVec) : 1, 0.0);

      thrust::device_vector<double> rhoSpinPolarizedDevice;
      thrust::device_vector<double> gradRhoSpinPolarizedDeviceX;
      thrust::device_vector<double> gradRhoSpinPolarizedDeviceY;
      thrust::device_vector<double> gradRhoSpinPolarizedDeviceZ;
      if (dftParameters::spinPolarized == 1)
        {
          rhoSpinPolarizedDevice.resize(totalLocallyOwnedCells * numQuadPoints,
                                        0.0);
          gradRhoSpinPolarizedDeviceX.resize(totalLocallyOwnedCells *
                                               numQuadPoints,
                                             0.0);
          gradRhoSpinPolarizedDeviceY.resize(totalLocallyOwnedCells *
                                               numQuadPoints,
                                             0.0);
          gradRhoSpinPolarizedDeviceZ.resize(totalLocallyOwnedCells *
                                               numQuadPoints,
                                             0.0);
        }

      std::vector<double>           partialOccupVec(BVec, 0.0);
      thrust::device_vector<double> partialOccupVecDevice(BVec, 0.0);

      // distributedGPUVec<double> & cudaFlattenedArrayBlock =
      // operatorMatrix.getBlockCUDADealiiVector();

      distributedGPUVec<double> &cudaFlattenedArrayBlock =
        operatorMatrix.getParallelChebyBlockVectorDevice();

      const unsigned int numGhosts =
        cudaFlattenedArrayBlock.get_partitioner()->n_ghost_indices();

      thrust::device_vector<double> &cellWaveFunctionMatrix =
        operatorMatrix.getCellWaveFunctionMatrix();

      for (unsigned int jvec = 0; jvec < totalNumWaveFunctions; jvec += BVec)
        {
          if ((jvec + BVec) <=
                bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1] &&
              (jvec + BVec) >
                bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
            {
              if (spectrumSplit)
                {
                  thrust::fill(partialOccupVecDevice.begin(),
                               partialOccupVecDevice.end(),
                               1.0);
                }
              else
                {
                  if (dftParameters::constraintMagnetization)
                    {
                      for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                           ++iEigenVec)
                        {
                          if (eigenValues[0][jvec + iEigenVec] > fermiEnergyUp)
                            partialOccupVec[iEigenVec] = 0.0;
                          else
                            partialOccupVec[iEigenVec] = 1.0;
                        }
                    }
                  else
                    {
                      for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                           ++iEigenVec)
                        {
                          partialOccupVec[iEigenVec] =
                            dftUtils::getPartialOccupancy(
                              eigenValues[0][jvec + iEigenVec],
                              fermiEnergy,
                              C_kb,
                              dftParameters::TVal);
                        }
                    }
                  partialOccupVecDevice = partialOccupVec;
                }

              stridedCopyToBlockKernel<<<(BVec + 255) / 256 * numLocalDofs,
                                         256>>>(BVec,
                                                X,
                                                numLocalDofs,
                                                totalNumWaveFunctions,
                                                cudaFlattenedArrayBlock.begin(),
                                                jvec);


              cudaFlattenedArrayBlock.update_ghost_values();

              (operatorMatrix.getOverloadedConstraintMatrix())
                ->distribute(cudaFlattenedArrayBlock, BVec);

              for (int iblock = 0; iblock < (numCellBlocks + 1); iblock++)
                {
                  const unsigned int currentCellsBlockSize =
                    (iblock == numCellBlocks) ? remCellBlockSize :
                                                cellsBlockSize;
                  if (currentCellsBlockSize > 0)
                    {
                      const unsigned int startingCellId =
                        iblock * cellsBlockSize;

                      copyCUDAKernel<<<(BVec + 255) / 256 *
                                         currentCellsBlockSize *
                                         numNodesPerElement,
                                       256>>>(
                        BVec,
                        currentCellsBlockSize * numNodesPerElement,
                        cudaFlattenedArrayBlock.begin(),
                        thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
                        thrust::raw_pointer_cast(
                          &(operatorMatrix
                              .getFlattenedArrayCellLocalProcIndexIdMap()
                                [startingCellId * numNodesPerElement])));

                      double scalarCoeffAlpha = 1.0, scalarCoeffBeta = 0.0;
                      int    strideA = BVec * numNodesPerElement;
                      int    strideB = 0;
                      int    strideC = BVec * numQuadPoints;


                      hipblasDgemmStridedBatched(
                        operatorMatrix.getCublasHandle(),
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        BVec,
                        numQuadPoints,
                        numNodesPerElement,
                        &scalarCoeffAlpha,
                        thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
                        BVec,
                        strideA,
                        thrust::raw_pointer_cast(
                          &(operatorMatrix.getShapeFunctionValuesInverted(
                            use2pPlusOneGLQuad)[0])),
                        numNodesPerElement,
                        strideB,
                        &scalarCoeffBeta,
                        thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
                        BVec,
                        strideC,
                        currentCellsBlockSize);



                      if (isEvaluateGradRho)
                        {
                          strideB = numNodesPerElement * numQuadPoints;

                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getShapeFunctionGradientValuesXInverted()
                                    [startingCellId * numNodesPerElement *
                                     numQuadPoints])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceX[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);


                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getShapeFunctionGradientValuesYInverted()
                                    [startingCellId * numNodesPerElement *
                                     numQuadPoints])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceY[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);

                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getShapeFunctionGradientValuesZInverted()
                                    [startingCellId * numNodesPerElement *
                                     numQuadPoints])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceZ[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);
                        }



                      computeRhoGradRhoFromInterpolatedValues<<<
                        (BVec + 255) / 256 * numQuadPoints *
                          currentCellsBlockSize,
                        256>>>(currentCellsBlockSize * numQuadPoints * BVec,
                               thrust::raw_pointer_cast(
                                 &rhoWfcContributionsDevice[0]),
                               thrust::raw_pointer_cast(
                                 &gradRhoWfcContributionsDeviceX[0]),
                               thrust::raw_pointer_cast(
                                 &gradRhoWfcContributionsDeviceY[0]),
                               thrust::raw_pointer_cast(
                                 &gradRhoWfcContributionsDeviceZ[0]),
                               isEvaluateGradRho);


                      hipblasDgemm(
                        operatorMatrix.getCublasHandle(),
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        1,
                        currentCellsBlockSize * numQuadPoints,
                        BVec,
                        &scalarCoeffAlphaRho,
                        thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                        1,
                        thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
                        BVec,
                        &scalarCoeffBetaRho,
                        thrust::raw_pointer_cast(
                          &rhoDevice[startingCellId * numQuadPoints]),
                        1);


                      if (isEvaluateGradRho)
                        {
                          hipblasDgemm(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            1,
                            currentCellsBlockSize * numQuadPoints,
                            BVec,
                            &scalarCoeffAlphaGradRho,
                            thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                            1,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceX[0]),
                            BVec,
                            &scalarCoeffBetaGradRho,
                            thrust::raw_pointer_cast(
                              &gradRhoDeviceX[startingCellId * numQuadPoints]),
                            1);


                          hipblasDgemm(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            1,
                            currentCellsBlockSize * numQuadPoints,
                            BVec,
                            &scalarCoeffAlphaGradRho,
                            thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                            1,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceY[0]),
                            BVec,
                            &scalarCoeffBetaGradRho,
                            thrust::raw_pointer_cast(
                              &gradRhoDeviceY[startingCellId * numQuadPoints]),
                            1);

                          hipblasDgemm(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            1,
                            currentCellsBlockSize * numQuadPoints,
                            BVec,
                            &scalarCoeffAlphaGradRho,
                            thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                            1,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceZ[0]),
                            BVec,
                            &scalarCoeffBetaGradRho,
                            thrust::raw_pointer_cast(
                              &gradRhoDeviceZ[startingCellId * numQuadPoints]),
                            1);
                        }
                    } // non-trivial cell block check
                }     // cells block loop
            }         // band parallelizatoin check
        }             // wave function block loop

      if (spectrumSplit)
        for (unsigned int jvec = 0; jvec < Nfr; jvec += BVec)
          if ((jvec + totalNumWaveFunctions - Nfr + BVec) <=
                bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1] &&
              (jvec + totalNumWaveFunctions - Nfr + BVec) >
                bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
            {
              if (dftParameters::constraintMagnetization)
                {
                  for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                       ++iEigenVec)
                    {
                      if (eigenValues[0][(totalNumWaveFunctions - Nfr) + jvec +
                                         iEigenVec] > fermiEnergyUp)
                        partialOccupVec[iEigenVec] = -1.0;
                      else
                        partialOccupVec[iEigenVec] = 0.0;
                    }
                }
              else
                {
                  for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                       ++iEigenVec)
                    {
                      partialOccupVec[iEigenVec] =
                        dftUtils::getPartialOccupancy(
                          eigenValues[0][(totalNumWaveFunctions - Nfr) + jvec +
                                         iEigenVec],
                          fermiEnergy,
                          C_kb,
                          dftParameters::TVal) -
                        1.0;
                    }
                }

              partialOccupVecDevice = partialOccupVec;

              stridedCopyToBlockKernel<<<(BVec + 255) / 256 * numLocalDofs,
                                         256>>>(BVec,
                                                XFrac,
                                                numLocalDofs,
                                                Nfr,
                                                cudaFlattenedArrayBlock.begin(),
                                                jvec);


              cudaFlattenedArrayBlock.update_ghost_values();

              (operatorMatrix.getOverloadedConstraintMatrix())
                ->distribute(cudaFlattenedArrayBlock, BVec);

              for (int iblock = 0; iblock < (numCellBlocks + 1); iblock++)
                {
                  const unsigned int currentCellsBlockSize =
                    (iblock == numCellBlocks) ? remCellBlockSize :
                                                cellsBlockSize;
                  if (currentCellsBlockSize > 0)
                    {
                      const unsigned int startingCellId =
                        iblock * cellsBlockSize;

                      copyCUDAKernel<<<(BVec + 255) / 256 *
                                         currentCellsBlockSize *
                                         numNodesPerElement,
                                       256>>>(
                        BVec,
                        currentCellsBlockSize * numNodesPerElement,
                        cudaFlattenedArrayBlock.begin(),
                        thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
                        thrust::raw_pointer_cast(
                          &(operatorMatrix
                              .getFlattenedArrayCellLocalProcIndexIdMap()
                                [startingCellId * numNodesPerElement])));

                      double scalarCoeffAlpha = 1.0, scalarCoeffBeta = 0.0;
                      int    strideA = BVec * numNodesPerElement;
                      int    strideB = 0;
                      int    strideC = BVec * numQuadPoints;


                      hipblasDgemmStridedBatched(
                        operatorMatrix.getCublasHandle(),
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        BVec,
                        numQuadPoints,
                        numNodesPerElement,
                        &scalarCoeffAlpha,
                        thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
                        BVec,
                        strideA,
                        thrust::raw_pointer_cast(
                          &(operatorMatrix.getShapeFunctionValuesInverted(
                            use2pPlusOneGLQuad)[0])),
                        numNodesPerElement,
                        strideB,
                        &scalarCoeffBeta,
                        thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
                        BVec,
                        strideC,
                        currentCellsBlockSize);



                      if (isEvaluateGradRho)
                        {
                          strideB = numNodesPerElement * numQuadPoints;

                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getShapeFunctionGradientValuesXInverted()
                                    [startingCellId * numNodesPerElement *
                                     numQuadPoints])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceX[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);


                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getShapeFunctionGradientValuesYInverted()
                                    [startingCellId * numNodesPerElement *
                                     numQuadPoints])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceY[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);

                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getShapeFunctionGradientValuesZInverted()
                                    [startingCellId * numNodesPerElement *
                                     numQuadPoints])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceZ[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);
                        }



                      computeRhoGradRhoFromInterpolatedValues<<<
                        (BVec + 255) / 256 * numQuadPoints *
                          currentCellsBlockSize,
                        256>>>(currentCellsBlockSize * numQuadPoints * BVec,
                               thrust::raw_pointer_cast(
                                 &rhoWfcContributionsDevice[0]),
                               thrust::raw_pointer_cast(
                                 &gradRhoWfcContributionsDeviceX[0]),
                               thrust::raw_pointer_cast(
                                 &gradRhoWfcContributionsDeviceY[0]),
                               thrust::raw_pointer_cast(
                                 &gradRhoWfcContributionsDeviceZ[0]),
                               isEvaluateGradRho);


                      hipblasDgemm(
                        operatorMatrix.getCublasHandle(),
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        1,
                        currentCellsBlockSize * numQuadPoints,
                        BVec,
                        &scalarCoeffAlphaRho,
                        thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                        1,
                        thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
                        BVec,
                        &scalarCoeffBetaRho,
                        thrust::raw_pointer_cast(
                          &rhoDevice[startingCellId * numQuadPoints]),
                        1);


                      if (isEvaluateGradRho)
                        {
                          hipblasDgemm(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            1,
                            currentCellsBlockSize * numQuadPoints,
                            BVec,
                            &scalarCoeffAlphaGradRho,
                            thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                            1,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceX[0]),
                            BVec,
                            &scalarCoeffBetaGradRho,
                            thrust::raw_pointer_cast(
                              &gradRhoDeviceX[startingCellId * numQuadPoints]),
                            1);


                          hipblasDgemm(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            1,
                            currentCellsBlockSize * numQuadPoints,
                            BVec,
                            &scalarCoeffAlphaGradRho,
                            thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                            1,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceY[0]),
                            BVec,
                            &scalarCoeffBetaGradRho,
                            thrust::raw_pointer_cast(
                              &gradRhoDeviceY[startingCellId * numQuadPoints]),
                            1);

                          hipblasDgemm(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            1,
                            currentCellsBlockSize * numQuadPoints,
                            BVec,
                            &scalarCoeffAlphaGradRho,
                            thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
                            1,
                            thrust::raw_pointer_cast(
                              &gradRhoWfcContributionsDeviceZ[0]),
                            BVec,
                            &scalarCoeffBetaGradRho,
                            thrust::raw_pointer_cast(
                              &gradRhoDeviceZ[startingCellId * numQuadPoints]),
                            1);
                        }
                    } // non-tivial cells block
                }     // cells block loop
            }         // spectrum split block


      if (dftParameters::spinPolarized == 1)
        {
          for (unsigned int jvec = 0; jvec < totalNumWaveFunctions;
               jvec += BVec)
            {
              if ((jvec + BVec) <=
                    bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1] &&
                  (jvec + BVec) >
                    bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
                {
                  if (spectrumSplit)
                    {
                      thrust::fill(partialOccupVecDevice.begin(),
                                   partialOccupVecDevice.end(),
                                   1.0);
                    }
                  else
                    {
                      if (dftParameters::constraintMagnetization)
                        {
                          for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                               ++iEigenVec)
                            {
                              if (eigenValues[0][jvec + iEigenVec +
                                                 totalNumWaveFunctions] >
                                  fermiEnergyDown)
                                partialOccupVec[iEigenVec] = 0.0;
                              else
                                partialOccupVec[iEigenVec] = 1.0;
                            }
                        }
                      else
                        {
                          for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                               ++iEigenVec)
                            {
                              partialOccupVec[iEigenVec] =
                                dftUtils::getPartialOccupancy(
                                  eigenValues[0][jvec + iEigenVec +
                                                 totalNumWaveFunctions],
                                  fermiEnergy,
                                  C_kb,
                                  dftParameters::TVal);
                            }
                        }
                      partialOccupVecDevice = partialOccupVec;
                    }

                  stridedCopyToBlockKernel<<<(BVec + 255) / 256 * numLocalDofs,
                                             256>>>(
                    BVec,
                    X + numLocalDofs * totalNumWaveFunctions,
                    numLocalDofs,
                    totalNumWaveFunctions,
                    cudaFlattenedArrayBlock.begin(),
                    jvec);


                  cudaFlattenedArrayBlock.update_ghost_values();

                  (operatorMatrix.getOverloadedConstraintMatrix())
                    ->distribute(cudaFlattenedArrayBlock, BVec);

                  for (int iblock = 0; iblock < (numCellBlocks + 1); iblock++)
                    {
                      const unsigned int currentCellsBlockSize =
                        (iblock == numCellBlocks) ? remCellBlockSize :
                                                    cellsBlockSize;
                      if (currentCellsBlockSize > 0)
                        {
                          const unsigned int startingCellId =
                            iblock * cellsBlockSize;

                          copyCUDAKernel<<<(BVec + 255) / 256 *
                                             currentCellsBlockSize *
                                             numNodesPerElement,
                                           256>>>(
                            BVec,
                            currentCellsBlockSize * numNodesPerElement,
                            cudaFlattenedArrayBlock.begin(),
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getFlattenedArrayCellLocalProcIndexIdMap()
                                    [startingCellId * numNodesPerElement])));

                          double scalarCoeffAlpha = 1.0, scalarCoeffBeta = 0.0;
                          int    strideA = BVec * numNodesPerElement;
                          int    strideB = 0;
                          int    strideC = BVec * numQuadPoints;


                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix.getShapeFunctionValuesInverted(
                                use2pPlusOneGLQuad)[0])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &rhoWfcContributionsDevice[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);



                          if (isEvaluateGradRho)
                            {
                              strideB = numNodesPerElement * numQuadPoints;

                              hipblasDgemmStridedBatched(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                BVec,
                                numQuadPoints,
                                numNodesPerElement,
                                &scalarCoeffAlpha,
                                thrust::raw_pointer_cast(
                                  &cellWaveFunctionMatrix[0]),
                                BVec,
                                strideA,
                                thrust::raw_pointer_cast(
                                  &(operatorMatrix
                                      .getShapeFunctionGradientValuesXInverted()
                                        [startingCellId * numNodesPerElement *
                                         numQuadPoints])),
                                numNodesPerElement,
                                strideB,
                                &scalarCoeffBeta,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceX[0]),
                                BVec,
                                strideC,
                                currentCellsBlockSize);


                              hipblasDgemmStridedBatched(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                BVec,
                                numQuadPoints,
                                numNodesPerElement,
                                &scalarCoeffAlpha,
                                thrust::raw_pointer_cast(
                                  &cellWaveFunctionMatrix[0]),
                                BVec,
                                strideA,
                                thrust::raw_pointer_cast(
                                  &(operatorMatrix
                                      .getShapeFunctionGradientValuesYInverted()
                                        [startingCellId * numNodesPerElement *
                                         numQuadPoints])),
                                numNodesPerElement,
                                strideB,
                                &scalarCoeffBeta,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceY[0]),
                                BVec,
                                strideC,
                                currentCellsBlockSize);

                              hipblasDgemmStridedBatched(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                BVec,
                                numQuadPoints,
                                numNodesPerElement,
                                &scalarCoeffAlpha,
                                thrust::raw_pointer_cast(
                                  &cellWaveFunctionMatrix[0]),
                                BVec,
                                strideA,
                                thrust::raw_pointer_cast(
                                  &(operatorMatrix
                                      .getShapeFunctionGradientValuesZInverted()
                                        [startingCellId * numNodesPerElement *
                                         numQuadPoints])),
                                numNodesPerElement,
                                strideB,
                                &scalarCoeffBeta,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceZ[0]),
                                BVec,
                                strideC,
                                currentCellsBlockSize);
                            }



                          computeRhoGradRhoFromInterpolatedValues<<<
                            (BVec + 255) / 256 * numQuadPoints *
                              currentCellsBlockSize,
                            256>>>(currentCellsBlockSize * numQuadPoints * BVec,
                                   thrust::raw_pointer_cast(
                                     &rhoWfcContributionsDevice[0]),
                                   thrust::raw_pointer_cast(
                                     &gradRhoWfcContributionsDeviceX[0]),
                                   thrust::raw_pointer_cast(
                                     &gradRhoWfcContributionsDeviceY[0]),
                                   thrust::raw_pointer_cast(
                                     &gradRhoWfcContributionsDeviceZ[0]),
                                   isEvaluateGradRho);


                          hipblasDgemm(operatorMatrix.getCublasHandle(),
                                      HIPBLAS_OP_N,
                                      HIPBLAS_OP_N,
                                      1,
                                      currentCellsBlockSize * numQuadPoints,
                                      BVec,
                                      &scalarCoeffAlphaRho,
                                      thrust::raw_pointer_cast(
                                        &partialOccupVecDevice[0]),
                                      1,
                                      thrust::raw_pointer_cast(
                                        &rhoWfcContributionsDevice[0]),
                                      BVec,
                                      &scalarCoeffBetaRho,
                                      thrust::raw_pointer_cast(
                                        &rhoSpinPolarizedDevice[startingCellId *
                                                                numQuadPoints]),
                                      1);


                          if (isEvaluateGradRho)
                            {
                              hipblasDgemm(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                1,
                                currentCellsBlockSize * numQuadPoints,
                                BVec,
                                &scalarCoeffAlphaGradRho,
                                thrust::raw_pointer_cast(
                                  &partialOccupVecDevice[0]),
                                1,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceX[0]),
                                BVec,
                                &scalarCoeffBetaGradRho,
                                thrust::raw_pointer_cast(
                                  &gradRhoSpinPolarizedDeviceX[startingCellId *
                                                               numQuadPoints]),
                                1);


                              hipblasDgemm(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                1,
                                currentCellsBlockSize * numQuadPoints,
                                BVec,
                                &scalarCoeffAlphaGradRho,
                                thrust::raw_pointer_cast(
                                  &partialOccupVecDevice[0]),
                                1,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceY[0]),
                                BVec,
                                &scalarCoeffBetaGradRho,
                                thrust::raw_pointer_cast(
                                  &gradRhoSpinPolarizedDeviceY[startingCellId *
                                                               numQuadPoints]),
                                1);

                              hipblasDgemm(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                1,
                                currentCellsBlockSize * numQuadPoints,
                                BVec,
                                &scalarCoeffAlphaGradRho,
                                thrust::raw_pointer_cast(
                                  &partialOccupVecDevice[0]),
                                1,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceZ[0]),
                                BVec,
                                &scalarCoeffBetaGradRho,
                                thrust::raw_pointer_cast(
                                  &gradRhoSpinPolarizedDeviceZ[startingCellId *
                                                               numQuadPoints]),
                                1);
                            }
                        } // non-trivial cell block
                    }     // cells block loop
                }         // band parallelization check
            }             // wavefunction blocks loop

          if (spectrumSplit)
            for (unsigned int jvec = 0; jvec < Nfr; jvec += BVec)
              if ((jvec + totalNumWaveFunctions - Nfr + BVec) <=
                    bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1] &&
                  (jvec + totalNumWaveFunctions - Nfr + BVec) >
                    bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
                {
                  if (dftParameters::constraintMagnetization)
                    {
                      for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                           ++iEigenVec)
                        {
                          if (eigenValues[0]
                                         [(totalNumWaveFunctions - Nfr) + jvec +
                                          iEigenVec + totalNumWaveFunctions] >
                              fermiEnergyDown)
                            partialOccupVec[iEigenVec] = -1.0;
                          else
                            partialOccupVec[iEigenVec] = 0.0;
                        }
                    }
                  else
                    {
                      for (unsigned int iEigenVec = 0; iEigenVec < BVec;
                           ++iEigenVec)
                        {
                          partialOccupVec[iEigenVec] =
                            dftUtils::getPartialOccupancy(
                              eigenValues[0]
                                         [(totalNumWaveFunctions - Nfr) + jvec +
                                          iEigenVec + totalNumWaveFunctions],
                              fermiEnergy,
                              C_kb,
                              dftParameters::TVal) -
                            1.0;
                        }
                    }

                  partialOccupVecDevice = partialOccupVec;

                  stridedCopyToBlockKernel<<<(BVec + 255) / 256 * numLocalDofs,
                                             256>>>(
                    BVec,
                    XFrac + numLocalDofs * Nfr,
                    numLocalDofs,
                    Nfr,
                    cudaFlattenedArrayBlock.begin(),
                    jvec);


                  cudaFlattenedArrayBlock.update_ghost_values();

                  (operatorMatrix.getOverloadedConstraintMatrix())
                    ->distribute(cudaFlattenedArrayBlock, BVec);

                  for (int iblock = 0; iblock < (numCellBlocks + 1); iblock++)
                    {
                      const unsigned int currentCellsBlockSize =
                        (iblock == numCellBlocks) ? remCellBlockSize :
                                                    cellsBlockSize;
                      if (currentCellsBlockSize > 0)
                        {
                          const unsigned int startingCellId =
                            iblock * cellsBlockSize;

                          copyCUDAKernel<<<(BVec + 255) / 256 *
                                             currentCellsBlockSize *
                                             numNodesPerElement,
                                           256>>>(
                            BVec,
                            currentCellsBlockSize * numNodesPerElement,
                            cudaFlattenedArrayBlock.begin(),
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            thrust::raw_pointer_cast(
                              &(operatorMatrix
                                  .getFlattenedArrayCellLocalProcIndexIdMap()
                                    [startingCellId * numNodesPerElement])));

                          double scalarCoeffAlpha = 1.0, scalarCoeffBeta = 0.0;
                          int    strideA = BVec * numNodesPerElement;
                          int    strideB = 0;
                          int    strideC = BVec * numQuadPoints;


                          hipblasDgemmStridedBatched(
                            operatorMatrix.getCublasHandle(),
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            BVec,
                            numQuadPoints,
                            numNodesPerElement,
                            &scalarCoeffAlpha,
                            thrust::raw_pointer_cast(
                              &cellWaveFunctionMatrix[0]),
                            BVec,
                            strideA,
                            thrust::raw_pointer_cast(
                              &(operatorMatrix.getShapeFunctionValuesInverted(
                                use2pPlusOneGLQuad)[0])),
                            numNodesPerElement,
                            strideB,
                            &scalarCoeffBeta,
                            thrust::raw_pointer_cast(
                              &rhoWfcContributionsDevice[0]),
                            BVec,
                            strideC,
                            currentCellsBlockSize);



                          if (isEvaluateGradRho)
                            {
                              strideB = numNodesPerElement * numQuadPoints;

                              hipblasDgemmStridedBatched(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                BVec,
                                numQuadPoints,
                                numNodesPerElement,
                                &scalarCoeffAlpha,
                                thrust::raw_pointer_cast(
                                  &cellWaveFunctionMatrix[0]),
                                BVec,
                                strideA,
                                thrust::raw_pointer_cast(
                                  &(operatorMatrix
                                      .getShapeFunctionGradientValuesXInverted()
                                        [startingCellId * numNodesPerElement *
                                         numQuadPoints])),
                                numNodesPerElement,
                                strideB,
                                &scalarCoeffBeta,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceX[0]),
                                BVec,
                                strideC,
                                currentCellsBlockSize);


                              hipblasDgemmStridedBatched(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                BVec,
                                numQuadPoints,
                                numNodesPerElement,
                                &scalarCoeffAlpha,
                                thrust::raw_pointer_cast(
                                  &cellWaveFunctionMatrix[0]),
                                BVec,
                                strideA,
                                thrust::raw_pointer_cast(
                                  &(operatorMatrix
                                      .getShapeFunctionGradientValuesYInverted()
                                        [startingCellId * numNodesPerElement *
                                         numQuadPoints])),
                                numNodesPerElement,
                                strideB,
                                &scalarCoeffBeta,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceY[0]),
                                BVec,
                                strideC,
                                currentCellsBlockSize);

                              hipblasDgemmStridedBatched(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                BVec,
                                numQuadPoints,
                                numNodesPerElement,
                                &scalarCoeffAlpha,
                                thrust::raw_pointer_cast(
                                  &cellWaveFunctionMatrix[0]),
                                BVec,
                                strideA,
                                thrust::raw_pointer_cast(
                                  &(operatorMatrix
                                      .getShapeFunctionGradientValuesZInverted()
                                        [startingCellId * numNodesPerElement *
                                         numQuadPoints])),
                                numNodesPerElement,
                                strideB,
                                &scalarCoeffBeta,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceZ[0]),
                                BVec,
                                strideC,
                                currentCellsBlockSize);
                            }



                          computeRhoGradRhoFromInterpolatedValues<<<
                            (BVec + 255) / 256 * numQuadPoints *
                              currentCellsBlockSize,
                            256>>>(currentCellsBlockSize * numQuadPoints * BVec,
                                   thrust::raw_pointer_cast(
                                     &rhoWfcContributionsDevice[0]),
                                   thrust::raw_pointer_cast(
                                     &gradRhoWfcContributionsDeviceX[0]),
                                   thrust::raw_pointer_cast(
                                     &gradRhoWfcContributionsDeviceY[0]),
                                   thrust::raw_pointer_cast(
                                     &gradRhoWfcContributionsDeviceZ[0]),
                                   isEvaluateGradRho);


                          hipblasDgemm(operatorMatrix.getCublasHandle(),
                                      HIPBLAS_OP_N,
                                      HIPBLAS_OP_N,
                                      1,
                                      currentCellsBlockSize * numQuadPoints,
                                      BVec,
                                      &scalarCoeffAlphaRho,
                                      thrust::raw_pointer_cast(
                                        &partialOccupVecDevice[0]),
                                      1,
                                      thrust::raw_pointer_cast(
                                        &rhoWfcContributionsDevice[0]),
                                      BVec,
                                      &scalarCoeffBetaRho,
                                      thrust::raw_pointer_cast(
                                        &rhoSpinPolarizedDevice[startingCellId *
                                                                numQuadPoints]),
                                      1);


                          if (isEvaluateGradRho)
                            {
                              hipblasDgemm(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                1,
                                currentCellsBlockSize * numQuadPoints,
                                BVec,
                                &scalarCoeffAlphaGradRho,
                                thrust::raw_pointer_cast(
                                  &partialOccupVecDevice[0]),
                                1,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceX[0]),
                                BVec,
                                &scalarCoeffBetaGradRho,
                                thrust::raw_pointer_cast(
                                  &gradRhoSpinPolarizedDeviceX[startingCellId *
                                                               numQuadPoints]),
                                1);


                              hipblasDgemm(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                1,
                                currentCellsBlockSize * numQuadPoints,
                                BVec,
                                &scalarCoeffAlphaGradRho,
                                thrust::raw_pointer_cast(
                                  &partialOccupVecDevice[0]),
                                1,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceY[0]),
                                BVec,
                                &scalarCoeffBetaGradRho,
                                thrust::raw_pointer_cast(
                                  &gradRhoSpinPolarizedDeviceY[startingCellId *
                                                               numQuadPoints]),
                                1);

                              hipblasDgemm(
                                operatorMatrix.getCublasHandle(),
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                1,
                                currentCellsBlockSize * numQuadPoints,
                                BVec,
                                &scalarCoeffAlphaGradRho,
                                thrust::raw_pointer_cast(
                                  &partialOccupVecDevice[0]),
                                1,
                                thrust::raw_pointer_cast(
                                  &gradRhoWfcContributionsDeviceZ[0]),
                                BVec,
                                &scalarCoeffBetaGradRho,
                                thrust::raw_pointer_cast(
                                  &gradRhoSpinPolarizedDeviceZ[startingCellId *
                                                               numQuadPoints]),
                                1);
                            }
                        } // non-trivial cell block
                    }     // cells block loop
                }         // spectrum-split check
        }                 // spin polarized

      // do cuda memcopy to host
      std::vector<double> rhoHost;
      std::vector<double> gradRhoHostX;
      std::vector<double> gradRhoHostY;
      std::vector<double> gradRhoHostZ;

      std::vector<double> rhoSpinPolarizedHost;
      std::vector<double> gradRhoSpinPolarizedHostX;
      std::vector<double> gradRhoSpinPolarizedHostY;
      std::vector<double> gradRhoSpinPolarizedHostZ;

      if (dftParameters::spinPolarized == 1)
        {
          rhoSpinPolarizedHost.resize(totalLocallyOwnedCells * numQuadPoints,
                                      0.0);
          hipMemcpy(&rhoSpinPolarizedHost[0],
                     thrust::raw_pointer_cast(&rhoSpinPolarizedDevice[0]),
                     totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                     hipMemcpyDeviceToHost);

          if (isEvaluateGradRho)
            {
              gradRhoSpinPolarizedHostX.resize(totalLocallyOwnedCells *
                                                 numQuadPoints,
                                               0.0);
              hipMemcpy(
                &gradRhoSpinPolarizedHostX[0],
                thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceX[0]),
                totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                hipMemcpyDeviceToHost);

              gradRhoSpinPolarizedHostY.resize(totalLocallyOwnedCells *
                                                 numQuadPoints,
                                               0.0);
              hipMemcpy(
                &gradRhoSpinPolarizedHostY[0],
                thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceY[0]),
                totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                hipMemcpyDeviceToHost);

              gradRhoSpinPolarizedHostZ.resize(totalLocallyOwnedCells *
                                                 numQuadPoints,
                                               0.0);
              hipMemcpy(
                &gradRhoSpinPolarizedHostZ[0],
                thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceZ[0]),
                totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                hipMemcpyDeviceToHost);
            }
        }

      rhoHost.resize(totalLocallyOwnedCells * numQuadPoints, 0.0);
      hipMemcpy(&rhoHost[0],
                 thrust::raw_pointer_cast(&rhoDevice[0]),
                 totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                 hipMemcpyDeviceToHost);

      if (isEvaluateGradRho)
        {
          gradRhoHostX.resize(totalLocallyOwnedCells * numQuadPoints, 0.0);
          hipMemcpy(&gradRhoHostX[0],
                     thrust::raw_pointer_cast(&gradRhoDeviceX[0]),
                     totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                     hipMemcpyDeviceToHost);

          gradRhoHostY.resize(totalLocallyOwnedCells * numQuadPoints, 0.0);
          hipMemcpy(&gradRhoHostY[0],
                     thrust::raw_pointer_cast(&gradRhoDeviceY[0]),
                     totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                     hipMemcpyDeviceToHost);

          gradRhoHostZ.resize(totalLocallyOwnedCells * numQuadPoints, 0.0);
          hipMemcpy(&gradRhoHostZ[0],
                     thrust::raw_pointer_cast(&gradRhoDeviceZ[0]),
                     totalLocallyOwnedCells * numQuadPoints * sizeof(double),
                     hipMemcpyDeviceToHost);
        }

      unsigned int                                         iElem = 0;
      typename dealii::DoFHandler<3>::active_cell_iterator cell =
        dofHandler.begin_active();
      typename dealii::DoFHandler<3>::active_cell_iterator endc =
        dofHandler.end();
      for (; cell != endc; ++cell)
        if (cell->is_locally_owned())
          {
            const dealii::CellId cellid = cell->id();

            std::vector<double>  dummy(1);
            std::vector<double> &tempRhoQuads = (*rhoValues)[cellid];
            std::vector<double> &tempGradRhoQuads =
              isEvaluateGradRho ? (*gradRhoValues)[cellid] : dummy;

            std::vector<double> &tempRhoQuadsSP =
              (dftParameters::spinPolarized == 1) ?
                (*rhoValuesSpinPolarized)[cellid] :
                dummy;
            std::vector<double> &tempGradRhoQuadsSP =
              ((dftParameters::spinPolarized == 1) && isEvaluateGradRho) ?
                (*gradRhoValuesSpinPolarized)[cellid] :
                dummy;

            for (unsigned int q = 0; q < numQuadPoints; ++q)
              {
                if (dftParameters::spinPolarized == 1)
                  {
                    tempRhoQuadsSP[2 * q] = rhoHost[iElem * numQuadPoints + q];
                    tempRhoQuadsSP[2 * q + 1] =
                      rhoSpinPolarizedHost[iElem * numQuadPoints + q];

                    if (isEvaluateGradRho)
                      {
                        tempGradRhoQuadsSP[6 * q] =
                          gradRhoHostX[iElem * numQuadPoints + q];
                        tempGradRhoQuadsSP[6 * q + 3] =
                          gradRhoSpinPolarizedHostX[iElem * numQuadPoints + q];
                        tempGradRhoQuadsSP[6 * q + 1] =
                          gradRhoHostY[iElem * numQuadPoints + q];
                        tempGradRhoQuadsSP[6 * q + 3 + 1] =
                          gradRhoSpinPolarizedHostY[iElem * numQuadPoints + q];
                        tempGradRhoQuadsSP[6 * q + 2] =
                          gradRhoHostZ[iElem * numQuadPoints + q];
                        tempGradRhoQuadsSP[6 * q + 3 + 2] =
                          gradRhoSpinPolarizedHostZ[iElem * numQuadPoints + q];
                      }

                    tempRhoQuads[q] =
                      rhoHost[iElem * numQuadPoints + q] +
                      rhoSpinPolarizedHost[iElem * numQuadPoints + q];

                    if (isEvaluateGradRho)
                      {
                        tempGradRhoQuads[3 * q] =
                          gradRhoHostX[iElem * numQuadPoints + q] +
                          gradRhoSpinPolarizedHostX[iElem * numQuadPoints + q];
                        tempGradRhoQuads[3 * q + 1] =
                          gradRhoHostY[iElem * numQuadPoints + q] +
                          gradRhoSpinPolarizedHostY[iElem * numQuadPoints + q];
                        tempGradRhoQuads[3 * q + 2] =
                          gradRhoHostZ[iElem * numQuadPoints + q] +
                          gradRhoSpinPolarizedHostZ[iElem * numQuadPoints + q];
                      }
                  }
                else
                  {
                    tempRhoQuads[q] = rhoHost[iElem * numQuadPoints + q];


                    if (isEvaluateGradRho)
                      {
                        tempGradRhoQuads[3 * q] =
                          gradRhoHostX[iElem * numQuadPoints + q];
                        tempGradRhoQuads[3 * q + 1] =
                          gradRhoHostY[iElem * numQuadPoints + q];
                        tempGradRhoQuads[3 * q + 2] =
                          gradRhoHostZ[iElem * numQuadPoints + q];
                      }
                  }
              }
            iElem++;
          }



      // gather density from all inter communicators
      sumRhoData(dofHandler,
                 rhoValues,
                 gradRhoValues,
                 rhoValuesSpinPolarized,
                 gradRhoValuesSpinPolarized,
                 isEvaluateGradRho,
                 interBandGroupComm);

      sumRhoData(dofHandler,
                 rhoValues,
                 gradRhoValues,
                 rhoValuesSpinPolarized,
                 gradRhoValuesSpinPolarized,
                 isEvaluateGradRho,
                 interpoolcomm);

      hipDeviceSynchronize();
      MPI_Barrier(MPI_COMM_WORLD);
      gpu_time = MPI_Wtime() - gpu_time;

      if (this_process == 0 && dftParameters::verbosity >= 2)
        std::cout << "Time for compute rho on GPU: " << gpu_time << std::endl;
#endif
    }

  } // namespace CUDA
} // namespace dftfe
