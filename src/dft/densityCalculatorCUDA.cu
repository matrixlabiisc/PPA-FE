#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2018 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das
//

//source file for electron density related computations
#include <densityCalculatorCUDA.h>
#include <dftParameters.h>
#include <dftUtils.h>
#include <constants.h>
#include <vectorUtilities.h>

namespace dftfe
{
   namespace CUDA
   {

       namespace
       {
	    void sumRhoData(const dealii::DoFHandler<3> & dofHandler,
                               std::map<dealii::CellId, std::vector<double> > * rhoValues,
			       std::map<dealii::CellId, std::vector<double> > * gradRhoValues,
			       std::map<dealii::CellId, std::vector<double> > * rhoValuesSpinPolarized,
			       std::map<dealii::CellId, std::vector<double> > * gradRhoValuesSpinPolarized,
			       const bool isGradRhoDataPresent,
			       const MPI_Comm &interComm)
	    {
	         typename dealii::DoFHandler<3>::active_cell_iterator cell = dofHandler.begin_active(), endc = dofHandler.end();

	         //gather density from inter communicator
	         if (dealii::Utilities::MPI::n_mpi_processes(interComm)>1)
	          for (; cell!=endc; ++cell)
		    if (cell->is_locally_owned())
		    {
			    const dealii::CellId cellId=cell->id();

			    dealii::Utilities::MPI::sum((*rhoValues)[cellId],
							interComm,
							(*rhoValues)[cellId]);
			    if(isGradRhoDataPresent)
			       dealii::Utilities::MPI::sum((*gradRhoValues)[cellId],
							   interComm,
							   (*gradRhoValues)[cellId]);

			    if (dftParameters::spinPolarized==1)
			    {
				dealii::Utilities::MPI::sum((*rhoValuesSpinPolarized)[cellId],
							    interComm,
							    (*rhoValuesSpinPolarized)[cellId]);
				if(isGradRhoDataPresent)
				   dealii::Utilities::MPI::sum((*gradRhoValuesSpinPolarized)[cellId],
							       interComm,
							       (*gradRhoValuesSpinPolarized)[cellId]);
			    }
		      }
	   }

         __global__
         void stridedCopyToBlockKernel(const unsigned int BVec,
                            const double *xVec,
                            const unsigned int M,
                            const unsigned int N,
                            double * yVec,
                            const unsigned int startingXVecId)
         {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = M*BVec;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      unsigned int blockIndex = index/BVec;
		      unsigned int intraBlockIndex=index-blockIndex*BVec;
		      yVec[index]
			      =xVec[blockIndex*N+startingXVecId+intraBlockIndex];
		   }

          }


          __global__
          void copyCUDAKernel(const unsigned int contiguousBlockSize,
                            const unsigned int numContiguousBlocks,
                            const double *copyFromVec,
                            double *copyToVec,
                            const dealii::types::global_dof_index *copyFromVecStartingContiguousBlockIds)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
		  const unsigned int numberEntries = numContiguousBlocks*contiguousBlockSize;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
		      unsigned int blockIndex = index/contiguousBlockSize;
		      unsigned int intraBlockIndex=index-blockIndex*contiguousBlockSize;
		      copyToVec[index]
			      =copyFromVec[copyFromVecStartingContiguousBlockIds[blockIndex]+intraBlockIndex];
		   }

          }



          __global__
          void computeRhoGradRhoFromInterpolatedValues(const unsigned int numberEntries,
			    double *rhoCellsWfcContributions,
                            double *gradRhoCellsWfcContributionsX,
                            double *gradRhoCellsWfcContributionsY,
                            double *gradRhoCellsWfcContributionsZ,
                            const bool isEvaluateGradRho)
          {

		  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;

		  for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
		   {
                      const double psi=rhoCellsWfcContributions[index];
                      rhoCellsWfcContributions[index]=psi*psi;

                      if (isEvaluateGradRho)
                      {
                        const double gradPsiX=gradRhoCellsWfcContributionsX[index];
                        gradRhoCellsWfcContributionsX[index]=2.0*psi*gradPsiX;
                         
                        const double gradPsiY=gradRhoCellsWfcContributionsY[index];
                        gradRhoCellsWfcContributionsY[index]=2.0*psi*gradPsiY;

                        const double gradPsiZ=gradRhoCellsWfcContributionsZ[index];
                        gradRhoCellsWfcContributionsZ[index]=2.0*psi*gradPsiZ;
                         
                      }
		   }

          }


      }

      void computeRhoFromPSI
                      (const double* X,
                       const double* XFrac,
                       const unsigned int totalNumWaveFunctions,
                       const unsigned int Nfr,
                       const unsigned int numLocalDofs,
                       const std::vector<std::vector<double>> & eigenValues,
                       const double fermiEnergy,
                       const double fermiEnergyUp,
                       const double fermiEnergyDown,
                       operatorDFTCUDAClass & operatorMatrix,
                       const dealii::DoFHandler<3> & dofHandler,
                       const unsigned int totalLocallyOwnedCells,
                       const unsigned int numNodesPerElement,
                       const unsigned int numQuadPoints,
                       const std::vector<double> & kPointWeights,
                       std::map<dealii::CellId, std::vector<double> > * rhoValues,
                       std::map<dealii::CellId, std::vector<double> > * gradRhoValues,
                       std::map<dealii::CellId, std::vector<double> > * rhoValuesSpinPolarized,
                       std::map<dealii::CellId, std::vector<double> > * gradRhoValuesSpinPolarized,
                       const bool isEvaluateGradRho,
                       const MPI_Comm &interpoolcomm,
                       const MPI_Comm &interBandGroupComm,
                       const bool spectrumSplit,
                       const bool use2pPlusOneGLQuad)
      {
#ifdef USE_COMPLEX
        AssertThrow(false,dftUtils::ExcNotImplementedYet());
#else
           int this_process;
           MPI_Comm_rank(MPI_COMM_WORLD, &this_process);
           hipDeviceSynchronize();
           MPI_Barrier(MPI_COMM_WORLD);
           double gpu_time=MPI_Wtime();
           const unsigned int numKPoints=kPointWeights.size();

	   //band group parallelization data structures
	   const unsigned int numberBandGroups=
		dealii::Utilities::MPI::n_mpi_processes(interBandGroupComm);
	   const unsigned int bandGroupTaskId = dealii::Utilities::MPI::this_mpi_process(interBandGroupComm);
	   std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
	   dftUtils::createBandParallelizationIndices(interBandGroupComm,
						      totalNumWaveFunctions,
						      bandGroupLowHighPlusOneIndices);

           const unsigned int BVec=std::min(dftParameters::chebyWfcBlockSize,
                                                 totalNumWaveFunctions);
           
           const double scalarCoeffAlphaRho = (dftParameters::spinPolarized==1)?1.0:2.0;
           const double scalarCoeffBetaRho = 1.0;
           const double scalarCoeffAlphaGradRho = (dftParameters::spinPolarized==1)?1.0:2.0;
           const double scalarCoeffBetaGradRho = 1.0;

           thrust::device_vector<double> rhoDevice(totalLocallyOwnedCells*numQuadPoints,0.0);
           thrust::device_vector<double> rhoWfcContributionsDevice(totalLocallyOwnedCells*numQuadPoints*BVec,0.0);

           thrust::device_vector<double> gradRhoDeviceX(totalLocallyOwnedCells*numQuadPoints,0.0);
           thrust::device_vector<double> gradRhoDeviceY(totalLocallyOwnedCells*numQuadPoints,0.0);
           thrust::device_vector<double> gradRhoDeviceZ(totalLocallyOwnedCells*numQuadPoints,0.0);
           thrust::device_vector<double> gradRhoWfcContributionsDeviceX(totalLocallyOwnedCells*numQuadPoints*BVec,0.0);
           thrust::device_vector<double> gradRhoWfcContributionsDeviceY(totalLocallyOwnedCells*numQuadPoints*BVec,0.0);
           thrust::device_vector<double> gradRhoWfcContributionsDeviceZ(totalLocallyOwnedCells*numQuadPoints*BVec,0.0);

           std::vector<double> partialOccupVec(BVec,0.0);
           thrust::device_vector<double> partialOccupVecDevice(BVec,0.0);

           //distributedGPUVec<double> & cudaFlattenedArrayBlock = operatorMatrix.getBlockCUDADealiiVector();

           distributedGPUVec<double> cudaFlattenedArrayBlock;
           vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
   				           BVec,
				           cudaFlattenedArrayBlock);

           const unsigned int numGhosts=cudaFlattenedArrayBlock.get_partitioner()->n_ghost_indices();

           thrust::device_vector<double> & cellWaveFunctionMatrix = operatorMatrix.getCellWaveFunctionMatrix();
           
           for (unsigned int jvec = 0; jvec < totalNumWaveFunctions; jvec += BVec)
           {

                  if((jvec+BVec)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
	          (jvec+BVec)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
	          {

			  if (spectrumSplit)
			  {
				  thrust::fill(partialOccupVecDevice.begin(),partialOccupVecDevice.end(),1.0);
			  }
			  else
			  {
				  if(dftParameters::constraintMagnetization)
				  {
					  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
					  {
						if(eigenValues[0][jvec+iEigenVec] > fermiEnergyUp)
							partialOccupVec[iEigenVec] = 0.0 ;
					  }
				  }
				  else
				  {
					  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
					  {
						partialOccupVec[iEigenVec]=dftUtils::getPartialOccupancy
										    (eigenValues[0][jvec+iEigenVec],
										     fermiEnergy,
										     C_kb,
										     dftParameters::TVal);
					  }
				  }
				  partialOccupVecDevice=partialOccupVec;
			  }


			  stridedCopyToBlockKernel<<<(BVec+255)/256*numLocalDofs, 256>>>(BVec,
										  X,
										  numLocalDofs,
										  totalNumWaveFunctions,
										  cudaFlattenedArrayBlock.begin(),
										  jvec);


			  cudaFlattenedArrayBlock.update_ghost_values();
			
			  (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
										   BVec);


			  copyCUDAKernel<<<(BVec+255)/256*totalLocallyOwnedCells*numNodesPerElement,256>>>
									  (BVec,
									   totalLocallyOwnedCells*numNodesPerElement,
									   cudaFlattenedArrayBlock.begin(),
									   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
									   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));
			  
			  double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
			  int strideA = BVec*numNodesPerElement;
			  int strideB = 0;
			  int strideC = BVec*numQuadPoints;

			  
			  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
						    HIPBLAS_OP_N,
						    HIPBLAS_OP_N,
						    BVec,
						    numQuadPoints,
						    numNodesPerElement,
						    &scalarCoeffAlpha,
						    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
						    BVec,
						    strideA,
						    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted(use2pPlusOneGLQuad))[0]),
						    numNodesPerElement,
						    strideB,
						    &scalarCoeffBeta,
						    thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
						    BVec,
						    strideC,
						    totalLocallyOwnedCells);
			  


			  if(isEvaluateGradRho)
			  {
				  strideB=numNodesPerElement*numQuadPoints;

				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);


				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);

				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);
			  }

			 
			   
			  computeRhoGradRhoFromInterpolatedValues<<<(BVec+255)/256*numQuadPoints*totalLocallyOwnedCells, 256>>>
									  (totalLocallyOwnedCells*numQuadPoints*BVec,
									  thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
									  isEvaluateGradRho);
			  
			  
			  hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      totalLocallyOwnedCells*numQuadPoints,
				      BVec,
				      &scalarCoeffAlphaRho,
				      thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
				      1,
				      thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
				      BVec,
				      &scalarCoeffBetaRho,
				      thrust::raw_pointer_cast(&rhoDevice[0]),
				      1);

			  
			  if(isEvaluateGradRho)
			  {
				      
				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoDeviceX[0]),
					    1);


				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoDeviceY[0]),
					    1);

				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoDeviceZ[0]),
					    1);
				 
				 
			   }
		   }
           } 
          
           if (spectrumSplit)
	       for (unsigned int jvec = 0; jvec < Nfr; jvec += BVec)
                   if((jvec+totalNumWaveFunctions-Nfr+BVec)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
	             (jvec+totalNumWaveFunctions-Nfr+BVec)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
                   {

			  if(dftParameters::constraintMagnetization)
			  {
				  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
				  {
					if(eigenValues[0][(totalNumWaveFunctions-Nfr)+jvec+iEigenVec] > fermiEnergyUp)
						partialOccupVec[iEigenVec] = -1.0 ;
				  }
			  }
			  else
			  {
				  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
				  {
					partialOccupVec[iEigenVec]=dftUtils::getPartialOccupancy
									    (eigenValues[0][(totalNumWaveFunctions-Nfr)+jvec+iEigenVec],
									     fermiEnergy,
									     C_kb,
									     dftParameters::TVal)-1.0;
				  }
			  }

			  partialOccupVecDevice=partialOccupVec;

			  stridedCopyToBlockKernel<<<(BVec+255)/256*numLocalDofs, 256>>>(BVec,
										  XFrac,
										  numLocalDofs,
										  Nfr,
										  cudaFlattenedArrayBlock.begin(),
										  jvec);


			  cudaFlattenedArrayBlock.update_ghost_values();
			
			  (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
										   BVec);


			  copyCUDAKernel<<<(BVec+255)/256*totalLocallyOwnedCells*numNodesPerElement,256>>>
									  (BVec,
									   totalLocallyOwnedCells*numNodesPerElement,
									   cudaFlattenedArrayBlock.begin(),
									   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
									   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));
			  
			  double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
			  int strideA = BVec*numNodesPerElement;
			  int strideB = 0;
			  int strideC = BVec*numQuadPoints;

			  
			  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
						    HIPBLAS_OP_N,
						    HIPBLAS_OP_N,
						    BVec,
						    numQuadPoints,
						    numNodesPerElement,
						    &scalarCoeffAlpha,
						    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
						    BVec,
						    strideA,
						    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted(use2pPlusOneGLQuad))[0]),
						    numNodesPerElement,
						    strideB,
						    &scalarCoeffBeta,
						    thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
						    BVec,
						    strideC,
						    totalLocallyOwnedCells);
			  


			  if(isEvaluateGradRho)
			  {
				  strideB=numNodesPerElement*numQuadPoints;

				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);


				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);

				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);
			  }

			 
			   
			  computeRhoGradRhoFromInterpolatedValues<<<(BVec+255)/256*numQuadPoints*totalLocallyOwnedCells, 256>>>
									  (totalLocallyOwnedCells*numQuadPoints*BVec,
									  thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
									  isEvaluateGradRho);
			  
			  
			  hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      totalLocallyOwnedCells*numQuadPoints,
				      BVec,
				      &scalarCoeffAlphaRho,
				      thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
				      1,
				      thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
				      BVec,
				      &scalarCoeffBetaRho,
				      thrust::raw_pointer_cast(&rhoDevice[0]),
				      1);

			  
			  if(isEvaluateGradRho)
			  {
				      
				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoDeviceX[0]),
					    1);


				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoDeviceY[0]),
					    1);

				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoDeviceZ[0]),
					    1);
				 
				 
			   }
                    }
		    
 
           thrust::device_vector<double> rhoSpinPolarizedDevice;
           thrust::device_vector<double> gradRhoSpinPolarizedDeviceX;
           thrust::device_vector<double> gradRhoSpinPolarizedDeviceY;
           thrust::device_vector<double> gradRhoSpinPolarizedDeviceZ;

           if(dftParameters::spinPolarized==1)
           {
              rhoSpinPolarizedDevice.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
              gradRhoSpinPolarizedDeviceX.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
              gradRhoSpinPolarizedDeviceY.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
              gradRhoSpinPolarizedDeviceZ.resize(totalLocallyOwnedCells*numQuadPoints,0.0);

              for (unsigned int jvec = 0; jvec < totalNumWaveFunctions; jvec += BVec)
	      {
                  if((jvec+BVec)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
	          (jvec+BVec)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
	          {
			  if (spectrumSplit)
			  {
				  thrust::fill(partialOccupVecDevice.begin(),partialOccupVecDevice.end(),1.0);
			  }
                          else
			  {
				  if(dftParameters::constraintMagnetization)
				  {
					  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
					  {
						if(eigenValues[0][jvec+iEigenVec+totalNumWaveFunctions] > fermiEnergyDown)
							partialOccupVec[iEigenVec] = 0.0 ;
					  }
				  }
				  else
				  {
					  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
					  {
						partialOccupVec[iEigenVec]=dftUtils::getPartialOccupancy
										    (eigenValues[0][jvec+iEigenVec+totalNumWaveFunctions],
										     fermiEnergy,
										     C_kb,
										     dftParameters::TVal);
					  }
				  }
                                  partialOccupVecDevice=partialOccupVec;
                          }


			  stridedCopyToBlockKernel<<<(BVec+255)/256*numLocalDofs, 256>>>(BVec,
										  X+numLocalDofs*totalNumWaveFunctions,
										  numLocalDofs,
										  totalNumWaveFunctions,
										  cudaFlattenedArrayBlock.begin(),
										  jvec);

  
                          cudaFlattenedArrayBlock.update_ghost_values();
                        
			  (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
										   BVec);


			  copyCUDAKernel<<<(BVec+255)/256*totalLocallyOwnedCells*numNodesPerElement,256>>>
									  (BVec,
									   totalLocallyOwnedCells*numNodesPerElement,
									   cudaFlattenedArrayBlock.begin(),
									   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
									   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));
			  
			  double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
			  int strideA = BVec*numNodesPerElement;
			  int strideB = 0;
			  int strideC = BVec*numQuadPoints;

			  
			  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
						    HIPBLAS_OP_N,
						    HIPBLAS_OP_N,
						    BVec,
						    numQuadPoints,
						    numNodesPerElement,
						    &scalarCoeffAlpha,
						    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
						    BVec,
						    strideA,
						    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted(use2pPlusOneGLQuad))[0]),
						    numNodesPerElement,
						    strideB,
						    &scalarCoeffBeta,
						    thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
						    BVec,
						    strideC,
						    totalLocallyOwnedCells);
			  


			  if(isEvaluateGradRho)
			  {
				  strideB=numNodesPerElement*numQuadPoints;

				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);


				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);

				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);
			  }

			 
			   
			  computeRhoGradRhoFromInterpolatedValues<<<(BVec+255)/256*numQuadPoints*totalLocallyOwnedCells, 256>>>
									  (totalLocallyOwnedCells*numQuadPoints*BVec,
									  thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
									  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
									  isEvaluateGradRho);
			  
			  
			  hipblasDgemm(operatorMatrix.getCublasHandle(),
				      HIPBLAS_OP_N,
				      HIPBLAS_OP_N,
				      1,
				      totalLocallyOwnedCells*numQuadPoints,
				      BVec,
				      &scalarCoeffAlphaRho,
				      thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
				      1,
				      thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
				      BVec,
				      &scalarCoeffBetaRho,
				      thrust::raw_pointer_cast(&rhoSpinPolarizedDevice[0]),
				      1);

			  
			  if(isEvaluateGradRho)
			  {
				      
				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceX[0]),
					    1);


				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceY[0]),
					    1);

				hipblasDgemm(operatorMatrix.getCublasHandle(),
					    HIPBLAS_OP_N,
					    HIPBLAS_OP_N,
					    1,
					    totalLocallyOwnedCells*numQuadPoints,
					    BVec,
					    &scalarCoeffAlphaGradRho,
					    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					    1,
					    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
					    BVec,
					    &scalarCoeffBetaGradRho,
					    thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceZ[0]),
					    1);
				 
				 
			   }
                        }
		   }

                   if (spectrumSplit)     
	              for (unsigned int jvec = 0; jvec < Nfr; jvec += BVec)
                          if((jvec+totalNumWaveFunctions-Nfr+BVec)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
	                  (jvec+totalNumWaveFunctions-Nfr+BVec)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
			  {
				  if(dftParameters::constraintMagnetization)
				  {
					  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
					  {
						if(eigenValues[0][(totalNumWaveFunctions-Nfr)+jvec+iEigenVec+totalNumWaveFunctions] > fermiEnergyDown)
							partialOccupVec[iEigenVec] = -1.0 ;
					  }
				  }
				  else
				  {
					  for(unsigned int iEigenVec=0; iEigenVec<BVec; ++iEigenVec)
					  {
						partialOccupVec[iEigenVec]=dftUtils::getPartialOccupancy
										    (eigenValues[0][(totalNumWaveFunctions-Nfr)+jvec+iEigenVec+totalNumWaveFunctions],
										     fermiEnergy,
										     C_kb,
										     dftParameters::TVal)-1.0;
					  }
				  }

				  partialOccupVecDevice=partialOccupVec;

				  stridedCopyToBlockKernel<<<(BVec+255)/256*numLocalDofs, 256>>>(BVec,
											  XFrac+numLocalDofs*Nfr,
											  numLocalDofs,
											  Nfr,
											  cudaFlattenedArrayBlock.begin(),
											  jvec);

	  
				  cudaFlattenedArrayBlock.update_ghost_values();
				
				  (operatorMatrix.getOverloadedConstraintMatrix())->distribute(cudaFlattenedArrayBlock,
											   BVec);


				  copyCUDAKernel<<<(BVec+255)/256*totalLocallyOwnedCells*numNodesPerElement,256>>>
										  (BVec,
										   totalLocallyOwnedCells*numNodesPerElement,
										   cudaFlattenedArrayBlock.begin(),
										   thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
										   thrust::raw_pointer_cast(&(operatorMatrix.getFlattenedArrayCellLocalProcIndexIdMap())[0]));
				  
				  double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
				  int strideA = BVec*numNodesPerElement;
				  int strideB = 0;
				  int strideC = BVec*numQuadPoints;

				  
				  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
							    HIPBLAS_OP_N,
							    HIPBLAS_OP_N,
							    BVec,
							    numQuadPoints,
							    numNodesPerElement,
							    &scalarCoeffAlpha,
							    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
							    BVec,
							    strideA,
							    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionValuesInverted(use2pPlusOneGLQuad))[0]),
							    numNodesPerElement,
							    strideB,
							    &scalarCoeffBeta,
							    thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
							    BVec,
							    strideC,
							    totalLocallyOwnedCells);
				  


				  if(isEvaluateGradRho)
				  {
					  strideB=numNodesPerElement*numQuadPoints;

					  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
								    HIPBLAS_OP_N,
								    HIPBLAS_OP_N,
								    BVec,
								    numQuadPoints,
								    numNodesPerElement,
								    &scalarCoeffAlpha,
								    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
								    BVec,
								    strideA,
								    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesXInverted(use2pPlusOneGLQuad))[0]),
								    numNodesPerElement,
								    strideB,
								    &scalarCoeffBeta,
								    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
								    BVec,
								    strideC,
								    totalLocallyOwnedCells);


					  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
								    HIPBLAS_OP_N,
								    HIPBLAS_OP_N,
								    BVec,
								    numQuadPoints,
								    numNodesPerElement,
								    &scalarCoeffAlpha,
								    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
								    BVec,
								    strideA,
								    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesYInverted(use2pPlusOneGLQuad))[0]),
								    numNodesPerElement,
								    strideB,
								    &scalarCoeffBeta,
								    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
								    BVec,
								    strideC,
								    totalLocallyOwnedCells);

					  hipblasDgemmStridedBatched(operatorMatrix.getCublasHandle(),
								    HIPBLAS_OP_N,
								    HIPBLAS_OP_N,
								    BVec,
								    numQuadPoints,
								    numNodesPerElement,
								    &scalarCoeffAlpha,
								    thrust::raw_pointer_cast(&cellWaveFunctionMatrix[0]),
								    BVec,
								    strideA,
								    thrust::raw_pointer_cast(&(operatorMatrix.getShapeFunctionGradientValuesZInverted(use2pPlusOneGLQuad))[0]),
								    numNodesPerElement,
								    strideB,
								    &scalarCoeffBeta,
								    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
								    BVec,
								    strideC,
								    totalLocallyOwnedCells);
				  }

				 
				   
				  computeRhoGradRhoFromInterpolatedValues<<<(BVec+255)/256*numQuadPoints*totalLocallyOwnedCells, 256>>>
										  (totalLocallyOwnedCells*numQuadPoints*BVec,
										  thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
										  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
										  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
										  thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
										  isEvaluateGradRho);
				  
				  
				  hipblasDgemm(operatorMatrix.getCublasHandle(),
					      HIPBLAS_OP_N,
					      HIPBLAS_OP_N,
					      1,
					      totalLocallyOwnedCells*numQuadPoints,
					      BVec,
					      &scalarCoeffAlphaRho,
					      thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
					      1,
					      thrust::raw_pointer_cast(&rhoWfcContributionsDevice[0]),
					      BVec,
					      &scalarCoeffBetaRho,
					      thrust::raw_pointer_cast(&rhoSpinPolarizedDevice[0]),
					      1);

				  
				  if(isEvaluateGradRho)
				  {
					      
					hipblasDgemm(operatorMatrix.getCublasHandle(),
						    HIPBLAS_OP_N,
						    HIPBLAS_OP_N,
						    1,
						    totalLocallyOwnedCells*numQuadPoints,
						    BVec,
						    &scalarCoeffAlphaGradRho,
						    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
						    1,
						    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceX[0]),
						    BVec,
						    &scalarCoeffBetaGradRho,
						    thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceX[0]),
						    1);


					hipblasDgemm(operatorMatrix.getCublasHandle(),
						    HIPBLAS_OP_N,
						    HIPBLAS_OP_N,
						    1,
						    totalLocallyOwnedCells*numQuadPoints,
						    BVec,
						    &scalarCoeffAlphaGradRho,
						    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
						    1,
						    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceY[0]),
						    BVec,
						    &scalarCoeffBetaGradRho,
						    thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceY[0]),
						    1);

					hipblasDgemm(operatorMatrix.getCublasHandle(),
						    HIPBLAS_OP_N,
						    HIPBLAS_OP_N,
						    1,
						    totalLocallyOwnedCells*numQuadPoints,
						    BVec,
						    &scalarCoeffAlphaGradRho,
						    thrust::raw_pointer_cast(&partialOccupVecDevice[0]),
						    1,
						    thrust::raw_pointer_cast(&gradRhoWfcContributionsDeviceZ[0]),
						    BVec,
						    &scalarCoeffBetaGradRho,
						    thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceZ[0]),
						    1);
					 
					 
				   }
                            }
			     
               
           }


           //do cuda memcopy to host
           std::vector<double> rhoHost;
           std::vector<double> gradRhoHostX;
           std::vector<double> gradRhoHostY;
           std::vector<double> gradRhoHostZ;

           std::vector<double> rhoSpinPolarizedHost;
           std::vector<double> gradRhoSpinPolarizedHostX;
           std::vector<double> gradRhoSpinPolarizedHostY;
           std::vector<double> gradRhoSpinPolarizedHostZ;

           if(dftParameters::spinPolarized==1)
           {
                   rhoSpinPolarizedHost.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
                   hipMemcpy(&rhoSpinPolarizedHost[0],
                              thrust::raw_pointer_cast(&rhoSpinPolarizedDevice[0]),
                              totalLocallyOwnedCells*numQuadPoints*sizeof(double),
                              hipMemcpyDeviceToHost);

                   if(isEvaluateGradRho)
                   {
			   gradRhoSpinPolarizedHostX.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
			   hipMemcpy(&gradRhoSpinPolarizedHostX[0],
				      thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceX[0]),
				      totalLocallyOwnedCells*numQuadPoints*sizeof(double),
				      hipMemcpyDeviceToHost);

			   gradRhoSpinPolarizedHostY.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
			   hipMemcpy(&gradRhoSpinPolarizedHostY[0],
				      thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceY[0]),
				      totalLocallyOwnedCells*numQuadPoints*sizeof(double),
				      hipMemcpyDeviceToHost);

			   gradRhoSpinPolarizedHostZ.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
			   hipMemcpy(&gradRhoSpinPolarizedHostZ[0],
				      thrust::raw_pointer_cast(&gradRhoSpinPolarizedDeviceZ[0]),
				      totalLocallyOwnedCells*numQuadPoints*sizeof(double),
				      hipMemcpyDeviceToHost);
                   }
           }
	   
           rhoHost.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
	   hipMemcpy(&rhoHost[0],
		      thrust::raw_pointer_cast(&rhoDevice[0]),
		      totalLocallyOwnedCells*numQuadPoints*sizeof(double),
		      hipMemcpyDeviceToHost);

           if(isEvaluateGradRho)
           {
		   gradRhoHostX.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
		   hipMemcpy(&gradRhoHostX[0],
			      thrust::raw_pointer_cast(&gradRhoDeviceX[0]),
			      totalLocallyOwnedCells*numQuadPoints*sizeof(double),
			      hipMemcpyDeviceToHost);

		   gradRhoHostY.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
		   hipMemcpy(&gradRhoHostY[0],
			      thrust::raw_pointer_cast(&gradRhoDeviceY[0]),
			      totalLocallyOwnedCells*numQuadPoints*sizeof(double),
			      hipMemcpyDeviceToHost);

		   gradRhoHostZ.resize(totalLocallyOwnedCells*numQuadPoints,0.0);
		   hipMemcpy(&gradRhoHostZ[0],
			      thrust::raw_pointer_cast(&gradRhoDeviceZ[0]),
			      totalLocallyOwnedCells*numQuadPoints*sizeof(double),
			      hipMemcpyDeviceToHost);
           }

           unsigned int iElem=0;
           typename dealii::DoFHandler<3>::active_cell_iterator cell=dofHandler.begin_active();
           typename dealii::DoFHandler<3>::active_cell_iterator endc = dofHandler.end();
	   for(; cell!=endc; ++cell)
	      if(cell->is_locally_owned())
	       {
                 const dealii::CellId cellid=cell->id();
		 for (unsigned int q=0; q<numQuadPoints; ++q)
		 {
		    if(dftParameters::spinPolarized==1)
		    {
                             
			    (*rhoValuesSpinPolarized)[cellid][2*q]=rhoHost[iElem*numQuadPoints+q];
			    (*rhoValuesSpinPolarized)[cellid][2*q+1]=rhoSpinPolarizedHost[iElem*numQuadPoints+q];

			    if(isEvaluateGradRho)
                            {
                                 (*gradRhoValuesSpinPolarized)[cellid][6*q]= gradRhoHostX[iElem*numQuadPoints+q];
                                 (*gradRhoValuesSpinPolarized)[cellid][6*q+3]= gradRhoSpinPolarizedHostX[iElem*numQuadPoints+q];
                                 (*gradRhoValuesSpinPolarized)[cellid][6*q+1]= gradRhoHostY[iElem*numQuadPoints+q];
                                 (*gradRhoValuesSpinPolarized)[cellid][6*q+3+1]= gradRhoSpinPolarizedHostY[iElem*numQuadPoints+q];
                                 (*gradRhoValuesSpinPolarized)[cellid][6*q+2]= gradRhoHostZ[iElem*numQuadPoints+q];
                                 (*gradRhoValuesSpinPolarized)[cellid][6*q+3+2]= gradRhoSpinPolarizedHostZ[iElem*numQuadPoints+q];

                            }

			    (*rhoValues)[cellid][q]= rhoHost[iElem*numQuadPoints+q] 
                                                     + rhoSpinPolarizedHost[iElem*numQuadPoints+q];

                            if(isEvaluateGradRho)
                            {
                                 (*gradRhoValues)[cellid][3*q]= gradRhoHostX[iElem*numQuadPoints+q]
                                       + gradRhoSpinPolarizedHostX[iElem*numQuadPoints+q];
                                 (*gradRhoValues)[cellid][3*q+1]= gradRhoHostY[iElem*numQuadPoints+q]
                                       + gradRhoSpinPolarizedHostY[iElem*numQuadPoints+q];
                                 (*gradRhoValues)[cellid][3*q+2]= gradRhoHostZ[iElem*numQuadPoints+q]
                                       + gradRhoSpinPolarizedHostZ[iElem*numQuadPoints+q];
                            }

                           
		     }
		     else
		     {
			    (*rhoValues)[cellid][q] = rhoHost[iElem*numQuadPoints+q];

                             
			     if(isEvaluateGradRho)
                             {
				 (*gradRhoValues)[cellid][3*q]= gradRhoHostX[iElem*numQuadPoints+q];
                                 (*gradRhoValues)[cellid][3*q+1]= gradRhoHostY[iElem*numQuadPoints+q];
                                 (*gradRhoValues)[cellid][3*q+2]= gradRhoHostZ[iElem*numQuadPoints+q];
                             }
                             
		     }
		 }
                 iElem++;
               }

           

           //gather density from all inter communicators
           sumRhoData(dofHandler,
                      rhoValues,
                      gradRhoValues,
                      rhoValuesSpinPolarized,
                      gradRhoValuesSpinPolarized,
                      isEvaluateGradRho,
                      interBandGroupComm);

           sumRhoData(dofHandler,
                      rhoValues,
                      gradRhoValues,
                      rhoValuesSpinPolarized,
                      gradRhoValuesSpinPolarized,
                      isEvaluateGradRho,
                      interpoolcomm);

	   hipDeviceSynchronize();
           MPI_Barrier(MPI_COMM_WORLD);
	   gpu_time = MPI_Wtime() - gpu_time;
 
           if (this_process==0 && dftParameters::verbosity>=2)
	      std::cout<<"Time for compute rho on GPU: "<<gpu_time<<std::endl;
#endif
      }

   }
}
