#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2018 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Phani Motamarri, Sambit Das

#include <chebyshevOrthogonalizedSubspaceIterationSolverCUDA.h>
#include <linearAlgebraOperations.h>
#include <linearAlgebraOperationsCUDA.h>
#include <vectorUtilities.h>
#include <dftUtils.h>
#include <dftParameters.h>

static const unsigned int order_lookup[][2] = {
  {500, 24}, // <= 500 ~> chebyshevOrder = 24
  {750, 30},
  {1000, 39},
  {750, 30},
  {1000, 39},
  {1500, 50},
  {2000, 53},
  {3000, 57},
  {4000, 62},
  {5000, 69},
  {9000, 77},
  {14000, 104},
  {20000, 119},
  {30000, 162},
  {50000, 300},
  {80000, 450},
  {100000, 550},
  {200000, 700},
  {500000, 1000}
};

namespace dftfe
{


   namespace 
   {
	__global__
	void stridedCopyToBlockKernel(const unsigned int BVec, 
				    const unsigned int M, 
				    const double *xVec,
				    const unsigned int N,
				    double *yVec,
				    const unsigned int startingXVecId)
	{
	  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	  const unsigned int numGangsPerBVec
		    =(BVec+blockDim.x-1)/blockDim.x;
	  const unsigned int gangBlockId=blockIdx.x/numGangsPerBVec;
	  const unsigned int localThreadId=globalThreadId-gangBlockId*numGangsPerBVec*blockDim.x;

	  if (globalThreadId<M*numGangsPerBVec*blockDim.x && localThreadId<BVec)
	  {
	    *(yVec+gangBlockId*BVec+localThreadId)=*(xVec+gangBlockId*N+startingXVecId+localThreadId); 
	  }
	}

	__global__
	void stridedCopyFromBlockKernel(const unsigned int BVec, 
					const unsigned int M, 
					const double *xVec,
					const unsigned int N,
					double *yVec,
					const unsigned int startingXVecId)
	{
	  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	  const unsigned int numGangsPerBVec
		    =(BVec+blockDim.x-1)/blockDim.x;
	  const unsigned int gangBlockId=blockIdx.x/numGangsPerBVec;
	  const unsigned int localThreadId=globalThreadId-gangBlockId*numGangsPerBVec*blockDim.x;

	  if (globalThreadId<M*numGangsPerBVec*blockDim.x && localThreadId<BVec)
	  {
	     *(yVec+gangBlockId*N+startingXVecId+localThreadId) = *(xVec+gangBlockId*BVec+localThreadId);
	  }
	}


	__global__
	void scaleCUDAKernel(const unsigned int contiguousBlockSize,
			   const unsigned int numContiguousBlocks,
			   const double scalar,
			   double *srcArray,
			   const double *scalingVector)
	{

	    const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	    const unsigned int numGangsPerContiguousBlock = (contiguousBlockSize + (blockDim.x-1))/blockDim.x;
	    const unsigned int gangBlockId = blockIdx.x/numGangsPerContiguousBlock;
	    const unsigned int localThreadId = globalThreadId-gangBlockId*numGangsPerContiguousBlock*blockDim.x;
	    if(globalThreadId < numContiguousBlocks*numGangsPerContiguousBlock*blockDim.x && localThreadId < contiguousBlockSize)
	      {
		*(srcArray+(localThreadId+gangBlockId*contiguousBlockSize)) = *(srcArray+(localThreadId+gangBlockId*contiguousBlockSize)) * (*(scalingVector+gangBlockId)*scalar); 

	      }

	}

	__global__
	void setZeroKernel(const unsigned int BVec, 
		           const unsigned int M, 
			   const unsigned int N,
			   double *yVec,
			   const unsigned int startingXVecId)
	{
	  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	  const unsigned int numGangsPerBVec
		    =(BVec+blockDim.x-1)/blockDim.x;
	  const unsigned int gangBlockId=blockIdx.x/numGangsPerBVec;
	  const unsigned int localThreadId=globalThreadId-gangBlockId*numGangsPerBVec*blockDim.x;

	  if (globalThreadId<M*numGangsPerBVec*blockDim.x && localThreadId<BVec)
	  {
	    *(yVec+gangBlockId*N+startingXVecId+localThreadId)=0.0;  
	  }
	}

	namespace internal
	{
	      unsigned int setChebyshevOrder(const unsigned int upperBoundUnwantedSpectrum)
	      {
		for(int i=0; i<sizeof(order_lookup)/sizeof(order_lookup[0]); i++) {
		  if(upperBoundUnwantedSpectrum <= order_lookup[i][0])
		    return order_lookup[i][1];
		}
		return 1250;
	      }
	}
   }

  //
  // Constructor.
  //
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::chebyshevOrthogonalizedSubspaceIterationSolverCUDA
  (const MPI_Comm &mpi_comm,
   double lowerBoundWantedSpectrum,
   double lowerBoundUnWantedSpectrum):
    d_lowerBoundWantedSpectrum(lowerBoundWantedSpectrum),
    d_lowerBoundUnWantedSpectrum(lowerBoundUnWantedSpectrum),
    pcout(std::cout, (dealii::Utilities::MPI::this_mpi_process(MPI_COMM_WORLD) == 0)),
    computing_timer(mpi_comm,
	            pcout,
		    dftParameters::reproducible_output ||
		    dftParameters::verbosity<4? dealii::TimerOutput::never : dealii::TimerOutput::summary,
		    dealii::TimerOutput::wall_times)
  {

  }

  //
  // Destructor.
  //
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::~chebyshevOrthogonalizedSubspaceIterationSolverCUDA()
  {

    //
    //
    //
    return;

  }

  //
  //reinitialize spectrum bounds
  //
  void
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::reinitSpectrumBounds(double lowerBoundWantedSpectrum,
								       double lowerBoundUnWantedSpectrum)
  {
    d_lowerBoundWantedSpectrum = lowerBoundWantedSpectrum;
    d_lowerBoundUnWantedSpectrum = lowerBoundUnWantedSpectrum;
  }


  //
  // solve
  //
  void
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::solve(operatorDFTCUDAClass  & operatorMatrix,
							    double* eigenVectorsFlattenedCUDA,
                                                            double* eigenVectorsRotFracDensityFlattenedCUDA,
                                                            const unsigned int flattenedSize,
							    vectorType  & tempEigenVec,
							    const unsigned int totalNumberWaveFunctions,
							    std::vector<double>        & eigenValues,
							    std::vector<double>        & residualNorms,
							    const MPI_Comm &interBandGroupComm,
                                                            dealii::ScaLAPACKMatrix<double> & projHamPar,
                                                            dealii::ScaLAPACKMatrix<double> & overlapMatPar,
                                                            const std::shared_ptr< const dealii::Utilities::MPI::ProcessGrid> & processGrid,
                                                            const bool useMixedPrecOverall,
                                                            const bool isFirstScf,
                                                            const bool useFullMassMatrixGEP,
                                                            const bool isElpaStep1,
                                                            const bool isElpaStep2)
  {
#ifdef USE_COMPLEX
        AssertThrow(false,dftUtils::ExcNotImplementedYet());
#else
    double gpu_time, start_time, sub_gpu_time;
    int this_process;

    MPI_Comm_rank(MPI_COMM_WORLD, &this_process);


    hipblasHandle_t & cublasHandle =
    operatorMatrix.getCublasHandle();

    //
    //allocate memory for full flattened array on device and fill it up
    //
    const unsigned int localVectorSize = flattenedSize/totalNumberWaveFunctions;
    
    hipDeviceSynchronize(); 
    MPI_Barrier(MPI_COMM_WORLD);
    start_time = MPI_Wtime();
   
    //band group parallelization data structures
    const unsigned int numberBandGroups=
      dealii::Utilities::MPI::n_mpi_processes(interBandGroupComm);


    const unsigned int bandGroupTaskId = dealii::Utilities::MPI::this_mpi_process(interBandGroupComm);
    std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
    dftUtils::createBandParallelizationIndices(interBandGroupComm,
					       totalNumberWaveFunctions,
					       bandGroupLowHighPlusOneIndices);


    const unsigned int vectorsBlockSize=std::min(dftParameters::chebyWfcBlockSize,
						 totalNumberWaveFunctions);

    cudaVectorType cudaFlattenedArrayBlock;
    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
				    vectorsBlockSize,
				    cudaFlattenedArrayBlock);


    cudaVectorType YArray;
    YArray.reinit(cudaFlattenedArrayBlock);

    cudaVectorTypeFloat cudaFlattenedFloatArrayBlock;
    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
                                    vectorsBlockSize,
                                    cudaFlattenedFloatArrayBlock);


    cudaVectorType projectorKetTimesVector;
    vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
				    vectorsBlockSize,
				    projectorKetTimesVector);


    cudaVectorType cudaFlattenedArrayBlock2;
    if (dftParameters::overlapComputeCommunCheby || dftParameters::chebyCommunAvoidanceAlgo)
           cudaFlattenedArrayBlock2.reinit(cudaFlattenedArrayBlock);


    cudaVectorType YArray2;
    if (dftParameters::overlapComputeCommunCheby)
           YArray2.reinit(cudaFlattenedArrayBlock2);


    cudaVectorType projectorKetTimesVector2;
    if (dftParameters::overlapComputeCommunCheby)
           projectorKetTimesVector2.reinit(projectorKetTimesVector);

    if(!isElpaStep2)
    {
	    computing_timer.enter_section("Lanczos k-step Upper Bound");
	    operatorMatrix.reinit(1);
	    const double upperBoundUnwantedSpectrum =linearAlgebraOperationsCUDA::lanczosUpperBoundEigenSpectrum(operatorMatrix,
													      tempEigenVec);
	    computing_timer.exit_section("Lanczos k-step Upper Bound");
            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
	    gpu_time = MPI_Wtime();
	    unsigned int chebyshevOrder = dftParameters::chebyshevOrder;

	    //
	    //set Chebyshev order
	    //
	    if(chebyshevOrder == 0)
	      chebyshevOrder=internal::setChebyshevOrder(upperBoundUnwantedSpectrum);

            chebyshevOrder = (isFirstScf && dftParameters::isPseudopotential)?chebyshevOrder*dftParameters::chebyshevFilterPolyDegreeFirstScfScalingFactor:chebyshevOrder;


	    if(dftParameters::lowerBoundUnwantedFracUpper > 1e-6)
	      d_lowerBoundUnWantedSpectrum=dftParameters::lowerBoundUnwantedFracUpper*upperBoundUnwantedSpectrum;

	    //
	    //output statements
	    //
	    if (dftParameters::verbosity>=2)
	      {
		char buffer[100];

		sprintf(buffer, "%s:%18.10e\n", "upper bound of unwanted spectrum", upperBoundUnwantedSpectrum);
		pcout << buffer;
		sprintf(buffer, "%s:%18.10e\n", "lower bound of unwanted spectrum", d_lowerBoundUnWantedSpectrum);
		pcout << buffer;
		sprintf(buffer, "%s: %u\n\n", "Chebyshev polynomial degree", chebyshevOrder);
		pcout << buffer;
	      }


	    //
	    //scale the eigenVectors (initial guess of single atom wavefunctions or previous guess) to convert into Lowden Orthonormalized FE basis
	    //multiply by M^{1/2}
	    scaleCUDAKernel<<<(totalNumberWaveFunctions+255)/256*localVectorSize,256>>>(totalNumberWaveFunctions,
										        localVectorSize,
											1.0,
											eigenVectorsFlattenedCUDA,
											operatorMatrix.getSqrtMassVec());

            //two blocks of wavefunctions are filtered simultaneously when overlap compute communication in chebyshev
            //filtering is toggled on
            const unsigned int numSimultaneousBlocks=dftParameters::overlapComputeCommunCheby?2:1;
            unsigned int numSimultaneousBlocksCurrent=numSimultaneousBlocks;
            const unsigned int numWfcsInBandGroup=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1]-bandGroupLowHighPlusOneIndices[2*bandGroupTaskId];
            int startIndexBandParal=totalNumberWaveFunctions;
            int numVectorsBandParal=0;
	    for (unsigned int jvec = 0; jvec < totalNumberWaveFunctions; jvec += numSimultaneousBlocksCurrent*vectorsBlockSize)
	    {

		// Correct block dimensions if block "goes off edge of" the matrix
		const unsigned int BVec = vectorsBlockSize;//std::min(vectorsBlockSize, totalNumberWaveFunctions-jvec);
              
                //handle edge case when total number of blocks in a given band group is not even in case of 
                //overlapping computation and communciation in chebyshev filtering 
                const unsigned int leftIndexBandGroupMargin=(jvec/numWfcsInBandGroup)*numWfcsInBandGroup;
                numSimultaneousBlocksCurrent
                     =((jvec+numSimultaneousBlocks*BVec-leftIndexBandGroupMargin)<=numWfcsInBandGroup && numSimultaneousBlocks==2)?2:1;

        	if ((jvec+numSimultaneousBlocksCurrent*BVec)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
	         (jvec+numSimultaneousBlocksCurrent*BVec)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
		{

	                if (jvec<startIndexBandParal)
		           startIndexBandParal=jvec;
	                numVectorsBandParal= jvec+numSimultaneousBlocksCurrent*BVec-startIndexBandParal;
			
                        //copy from vector containg all wavefunction vectors to current wavefunction vectors block
			stridedCopyToBlockKernel<<<(BVec+255)/256*localVectorSize, 256>>>(BVec,
											  localVectorSize,
											  eigenVectorsFlattenedCUDA,
											  totalNumberWaveFunctions,
											  cudaFlattenedArrayBlock.begin(),
											  jvec);

                        if (dftParameters::overlapComputeCommunCheby && numSimultaneousBlocksCurrent==2)
				stridedCopyToBlockKernel<<<(BVec+255)/256*localVectorSize, 256>>>(BVec,
												  localVectorSize,
												  eigenVectorsFlattenedCUDA,
												  totalNumberWaveFunctions,
												  cudaFlattenedArrayBlock2.begin(),
												  jvec+BVec);
			  
			 //
			 //call Chebyshev filtering function only for the current block or two simulataneous blocks
                         //(in case of overlap computation and communication) to be filtered and does in-place filtering
                         if (dftParameters::overlapComputeCommunCheby && numSimultaneousBlocksCurrent==2)
				 linearAlgebraOperationsCUDA::chebyshevFilter(operatorMatrix,
									      cudaFlattenedArrayBlock,
									      YArray,
									      cudaFlattenedFloatArrayBlock,
									      projectorKetTimesVector,
									      cudaFlattenedArrayBlock2,
									      YArray2,
									      projectorKetTimesVector2,
									      localVectorSize,
									      BVec,
									      chebyshevOrder,
									      d_lowerBoundUnWantedSpectrum,
									      upperBoundUnwantedSpectrum,
									      d_lowerBoundWantedSpectrum);	
                         else if (dftParameters::chebyCommunAvoidanceAlgo)
				 linearAlgebraOperationsCUDA::chebyshevFilterCommunAvoidance(operatorMatrix,
										   cudaFlattenedArrayBlock,
										   YArray,
                                                                                   cudaFlattenedArrayBlock2,
										   cudaFlattenedFloatArrayBlock,
										   projectorKetTimesVector,
										   localVectorSize,
										   BVec,
										   chebyshevOrder,
										   d_lowerBoundUnWantedSpectrum,
										   upperBoundUnwantedSpectrum,
										   d_lowerBoundWantedSpectrum);	
                         else 
				 linearAlgebraOperationsCUDA::chebyshevFilter(operatorMatrix,
										   cudaFlattenedArrayBlock,
										   YArray,
										   cudaFlattenedFloatArrayBlock,
										   projectorKetTimesVector,
										   localVectorSize,
										   BVec,
										   chebyshevOrder,
										   d_lowerBoundUnWantedSpectrum,
										   upperBoundUnwantedSpectrum,
										   d_lowerBoundWantedSpectrum);								  
		       //copy current wavefunction vectors block to vector containing all wavefunction vectors
		       stridedCopyFromBlockKernel<<<(BVec+255)/256*localVectorSize, 256>>>(BVec,
											   localVectorSize,
											   cudaFlattenedArrayBlock.begin(),
											   totalNumberWaveFunctions,
											   eigenVectorsFlattenedCUDA,
											   jvec);

                       if (dftParameters::overlapComputeCommunCheby && numSimultaneousBlocksCurrent==2)
			       stridedCopyFromBlockKernel<<<(BVec+255)/256*localVectorSize, 256>>>(BVec,
												   localVectorSize,
												   cudaFlattenedArrayBlock2.begin(),
												   totalNumberWaveFunctions,
												   eigenVectorsFlattenedCUDA,
												   jvec+BVec);
		}
                else
                {
                      //set to zero wavefunctions which wont go through chebyshev filtering inside a given band group
	              setZeroKernel<<<(numSimultaneousBlocksCurrent*BVec+255)/256*localVectorSize, 256>>>(numSimultaneousBlocksCurrent*BVec,
								             localVectorSize,
									     totalNumberWaveFunctions,
									     eigenVectorsFlattenedCUDA,
									     jvec);
                }

	    }//block loop

	    hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
	    gpu_time = MPI_Wtime() - gpu_time;
	    if (this_process==0 && dftParameters::verbosity>=2)
	        std::cout<<"Time for chebyshev filtering on GPU: "<<gpu_time<<std::endl;


	    if(dftParameters::verbosity >= 4)
	      pcout<<"ChebyShev Filtering Done: "<<std::endl;


            if (numberBandGroups>1)
            {
		    hipDeviceSynchronize();
                    MPI_Barrier(MPI_COMM_WORLD);
		    double band_paral_time=MPI_Wtime();

                    std::vector<double> eigenVectorsFlattened(totalNumberWaveFunctions*localVectorSize,0);

                    //hipDeviceSynchronize();
                    //double copytime=MPI_Wtime();
		    hipMemcpy(&eigenVectorsFlattened[0],
			       eigenVectorsFlattenedCUDA,
			       totalNumberWaveFunctions*localVectorSize*sizeof(double),
			       hipMemcpyDeviceToHost);
                    //hipDeviceSynchronize();
                    //copytime = MPI_Wtime() - copytime;
	            //if (this_process==0)
	            //   std::cout<<"copy time on GPU: "<<copytime<<std::endl;

		    MPI_Barrier(interBandGroupComm);

                    if (true)
                    {
		  	    MPI_Allreduce(MPI_IN_PLACE,
				      &eigenVectorsFlattened[0],
				      totalNumberWaveFunctions*localVectorSize,
				      MPI_DOUBLE,
				      MPI_SUM,
				      interBandGroupComm);
                    }
                    else
                    {
			    std::vector<double> eigenVectorsBandGroup(numVectorsBandParal*localVectorSize,0);
			    std::vector<double> eigenVectorsBandGroupTransposed(numVectorsBandParal*localVectorSize,0);
			    std::vector<double> eigenVectorsTransposed(totalNumberWaveFunctions*localVectorSize,0);

			    for(unsigned int iNode = 0; iNode < localVectorSize; ++iNode)
			       for(unsigned int iWave = 0; iWave < numVectorsBandParal; ++iWave)
				   eigenVectorsBandGroup[iNode*numVectorsBandParal+iWave]
				     = eigenVectorsFlattened[iNode*totalNumberWaveFunctions+startIndexBandParal+iWave];

			    
			    for(unsigned int iNode = 0; iNode < localVectorSize; ++iNode)
			       for(unsigned int iWave = 0; iWave < numVectorsBandParal; ++iWave)
				   eigenVectorsBandGroupTransposed[iWave*localVectorSize+iNode]
				     = eigenVectorsBandGroup[iNode*numVectorsBandParal+iWave];

			    std::vector<int> recvcounts(numberBandGroups,0);
			    std::vector<int> displs(numberBandGroups,0);

			    int recvcount=numVectorsBandParal*localVectorSize;
			    MPI_Allgather(&recvcount,
					  1,
					  MPI_INT,
					  &recvcounts[0],
					  1,
					  MPI_INT,
					  interBandGroupComm);

			    int displ=startIndexBandParal*localVectorSize;
			    MPI_Allgather(&displ,
					  1,
					  MPI_INT,
					  &displs[0],
					  1,
					  MPI_INT,
					  interBandGroupComm);

			    MPI_Allgatherv(&eigenVectorsBandGroupTransposed[0],
					   numVectorsBandParal*localVectorSize,
					   MPI_DOUBLE,
					   &eigenVectorsTransposed[0],
					   &recvcounts[0],
					   &displs[0],
					   dataTypes::mpi_type_id(&eigenVectorsTransposed[0]),
					   interBandGroupComm);

			    
			    for(unsigned int iNode = 0; iNode < localVectorSize; ++iNode)
			       for(unsigned int iWave = 0; iWave < totalNumberWaveFunctions; ++iWave)
				   eigenVectorsFlattened[iNode*totalNumberWaveFunctions+iWave]
				     = eigenVectorsTransposed[iWave*localVectorSize+iNode];
		    }
		    MPI_Barrier(interBandGroupComm);

                    //hipDeviceSynchronize();
                    //copytime=MPI_Wtime();
		    hipMemcpy(eigenVectorsFlattenedCUDA,
			       &eigenVectorsFlattened[0],
			       totalNumberWaveFunctions*localVectorSize*sizeof(double),
			       hipMemcpyHostToDevice);
                    //hipDeviceSynchronize();
                    //copytime = MPI_Wtime() - copytime;
	            //if (this_process==0)
	            //   std::cout<<"copy time on GPU: "<<copytime<<std::endl;
		    hipDeviceSynchronize();
                    MPI_Barrier(MPI_COMM_WORLD);
		    band_paral_time = MPI_Wtime() - band_paral_time;

		    if (this_process==0 && dftParameters::verbosity>=2)
			std::cout<<"Time for band parallelization communication: "<<band_paral_time<<std::endl;

            }

            //if (dftParameters::measureOnlyChebyTime)
            //  exit(0);

	    /*
	    int inc=1;
	    double result=0.0;
	    hipblasDnrm2(cublasHandle,
			flattenedSize,
			eigenVectorsFlattenedCUDA,
			inc, 
			&result);
	    result=result*result;
	    result=dealii::Utilities::MPI::sum(result,operatorMatrix.getMPICommunicator());
	    std::cout<<"l2 norm Chebyshev filtered x: "<<std::sqrt(result)<<std::endl;
	    */

            if (dftParameters::rrGEP==false)
            {
		    if(dftParameters::orthogType.compare("LW") == 0)
		    {

			AssertThrow(false,dealii::ExcMessage("Lowden Gram-Schmidt Orthonormalization Not implemented in CUDA:"));

		    }
		    else if (dftParameters::orthogType.compare("PGS") == 0)
		    {
			//gpu_time = MPI_Wtime();
			linearAlgebraOperationsCUDA::pseudoGramSchmidtOrthogonalization
				     (operatorMatrix,
				      eigenVectorsFlattenedCUDA,
				      localVectorSize,
				      totalNumberWaveFunctions,
				      operatorMatrix.getMPICommunicator(),
                                      interBandGroupComm,
				      cublasHandle,
				      useMixedPrecOverall);
				    
			//gpu_time = MPI_Wtime() - gpu_time;
			//if (this_process==0)
			//    std::cout<<"Time for PGS on GPU: "<<gpu_time<<std::endl;

		    }
		    else if (dftParameters::orthogType.compare("GS") == 0)
		    {

			AssertThrow(false,dealii::ExcMessage("Classical Gram-Schmidt Orthonormalization not implemented in CUDA:"));

		    }

		    if(dftParameters::verbosity >= 4)
		      pcout<<"Orthogonalization Done: "<<std::endl;

	    }

	    //gpu_time = MPI_Wtime();
	    for (unsigned int i=0;i<eigenValues.size();i++)
		   eigenValues[i]=0.0;

    }

    if (eigenValues.size()!=totalNumberWaveFunctions)
    {
            if (dftParameters::rrGEP==false)             
		    linearAlgebraOperationsCUDA::rayleighRitzSpectrumSplitDirect(operatorMatrix,
							      eigenVectorsFlattenedCUDA,
							      eigenVectorsRotFracDensityFlattenedCUDA,
							      cudaFlattenedArrayBlock,
                                                              cudaFlattenedFloatArrayBlock,
							      YArray,
							      projectorKetTimesVector,
							      localVectorSize,
							      totalNumberWaveFunctions,
							      totalNumberWaveFunctions-eigenValues.size(),
							      isElpaStep1,
							      isElpaStep2,
							      operatorMatrix.getMPICommunicator(),
							      &eigenValues[0],
							      cublasHandle,
							      projHamPar,
							      processGrid,
							      useMixedPrecOverall);
            else           
		    linearAlgebraOperationsCUDA::rayleighRitzGEPSpectrumSplitDirect(operatorMatrix,
							      eigenVectorsFlattenedCUDA,
							      eigenVectorsRotFracDensityFlattenedCUDA,
							      cudaFlattenedArrayBlock,
                                                              cudaFlattenedFloatArrayBlock,
							      YArray,
							      projectorKetTimesVector,
							      localVectorSize,
							      totalNumberWaveFunctions,
							      totalNumberWaveFunctions-eigenValues.size(),
							      isElpaStep1,
							      isElpaStep2,
							      operatorMatrix.getMPICommunicator(),
                                                              interBandGroupComm,
							      &eigenValues[0],
							      cublasHandle,
							      projHamPar,
                                                              overlapMatPar,
							      processGrid,
							      useMixedPrecOverall);

	     if (isElpaStep1)
	     {
		hipDeviceSynchronize();
                MPI_Barrier(MPI_COMM_WORLD);
		gpu_time = MPI_Wtime() - start_time;
		if (this_process==0 && dftParameters::verbosity>=2)
	            std::cout<<"Time for all steps of subspace iteration on GPU till ELPA step 1: "<<gpu_time<<std::endl; 
		return;
	     }

    }
    else
    {
            if (dftParameters::rrGEP==false)
		    linearAlgebraOperationsCUDA::rayleighRitz(operatorMatrix,
							      eigenVectorsFlattenedCUDA,
							      cudaFlattenedArrayBlock,
                                                              cudaFlattenedFloatArrayBlock,
							      YArray,
							      projectorKetTimesVector,
							      localVectorSize,
							      totalNumberWaveFunctions,
							      isElpaStep1,
							      isElpaStep2,
							      operatorMatrix.getMPICommunicator(),
                                                              interBandGroupComm,
							      &eigenValues[0],
							      cublasHandle,
							      projHamPar,
							      processGrid,
							      useMixedPrecOverall);
            else
		    linearAlgebraOperationsCUDA::rayleighRitzGEP(operatorMatrix,
							      eigenVectorsFlattenedCUDA,
							      cudaFlattenedArrayBlock,
                                                              cudaFlattenedFloatArrayBlock,
							      YArray,
							      projectorKetTimesVector,
							      localVectorSize,
							      totalNumberWaveFunctions,
							      isElpaStep1,
							      isElpaStep2,
							      operatorMatrix.getMPICommunicator(),
                                                              interBandGroupComm,
							      &eigenValues[0],
							      cublasHandle,
							      projHamPar,
                                                              overlapMatPar,
							      processGrid,
							      useMixedPrecOverall);
     

	     if (isElpaStep1)
	     {
		hipDeviceSynchronize();
                MPI_Barrier(MPI_COMM_WORLD);
		gpu_time = MPI_Wtime() - start_time;
		if (this_process==0 && dftParameters::verbosity>=2)
	            std::cout<<"Time for all steps of subspace iteration on GPU till ELPA step 1: "<<gpu_time<<std::endl; 
		return;
	     }

    }
     //gpu_time = MPI_Wtime() - gpu_time;
     //if (this_process==0)
     //    std::cout<<"Time for Rayleigh Ritz on GPU: "<<gpu_time<<std::endl;



    if(dftParameters::verbosity >= 4)
    {
	pcout<<"Rayleigh-Ritz Done: "<<std::endl;
	pcout<<std::endl;
    }

    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time = MPI_Wtime();
    if (eigenValues.size()!=totalNumberWaveFunctions)
      linearAlgebraOperationsCUDA::computeEigenResidualNorm(operatorMatrix,
      						        eigenVectorsRotFracDensityFlattenedCUDA,
						        cudaFlattenedArrayBlock,
						        YArray,
						        projectorKetTimesVector,
						        localVectorSize,
						        eigenValues.size(),
      						        eigenValues,
							operatorMatrix.getMPICommunicator(),
                                                        interBandGroupComm,
                                                        cublasHandle,
      						        residualNorms);
    else
      linearAlgebraOperationsCUDA::computeEigenResidualNorm(operatorMatrix,
      						        eigenVectorsFlattenedCUDA,
						        cudaFlattenedArrayBlock,
						        YArray,
						        projectorKetTimesVector,
						        localVectorSize,
						        totalNumberWaveFunctions,
      						        eigenValues,
							operatorMatrix.getMPICommunicator(),
                                                        interBandGroupComm,
                                                        cublasHandle,
      						        residualNorms,
                                                        true);

    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time = MPI_Wtime() - gpu_time;
    if (this_process==0 && dftParameters::verbosity>=2)
       std::cout<<"Time to compute residual norm: "<<gpu_time<<std::endl;

    //
    //scale the eigenVectors with M^{-1/2} to represent the wavefunctions in the usual FE basis
    //
    scaleCUDAKernel<<<(totalNumberWaveFunctions+255)/256*localVectorSize,256>>>(totalNumberWaveFunctions,
                                                                     localVectorSize,
                                                                     1.0,
                                                                     eigenVectorsFlattenedCUDA,
                                                                     operatorMatrix.getInvSqrtMassVec());

    if (eigenValues.size()!=totalNumberWaveFunctions)
	    scaleCUDAKernel<<<(eigenValues.size()+255)/256*localVectorSize,256>>>(eigenValues.size(),
									     localVectorSize,
									     1.0,
									     eigenVectorsRotFracDensityFlattenedCUDA,
									     operatorMatrix.getInvSqrtMassVec());

    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time = MPI_Wtime() - start_time;

    if (isElpaStep2)
       if (this_process==0 && dftParameters::verbosity>=2)
           std::cout<<"Time for ELPA step 2 on GPU: "<<gpu_time<<std::endl;
    else
       if (this_process==0 && dftParameters::verbosity>=2)
           std::cout<<"Time for all steps of subspace iteration on GPU: "<<gpu_time<<std::endl;
    return;
#endif
  }
}
