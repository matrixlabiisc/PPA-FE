#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2018 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Phani Motamarri, Sambit Das

#include <chebyshevOrthogonalizedSubspaceIterationSolverCUDA.h>
#include <linearAlgebraOperations.h>
#include <linearAlgebraOperationsCUDA.h>
#include <vectorUtilities.h>
#include <dftUtils.h>
#include <dftParameters.h>


namespace dftfe
{


   namespace 
   {
	__global__
	void stridedCopyToBlockKernel(const unsigned int BVec, 
				    const unsigned int M, 
				    const double *xVec,
				    const unsigned int N,
				    double *yVec,
				    const unsigned int startingXVecId)
	{
	  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	  const unsigned int numGangsPerBVec
		    =(BVec+blockDim.x-1)/blockDim.x;
	  const unsigned int gangBlockId=blockIdx.x/numGangsPerBVec;
	  const unsigned int localThreadId=globalThreadId-gangBlockId*numGangsPerBVec*blockDim.x;

	  if (globalThreadId<M*numGangsPerBVec*blockDim.x && localThreadId<BVec)
	  {
	    *(yVec+gangBlockId*BVec+localThreadId)=*(xVec+gangBlockId*N+startingXVecId+localThreadId); 
	  }
	}

	__global__
	void stridedCopyFromBlockKernel(const unsigned int BVec, 
					const unsigned int M, 
					const double *xVec,
					const unsigned int N,
					double *yVec,
					const unsigned int startingXVecId)
	{
	  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	  const unsigned int numGangsPerBVec
		    =(BVec+blockDim.x-1)/blockDim.x;
	  const unsigned int gangBlockId=blockIdx.x/numGangsPerBVec;
	  const unsigned int localThreadId=globalThreadId-gangBlockId*numGangsPerBVec*blockDim.x;

	  if (globalThreadId<M*numGangsPerBVec*blockDim.x && localThreadId<BVec)
	  {
	     *(yVec+gangBlockId*N+startingXVecId+localThreadId) = *(xVec+gangBlockId*BVec+localThreadId);
	  }
	}


	__global__
	void scaleCUDAKernel(const unsigned int contiguousBlockSize,
			   const unsigned int numContiguousBlocks,
			   const double scalar,
			   double *srcArray,
			   const double *scalingVector)
	{

	    const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	    const unsigned int numGangsPerContiguousBlock = (contiguousBlockSize + (blockDim.x-1))/blockDim.x;
	    const unsigned int gangBlockId = blockIdx.x/numGangsPerContiguousBlock;
	    const unsigned int localThreadId = globalThreadId-gangBlockId*numGangsPerContiguousBlock*blockDim.x;
	    if(globalThreadId < numContiguousBlocks*numGangsPerContiguousBlock*blockDim.x && localThreadId < contiguousBlockSize)
	      {
		*(srcArray+(localThreadId+gangBlockId*contiguousBlockSize)) = *(srcArray+(localThreadId+gangBlockId*contiguousBlockSize)) * (*(scalingVector+gangBlockId)*scalar); 

	      }

	}

	__global__
	void setZeroKernel(const unsigned int BVec, 
		           const unsigned int M, 
			   const unsigned int N,
			   double *yVec,
			   const unsigned int startingXVecId)
	{
	  const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	  const unsigned int numGangsPerBVec
		    =(BVec+blockDim.x-1)/blockDim.x;
	  const unsigned int gangBlockId=blockIdx.x/numGangsPerBVec;
	  const unsigned int localThreadId=globalThreadId-gangBlockId*numGangsPerBVec*blockDim.x;

	  if (globalThreadId<M*numGangsPerBVec*blockDim.x && localThreadId<BVec)
	  {
	    *(yVec+gangBlockId*N+startingXVecId+localThreadId)=0.0;  
	  }
	}
   }

  namespace internal
  {
      unsigned int setChebyshevOrder(const unsigned int upperBoundUnwantedSpectrum)
      {
	unsigned int chebyshevOrder;
	if(upperBoundUnwantedSpectrum <= 500)
	  chebyshevOrder = 24;
	else if(upperBoundUnwantedSpectrum > 500  && upperBoundUnwantedSpectrum <= 1000)
	  chebyshevOrder = 30;
	else if(upperBoundUnwantedSpectrum > 1000 && upperBoundUnwantedSpectrum <= 1500)
          chebyshevOrder = 34;		
        else if(upperBoundUnwantedSpectrum > 1500 && upperBoundUnwantedSpectrum <= 2000)
	  chebyshevOrder = 38;
	else if(upperBoundUnwantedSpectrum > 2000 && upperBoundUnwantedSpectrum <= 3000)
	  chebyshevOrder = 45;
	else if(upperBoundUnwantedSpectrum > 3000 && upperBoundUnwantedSpectrum <= 4000)
	  chebyshevOrder = 53;
        else if(upperBoundUnwantedSpectrum > 4000 && upperBoundUnwantedSpectrum <= 5000)
          chebyshevOrder = 60;
	else if(upperBoundUnwantedSpectrum > 5000 && upperBoundUnwantedSpectrum <= 9000)
	  chebyshevOrder = 68;
	else if(upperBoundUnwantedSpectrum > 9000 && upperBoundUnwantedSpectrum <= 14000)
	  chebyshevOrder = 94;
	else if(upperBoundUnwantedSpectrum > 14000 && upperBoundUnwantedSpectrum <= 20000)
	  chebyshevOrder = 109;
	else if(upperBoundUnwantedSpectrum > 20000 && upperBoundUnwantedSpectrum <= 30000)
	  chebyshevOrder = 162;
	else if(upperBoundUnwantedSpectrum > 30000 && upperBoundUnwantedSpectrum <= 50000)
	  chebyshevOrder = 300;
	else if(upperBoundUnwantedSpectrum > 50000 && upperBoundUnwantedSpectrum <= 80000)
	  chebyshevOrder = 450;
	else if(upperBoundUnwantedSpectrum > 80000 && upperBoundUnwantedSpectrum <= 1e5)
	  chebyshevOrder = 550;
	else if(upperBoundUnwantedSpectrum > 1e5 && upperBoundUnwantedSpectrum <= 2e5)
	  chebyshevOrder = 700;
	else if(upperBoundUnwantedSpectrum > 2e5 && upperBoundUnwantedSpectrum <= 5e5)
	  chebyshevOrder = 1000;
	else if(upperBoundUnwantedSpectrum > 5e5)
	  chebyshevOrder = 1250;

	return chebyshevOrder;
      }
  }

  //
  // Constructor.
  //
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::chebyshevOrthogonalizedSubspaceIterationSolverCUDA
  (const MPI_Comm &mpi_comm,
   double lowerBoundWantedSpectrum,
   double lowerBoundUnWantedSpectrum):
    d_lowerBoundWantedSpectrum(lowerBoundWantedSpectrum),
    d_lowerBoundUnWantedSpectrum(lowerBoundUnWantedSpectrum),
    pcout(std::cout, (dealii::Utilities::MPI::this_mpi_process(MPI_COMM_WORLD) == 0)),
    computing_timer(mpi_comm,
	            pcout,
		    dftParameters::reproducible_output ||
		    dftParameters::verbosity<4? dealii::TimerOutput::never : dealii::TimerOutput::summary,
		    dealii::TimerOutput::wall_times)
  {

  }

  //
  // Destructor.
  //
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::~chebyshevOrthogonalizedSubspaceIterationSolverCUDA()
  {

    //
    //
    //
    return;

  }

  //
  //reinitialize spectrum bounds
  //
  void
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::reinitSpectrumBounds(double lowerBoundWantedSpectrum,
								       double lowerBoundUnWantedSpectrum)
  {
    d_lowerBoundWantedSpectrum = lowerBoundWantedSpectrum;
    d_lowerBoundUnWantedSpectrum = lowerBoundUnWantedSpectrum;
  }


  //
  // solve
  //
  void
  chebyshevOrthogonalizedSubspaceIterationSolverCUDA::solve(operatorDFTCUDAClass  & operatorMatrix,
							    double* eigenVectorsFlattenedCUDA,
                                                            double* eigenVectorsRotFracDensityFlattenedCUDA,
                                                            const unsigned int flattenedSize,
							    vectorType  & tempEigenVec,
							    const unsigned int totalNumberWaveFunctions,
							    std::vector<double>        & eigenValues,
							    std::vector<double>        & residualNorms,
							    const MPI_Comm &interBandGroupComm,
                                                            dealii::ScaLAPACKMatrix<double> & projHamPar,
                                                            dealii::ScaLAPACKMatrix<double> & overlapMatPar,
                                                            const std::shared_ptr< const dealii::Utilities::MPI::ProcessGrid> & processGrid,
                                                            const bool useMixedPrecOverall,
                                                            const bool isFirstScf,
                                                            const bool useFullMassMatrixGEP,
                                                            const bool isElpaStep1,
                                                            const bool isElpaStep2)
  {
#ifdef USE_COMPLEX
        AssertThrow(false,dftUtils::ExcNotImplementedYet());
#else
    double gpu_time, start_time, sub_gpu_time;
    int this_process;

    MPI_Comm_rank(MPI_COMM_WORLD, &this_process);


    hipblasHandle_t & cublasHandle =
    operatorMatrix.getCublasHandle();

    //
    //allocate memory for full flattened array on device and fill it up
    //
    const unsigned int localVectorSize = flattenedSize/totalNumberWaveFunctions;
    
    hipDeviceSynchronize(); 
    MPI_Barrier(MPI_COMM_WORLD);
    start_time = MPI_Wtime();
   
    //band group parallelization data structures
    const unsigned int numberBandGroups=
      dealii::Utilities::MPI::n_mpi_processes(interBandGroupComm);


    const unsigned int bandGroupTaskId = dealii::Utilities::MPI::this_mpi_process(interBandGroupComm);
    std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
    dftUtils::createBandParallelizationIndices(interBandGroupComm,
					       totalNumberWaveFunctions,
					       bandGroupLowHighPlusOneIndices);


    const unsigned int vectorsBlockSize=std::min(dftParameters::chebyWfcBlockSize,
						 totalNumberWaveFunctions);

    cudaVectorType cudaFlattenedArrayBlock;
    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
				    vectorsBlockSize,
				    cudaFlattenedArrayBlock);


    cudaVectorType YArray;
    YArray.reinit(cudaFlattenedArrayBlock);

    cudaVectorTypeFloat cudaFlattenedFloatArrayBlock;
    vectorTools::createDealiiVector(operatorMatrix.getMatrixFreeData()->get_vector_partitioner(),
                                    vectorsBlockSize,
                                    cudaFlattenedFloatArrayBlock);


    cudaVectorType projectorKetTimesVector;
    vectorTools::createDealiiVector(operatorMatrix.getProjectorKetTimesVectorSingle().get_partitioner(),
				    vectorsBlockSize,
				    projectorKetTimesVector);



    if(!isElpaStep2)
    {
	    computing_timer.enter_section("Lanczos k-step Upper Bound");
	    operatorMatrix.reinit(1);
	    const double upperBoundUnwantedSpectrum =linearAlgebraOperationsCUDA::lanczosUpperBoundEigenSpectrum(operatorMatrix,
													      tempEigenVec);
	    computing_timer.exit_section("Lanczos k-step Upper Bound");
            hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
	    gpu_time = MPI_Wtime();
	    unsigned int chebyshevOrder = dftParameters::chebyshevOrder;

	    //
	    //set Chebyshev order
	    //
	    if(chebyshevOrder == 0)
	      chebyshevOrder=internal::setChebyshevOrder(upperBoundUnwantedSpectrum);

            chebyshevOrder = (isFirstScf && dftParameters::isPseudopotential)?chebyshevOrder*1.34:chebyshevOrder;


	    if(dftParameters::lowerBoundUnwantedFracUpper > 1e-6)
	      d_lowerBoundUnWantedSpectrum=dftParameters::lowerBoundUnwantedFracUpper*upperBoundUnwantedSpectrum;

	    //
	    //output statements
	    //
	    if (dftParameters::verbosity>=2)
	      {
		char buffer[100];

		sprintf(buffer, "%s:%18.10e\n", "upper bound of unwanted spectrum", upperBoundUnwantedSpectrum);
		pcout << buffer;
		sprintf(buffer, "%s:%18.10e\n", "lower bound of unwanted spectrum", d_lowerBoundUnWantedSpectrum);
		pcout << buffer;
		sprintf(buffer, "%s: %u\n\n", "Chebyshev polynomial degree", chebyshevOrder);
		pcout << buffer;
	      }


	    //
	    //scale the eigenVectors (initial guess of single atom wavefunctions or previous guess) to convert into Lowden Orthonormalized FE basis
	    //multiply by M^{1/2}
	    scaleCUDAKernel<<<(totalNumberWaveFunctions+255)/256*localVectorSize,256>>>(totalNumberWaveFunctions,
										        localVectorSize,
											1.0,
											eigenVectorsFlattenedCUDA,
											operatorMatrix.getSqrtMassVec());

	    //
	    //Set the constraints to zero
	    //
	    //operatorMatrix.getOverloadedConstraintMatrix()->set_zero(eigenVectorsFlattened,
	    //	                                                    totalNumberWaveFunctions);

            int startIndexBandParal=totalNumberWaveFunctions;
            int numVectorsBandParal=0;
	    for (unsigned int jvec = 0; jvec < totalNumberWaveFunctions; jvec += vectorsBlockSize)
	    {

		// Correct block dimensions if block "goes off edge of" the matrix
		const unsigned int BVec = std::min(vectorsBlockSize, totalNumberWaveFunctions-jvec);


        	if ((jvec+BVec)<=bandGroupLowHighPlusOneIndices[2*bandGroupTaskId+1] &&
	         (jvec+BVec)>bandGroupLowHighPlusOneIndices[2*bandGroupTaskId])
		{

	                if (jvec<startIndexBandParal)
		           startIndexBandParal=jvec;
	                numVectorsBandParal= jvec+BVec-startIndexBandParal;
			 
			stridedCopyToBlockKernel<<<(BVec+255)/256*localVectorSize, 256>>>(BVec,
											  localVectorSize,
											  eigenVectorsFlattenedCUDA,
											  totalNumberWaveFunctions,
											  cudaFlattenedArrayBlock.begin(),
											  jvec);
			  
			 //
			 //call Chebyshev filtering function only for the current block to be filtered
			 //and does in-place filtering
			 if (jvec+BVec<dftParameters::numAdaptiveFilterStates)
			 {
				const double chebyshevOrd=(double)chebyshevOrder;
				const double adaptiveOrder=0.5*chebyshevOrd
				  +jvec*0.3*chebyshevOrd/dftParameters::numAdaptiveFilterStates;
				linearAlgebraOperationsCUDA::chebyshevFilter(operatorMatrix,
									     cudaFlattenedArrayBlock,
									     YArray,
									     cudaFlattenedFloatArrayBlock,
									     projectorKetTimesVector,
									     localVectorSize,
									     BVec,
									     std::ceil(adaptiveOrder),
									     d_lowerBoundUnWantedSpectrum,
									     upperBoundUnwantedSpectrum,
									     d_lowerBoundWantedSpectrum);
									   
									   
			 }
			 else
			      linearAlgebraOperationsCUDA::chebyshevFilter(operatorMatrix,
									   cudaFlattenedArrayBlock,
									   YArray,
									   cudaFlattenedFloatArrayBlock,
									   projectorKetTimesVector,
									   localVectorSize,
									   BVec,
									   chebyshevOrder,
									   d_lowerBoundUnWantedSpectrum,
									   upperBoundUnwantedSpectrum,
									   d_lowerBoundWantedSpectrum);
									  
			 
		       stridedCopyFromBlockKernel<<<(BVec+255)/256*localVectorSize, 256>>>(BVec,
											   localVectorSize,
											   cudaFlattenedArrayBlock.begin(),
											   totalNumberWaveFunctions,
											   eigenVectorsFlattenedCUDA,
											   jvec);
		}
                else
                {
                      //set to zero wavefunctions which wont go through chebyshev filtering inside a given band group
		      setZeroKernel<<<(BVec+255)/256*localVectorSize, 256>>>(BVec,
		      					                     localVectorSize,
		      							     totalNumberWaveFunctions,
		      							     eigenVectorsFlattenedCUDA,
		      							     jvec);
                }

	    }//block loop

	    hipDeviceSynchronize();
            MPI_Barrier(MPI_COMM_WORLD);
	    gpu_time = MPI_Wtime() - start_time;
	    if (this_process==0 && dftParameters::verbosity>=2)
	        std::cout<<"Time for chebyshev filtering on GPU: "<<gpu_time<<std::endl;


	    if(dftParameters::verbosity >= 4)
	      pcout<<"ChebyShev Filtering Done: "<<std::endl;


            if (numberBandGroups>1)
            {
		    hipDeviceSynchronize();
                    MPI_Barrier(MPI_COMM_WORLD);
		    double band_paral_time=MPI_Wtime();

                    std::vector<double> eigenVectorsFlattened(totalNumberWaveFunctions*localVectorSize,0);

                    //hipDeviceSynchronize();
                    //double copytime=MPI_Wtime();
		    hipMemcpy(&eigenVectorsFlattened[0],
			       eigenVectorsFlattenedCUDA,
			       totalNumberWaveFunctions*localVectorSize*sizeof(double),
			       hipMemcpyDeviceToHost);
                    //hipDeviceSynchronize();
                    //copytime = MPI_Wtime() - copytime;
	            //if (this_process==0)
	            //   std::cout<<"copy time on GPU: "<<copytime<<std::endl;

		    MPI_Barrier(interBandGroupComm);

                    if (true)
                    {
		  	    MPI_Allreduce(MPI_IN_PLACE,
				      &eigenVectorsFlattened[0],
				      totalNumberWaveFunctions*localVectorSize,
				      MPI_DOUBLE,
				      MPI_SUM,
				      interBandGroupComm);
                    }
                    else
                    {
			    std::vector<double> eigenVectorsBandGroup(numVectorsBandParal*localVectorSize,0);
			    std::vector<double> eigenVectorsBandGroupTransposed(numVectorsBandParal*localVectorSize,0);
			    std::vector<double> eigenVectorsTransposed(totalNumberWaveFunctions*localVectorSize,0);

			    for(unsigned int iNode = 0; iNode < localVectorSize; ++iNode)
			       for(unsigned int iWave = 0; iWave < numVectorsBandParal; ++iWave)
				   eigenVectorsBandGroup[iNode*numVectorsBandParal+iWave]
				     = eigenVectorsFlattened[iNode*totalNumberWaveFunctions+startIndexBandParal+iWave];

			    
			    for(unsigned int iNode = 0; iNode < localVectorSize; ++iNode)
			       for(unsigned int iWave = 0; iWave < numVectorsBandParal; ++iWave)
				   eigenVectorsBandGroupTransposed[iWave*localVectorSize+iNode]
				     = eigenVectorsBandGroup[iNode*numVectorsBandParal+iWave];

			    std::vector<int> recvcounts(numberBandGroups,0);
			    std::vector<int> displs(numberBandGroups,0);

			    int recvcount=numVectorsBandParal*localVectorSize;
			    MPI_Allgather(&recvcount,
					  1,
					  MPI_INT,
					  &recvcounts[0],
					  1,
					  MPI_INT,
					  interBandGroupComm);

			    int displ=startIndexBandParal*localVectorSize;
			    MPI_Allgather(&displ,
					  1,
					  MPI_INT,
					  &displs[0],
					  1,
					  MPI_INT,
					  interBandGroupComm);

			    MPI_Allgatherv(&eigenVectorsBandGroupTransposed[0],
					   numVectorsBandParal*localVectorSize,
					   MPI_DOUBLE,
					   &eigenVectorsTransposed[0],
					   &recvcounts[0],
					   &displs[0],
					   dataTypes::mpi_type_id(&eigenVectorsTransposed[0]),
					   interBandGroupComm);

			    
			    for(unsigned int iNode = 0; iNode < localVectorSize; ++iNode)
			       for(unsigned int iWave = 0; iWave < totalNumberWaveFunctions; ++iWave)
				   eigenVectorsFlattened[iNode*totalNumberWaveFunctions+iWave]
				     = eigenVectorsTransposed[iWave*localVectorSize+iNode];
		    }
		    MPI_Barrier(interBandGroupComm);

                    //hipDeviceSynchronize();
                    //copytime=MPI_Wtime();
		    hipMemcpy(eigenVectorsFlattenedCUDA,
			       &eigenVectorsFlattened[0],
			       totalNumberWaveFunctions*localVectorSize*sizeof(double),
			       hipMemcpyHostToDevice);
                    //hipDeviceSynchronize();
                    //copytime = MPI_Wtime() - copytime;
	            //if (this_process==0)
	            //   std::cout<<"copy time on GPU: "<<copytime<<std::endl;
		    hipDeviceSynchronize();
                    MPI_Barrier(MPI_COMM_WORLD);
		    band_paral_time = MPI_Wtime() - band_paral_time;

		    if (this_process==0 && dftParameters::verbosity>=2)
			std::cout<<"Time for band parallelization communication: "<<band_paral_time<<std::endl;

            }

            //if (dftParameters::measureOnlyChebyTime)
            //  exit(0);

	    /*
	    int inc=1;
	    double result=0.0;
	    hipblasDnrm2(cublasHandle,
			flattenedSize,
			eigenVectorsFlattenedCUDA,
			inc, 
			&result);
	    result=result*result;
	    result=dealii::Utilities::MPI::sum(result,operatorMatrix.getMPICommunicator());
	    std::cout<<"l2 norm Chebyshev filtered x: "<<std::sqrt(result)<<std::endl;
	    */

            if (dftParameters::rrGEP==false)
            {
		    if(dftParameters::orthogType.compare("LW") == 0)
		    {

			AssertThrow(false,dealii::ExcMessage("Lowden Gram-Schmidt Orthonormalization Not implemented in CUDA:"));

		    }
		    else if (dftParameters::orthogType.compare("PGS") == 0)
		    {
			//gpu_time = MPI_Wtime();
			linearAlgebraOperationsCUDA::pseudoGramSchmidtOrthogonalization
				     (operatorMatrix,
				      eigenVectorsFlattenedCUDA,
				      localVectorSize,
				      totalNumberWaveFunctions,
				      operatorMatrix.getMPICommunicator(),
                                      interBandGroupComm,
				      cublasHandle,
				      useMixedPrecOverall);
				    
			//gpu_time = MPI_Wtime() - gpu_time;
			//if (this_process==0)
			//    std::cout<<"Time for PGS on GPU: "<<gpu_time<<std::endl;

		    }
		    else if (dftParameters::orthogType.compare("GS") == 0)
		    {

			AssertThrow(false,dealii::ExcMessage("Classical Gram-Schmidt Orthonormalization not implemented in CUDA:"));

		    }

		    if(dftParameters::verbosity >= 4)
		      pcout<<"Orthogonalization Done: "<<std::endl;

	    }

	    //gpu_time = MPI_Wtime();
	    for (unsigned int i=0;i<eigenValues.size();i++)
		   eigenValues[i]=0.0;

    }

    if (eigenValues.size()!=totalNumberWaveFunctions &&  dftParameters::rrGEP==false)
    {
	    linearAlgebraOperationsCUDA::rayleighRitzSpectrumSplitDirect(operatorMatrix,
						      eigenVectorsFlattenedCUDA,
                                                      eigenVectorsRotFracDensityFlattenedCUDA,
						      cudaFlattenedArrayBlock,
						      YArray,
						      projectorKetTimesVector,
						      localVectorSize,
						      totalNumberWaveFunctions,
                                                      totalNumberWaveFunctions-eigenValues.size(),
						      isElpaStep1,
						      isElpaStep2,
						      operatorMatrix.getMPICommunicator(),
						      &eigenValues[0],
						      cublasHandle,
						      projHamPar,
						      processGrid,
                                                      useMixedPrecOverall);

	     if (isElpaStep1)
	     {
		hipDeviceSynchronize();
                MPI_Barrier(MPI_COMM_WORLD);
		gpu_time = MPI_Wtime() - start_time;
		if (this_process==0 && dftParameters::verbosity>=2)
	            std::cout<<"Time for all steps of subspace iteration on GPU till ELPA step 1: "<<gpu_time<<std::endl; 
		return;
	     }

    }
    else
    {
            if (dftParameters::rrGEP==false)
		    linearAlgebraOperationsCUDA::rayleighRitz(operatorMatrix,
							      eigenVectorsFlattenedCUDA,
							      cudaFlattenedArrayBlock,
							      YArray,
							      projectorKetTimesVector,
							      localVectorSize,
							      totalNumberWaveFunctions,
							      isElpaStep1,
							      isElpaStep2,
							      operatorMatrix.getMPICommunicator(),
                                                              interBandGroupComm,
							      &eigenValues[0],
							      cublasHandle,
							      projHamPar,
							      processGrid,
							      useMixedPrecOverall);
            else if (dftParameters::rrGEP)
		    linearAlgebraOperationsCUDA::rayleighRitzGEP(operatorMatrix,
							      eigenVectorsFlattenedCUDA,
							      cudaFlattenedArrayBlock,
							      YArray,
							      projectorKetTimesVector,
							      localVectorSize,
							      totalNumberWaveFunctions,
							      isElpaStep1,
							      isElpaStep2,
							      operatorMatrix.getMPICommunicator(),
                                                              interBandGroupComm,
							      &eigenValues[0],
							      cublasHandle,
							      projHamPar,
                                                              overlapMatPar,
							      processGrid,
							      useMixedPrecOverall);
     

	     if (isElpaStep1)
	     {
		hipDeviceSynchronize();
                MPI_Barrier(MPI_COMM_WORLD);
		gpu_time = MPI_Wtime() - start_time;
		if (this_process==0 && dftParameters::verbosity>=2)
	            std::cout<<"Time for all steps of subspace iteration on GPU till ELPA step 1: "<<gpu_time<<std::endl; 
		return;
	     }

    }
     //gpu_time = MPI_Wtime() - gpu_time;
     //if (this_process==0)
     //    std::cout<<"Time for Rayleigh Ritz on GPU: "<<gpu_time<<std::endl;



    if(dftParameters::verbosity >= 4)
    {
	pcout<<"Rayleigh-Ritz Done: "<<std::endl;
	pcout<<std::endl;
    }

    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time = MPI_Wtime();
    if (eigenValues.size()!=totalNumberWaveFunctions)
      linearAlgebraOperationsCUDA::computeEigenResidualNorm(operatorMatrix,
      						        eigenVectorsRotFracDensityFlattenedCUDA,
						        cudaFlattenedArrayBlock,
						        YArray,
						        projectorKetTimesVector,
						        localVectorSize,
						        eigenValues.size(),
      						        eigenValues,
							operatorMatrix.getMPICommunicator(),
                                                        interBandGroupComm,
                                                        cublasHandle,
      						        residualNorms);
    else
      linearAlgebraOperationsCUDA::computeEigenResidualNorm(operatorMatrix,
      						        eigenVectorsFlattenedCUDA,
						        cudaFlattenedArrayBlock,
						        YArray,
						        projectorKetTimesVector,
						        localVectorSize,
						        totalNumberWaveFunctions,
      						        eigenValues,
							operatorMatrix.getMPICommunicator(),
                                                        interBandGroupComm,
                                                        cublasHandle,
      						        residualNorms,
                                                        true);

    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time = MPI_Wtime() - gpu_time;
    if (this_process==0 && dftParameters::verbosity>=2)
       std::cout<<"Time to compute residual norm: "<<gpu_time<<std::endl;

    //
    //scale the eigenVectors with M^{-1/2} to represent the wavefunctions in the usual FE basis
    //
    scaleCUDAKernel<<<(totalNumberWaveFunctions+255)/256*localVectorSize,256>>>(totalNumberWaveFunctions,
                                                                     localVectorSize,
                                                                     1.0,
                                                                     eigenVectorsFlattenedCUDA,
                                                                     operatorMatrix.getInvSqrtMassVec());

    if (eigenValues.size()!=totalNumberWaveFunctions)
	    scaleCUDAKernel<<<(eigenValues.size()+255)/256*localVectorSize,256>>>(eigenValues.size(),
									     localVectorSize,
									     1.0,
									     eigenVectorsRotFracDensityFlattenedCUDA,
									     operatorMatrix.getInvSqrtMassVec());

    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time = MPI_Wtime() - start_time;

    if (isElpaStep2)
       if (this_process==0 && dftParameters::verbosity>=2)
           std::cout<<"Time for ELPA step 2 on GPU: "<<gpu_time<<std::endl;
    else
       if (this_process==0 && dftParameters::verbosity>=2)
           std::cout<<"Time for all steps of subspace iteration on GPU: "<<gpu_time<<std::endl;
    return;
#endif
  }
}
