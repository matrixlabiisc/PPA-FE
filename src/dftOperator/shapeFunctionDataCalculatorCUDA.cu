#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2020 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE udistribution.
//
// ---------------------------------------------------------------------
//
// @author  Sambit Das
//

namespace shapeFuncCUDA
{
	__global__
		void computeShapeGradNINJIntegralContribution(const unsigned int numQuads,
				const unsigned int numNodesPerElem,
				const unsigned int numElems,
				const double * gradNQuadValuesXI,
				const double * gradNQuadValuesYI,
				const double * gradNQuadValuesZI,
				const double * gradNQuadValuesXJ,
				const double * gradNQuadValuesYJ,
				const double * gradNQuadValuesZJ,
				const double * jxwQuadValues,
				double * shapeGradNINJIntegralContribution)
		{

			const unsigned int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
			const unsigned int numberEntries = numElems*numNodesPerElem*numNodesPerElem*numQuads;

			for(unsigned int index = globalThreadId; index < numberEntries; index+= blockDim.x*gridDim.x)
			{
				const unsigned int blockIndex1 = index/numQuads;
				const unsigned int quadIndex=index-blockIndex1*numQuads;
				const unsigned int blockIndex2=blockIndex1/numNodesPerElem;
				const unsigned int cellId=blockIndex2/numNodesPerElem;
				const unsigned int idJ=cellId*numNodesPerElem*numQuads+(blockIndex1-blockIndex2*numNodesPerElem)*numQuads+quadIndex;
				const unsigned int idI=cellId*numNodesPerElem*numQuads+(blockIndex2-cellId*numNodesPerElem)*numQuads+quadIndex;


				shapeGradNINJIntegralContribution[index]=(gradNQuadValuesXI[idI]*gradNQuadValuesXJ[idJ]+gradNQuadValuesYI[idI]*gradNQuadValuesYJ[idJ]+gradNQuadValuesZI[idI]*gradNQuadValuesZJ[idJ])*jxwQuadValues[cellId*numQuads+quadIndex];
			}

		}

	void computeShapeGradNINJIntegral(hipblasHandle_t &handle,
			const unsigned int numQuads,
			const unsigned int numNodesPerElem,
			const unsigned int numElems,
			const thrust::device_vector<double> & gradNQuadValuesXD,
			const thrust::device_vector<double> & gradNQuadValuesYD,
			const thrust::device_vector<double> & gradNQuadValuesZD,
			const thrust::device_vector<double> & jxwQuadValuesD,
			thrust::device_vector<double> & shapeGradNINJIntegralD)
	{
		shapeGradNINJIntegralD.clear();
		shapeGradNINJIntegralD.resize(numElems*numNodesPerElem*numNodesPerElem,0.0);
		//thrust::device_vector<double> gradPsiQuadValuesXDJ=gradPsiQuadValuesXD;
		//thrust::device_vector<double> gradPsiQuadValuesYDJ=gradPsiQuadValuesYD;
		//thrust::device_vector<double> gradPsiQuadValuesZDJ=gradPsiQuadValuesZD;

		const int blockSize=1;
		const int numberBlocks=numElems/blockSize;
		const int remBlockSize=numElems-numberBlocks*blockSize;

		thrust::device_vector<double> shapeGradNINJIntegralContributionD(blockSize*numNodesPerElem*numNodesPerElem*numQuads,0.0);
		thrust::device_vector<double> onesVecD(numQuads,1.0); 
		for (int iblock=0; iblock<(numberBlocks+1); iblock++)
		{
			const int currentBlockSize= (iblock==numberBlocks)?remBlockSize:blockSize;
			if (currentBlockSize>0)
			{
				const int startingId=iblock*blockSize;

				computeShapeGradNINJIntegralContribution<<<(numQuads+255)/256*numNodesPerElem*numNodesPerElem*currentBlockSize,256>>>
					(numQuads,
					 numNodesPerElem,
					 currentBlockSize,
					 thrust::raw_pointer_cast(&gradNQuadValuesXD[startingId*numNodesPerElem*numQuads]),
					 thrust::raw_pointer_cast(&gradNQuadValuesYD[startingId*numNodesPerElem*numQuads]),
					 thrust::raw_pointer_cast(&gradNQuadValuesZD[startingId*numNodesPerElem*numQuads]),
					 thrust::raw_pointer_cast(&gradNQuadValuesXD[startingId*numNodesPerElem*numQuads]),
					 thrust::raw_pointer_cast(&gradNQuadValuesYD[startingId*numNodesPerElem*numQuads]),
					 thrust::raw_pointer_cast(&gradNQuadValuesZD[startingId*numNodesPerElem*numQuads]),
					 thrust::raw_pointer_cast(&jxwQuadValuesD[startingId*numQuads]),
					 thrust::raw_pointer_cast(&shapeGradNINJIntegralContributionD[0]));

				const double scalarCoeffAlpha = 1.0;
				const double scalarCoeffBeta = 0.0;



				hipblasDgemm(handle,
						HIPBLAS_OP_N,
						HIPBLAS_OP_N,
						1,
						currentBlockSize*numNodesPerElem*numNodesPerElem,
						numQuads,
						&scalarCoeffAlpha,
						thrust::raw_pointer_cast(&onesVecD[0]),
						1,
						thrust::raw_pointer_cast(&shapeGradNINJIntegralContributionD[0]),
						numQuads,
						&scalarCoeffBeta,
						thrust::raw_pointer_cast(&shapeGradNINJIntegralD[startingId*numNodesPerElem*numNodesPerElem]),
						1);
			}
		}
	} 
}

	template<unsigned int FEOrder,unsigned int FEOrderElectro>
void kohnShamDFTOperatorCUDAClass<FEOrder,FEOrderElectro>::preComputeShapeFunctionGradientIntegrals(const unsigned int lpspQuadratureId)
{

	//
	//get FE data
	//
	const unsigned int numberPhysicalCells = dftPtr->matrix_free_data.n_physical_cells();
	const Quadrature<3> &  quadrature=dftPtr->matrix_free_data.get_quadrature(dftPtr->d_densityQuadratureId);
	FEValues<3> fe_values(dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).get_fe(), quadrature, update_values | update_gradients | update_JxW_values);
	const unsigned int numberDofsPerElement = dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).get_fe().dofs_per_cell;
  const unsigned int numberDofsPerElementElectro = dftPtr->d_matrixFreeDataPRefined.get_dof_handler(dftPtr->d_baseDofHandlerIndexElectro).get_fe().dofs_per_cell;
	const unsigned int numberQuadraturePoints = quadrature.size();

	FEValues<3> fe_values_lpsp(dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).get_fe(), dftPtr->matrix_free_data.get_quadrature(lpspQuadratureId), update_values);
	const unsigned int numberQuadraturePointsLpsp = dftPtr->matrix_free_data.get_quadrature(lpspQuadratureId).size();
  d_numQuadPointsLpsp=numberQuadraturePointsLpsp;

	//
	//resize data members
	//
	//d_cellShapeFunctionGradientIntegralFlattened.clear();
	//d_cellShapeFunctionGradientIntegralFlattened.resize(numberPhysicalCells*numberDofsPerElement*numberDofsPerElement);

	d_cellJxWValues.clear();
	d_cellJxWValues.resize(numberPhysicalCells*numberQuadraturePoints);

	d_shapeFunctionValue.resize(numberQuadraturePoints*numberDofsPerElement,0.0);
	d_shapeFunctionValueInverted.resize(numberQuadraturePoints*numberDofsPerElement,0.0);

	d_shapeFunctionGradientValueX.resize(numberPhysicalCells*numberQuadraturePoints*numberDofsPerElement,0.0);
	d_shapeFunctionGradientValueXInverted.resize(numberPhysicalCells*numberQuadraturePoints*numberDofsPerElement,0.0);

	d_shapeFunctionGradientValueY.resize(numberPhysicalCells*numberQuadraturePoints*numberDofsPerElement,0.0);
	d_shapeFunctionGradientValueYInverted.resize(numberPhysicalCells*numberQuadraturePoints*numberDofsPerElement,0.0);

	d_shapeFunctionGradientValueZ.resize(numberPhysicalCells*numberQuadraturePoints*numberDofsPerElement,0.0);
	d_shapeFunctionGradientValueZInverted.resize(numberPhysicalCells*numberQuadraturePoints*numberDofsPerElement,0.0);

  std::vector<double> shapeFunctionValueLpsp(numberQuadraturePointsLpsp*numberDofsPerElement,0.0);
  std::vector<double> shapeFunctionValueInvertedLpsp(numberQuadraturePointsLpsp*numberDofsPerElement,0.0);



	typename dealii::DoFHandler<3>::active_cell_iterator cellPtr=dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).begin_active();
	typename dealii::DoFHandler<3>::active_cell_iterator endcPtr = dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).end();

	unsigned int iElem=0;
	for(; cellPtr!=endcPtr; ++cellPtr)
		if(cellPtr->is_locally_owned())
		{
			fe_values.reinit (cellPtr);

			for(unsigned int q_point = 0; q_point < numberQuadraturePoints; ++q_point)
				d_cellJxWValues[iElem*numberQuadraturePoints+q_point]=fe_values.JxW(q_point); 

			for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
				for(unsigned int q_point = 0; q_point < numberQuadraturePoints; ++q_point)
				{
					const dealii::Tensor<1,3,double> & shape_grad=fe_values.shape_grad(iNode,q_point);

					d_shapeFunctionGradientValueX[iElem*numberDofsPerElement*numberQuadraturePoints
						+iNode*numberQuadraturePoints+q_point]=shape_grad[0];
					d_shapeFunctionGradientValueXInverted[iElem*numberQuadraturePoints*numberDofsPerElement
						+q_point*numberDofsPerElement+iNode]=shape_grad[0];

					d_shapeFunctionGradientValueY[iElem*numberDofsPerElement*numberQuadraturePoints
						+iNode*numberQuadraturePoints+q_point]=shape_grad[1];
					d_shapeFunctionGradientValueYInverted[iElem*numberQuadraturePoints*numberDofsPerElement
						+q_point*numberDofsPerElement+iNode]=shape_grad[1];

					d_shapeFunctionGradientValueZ[iElem*numberDofsPerElement*numberQuadraturePoints
						+iNode*numberQuadraturePoints+q_point]=shape_grad[2];
					d_shapeFunctionGradientValueZInverted[iElem*numberQuadraturePoints*numberDofsPerElement
						+q_point*numberDofsPerElement+iNode]=shape_grad[2];

				}

			if(iElem == 0)
      {
        fe_values_lpsp.reinit(cellPtr);

				for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
					for(unsigned int q_point = 0; q_point < numberQuadraturePoints; ++q_point)
					{
						const double val=fe_values.shape_value(iNode,q_point);
						d_shapeFunctionValue[numberQuadraturePoints*iNode + q_point] = val;
						d_shapeFunctionValueInverted[q_point*numberDofsPerElement+iNode] = val;
					}

				for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
					for(unsigned int q_point = 0; q_point < numberQuadraturePointsLpsp; ++q_point)
          {
						const double val=fe_values_lpsp.shape_value(iNode,q_point);            
						shapeFunctionValueLpsp[numberQuadraturePointsLpsp*iNode + q_point] = val; 
            shapeFunctionValueInvertedLpsp[q_point*numberDofsPerElement+iNode] = val;  
          }
      }

			iElem++;
		}

	d_shapeFunctionValueDevice=d_shapeFunctionValue;
	d_shapeFunctionValueInvertedDevice=d_shapeFunctionValueInverted;

	d_shapeFunctionGradientValueXDevice=d_shapeFunctionGradientValueX;
	d_shapeFunctionGradientValueXInvertedDevice=d_shapeFunctionGradientValueXInverted;

	d_shapeFunctionGradientValueYDevice=d_shapeFunctionGradientValueY;
	d_shapeFunctionGradientValueYInvertedDevice=d_shapeFunctionGradientValueYInverted;

	d_shapeFunctionGradientValueZDevice=d_shapeFunctionGradientValueZ;
	d_shapeFunctionGradientValueZInvertedDevice=d_shapeFunctionGradientValueZInverted;

	d_shapeFunctionValueLpspDevice=shapeFunctionValueLpsp;
	d_shapeFunctionValueInvertedLpspDevice=shapeFunctionValueInvertedLpsp;  

	//d_cellShapeFunctionGradientIntegralFlattenedDevice=d_cellShapeFunctionGradientIntegralFlattened;
	d_cellJxWValuesDevice=d_cellJxWValues;

	hipDeviceSynchronize();
	MPI_Barrier(MPI_COMM_WORLD);
	double gpu_time=MPI_Wtime();


	QGauss<3>  quadraturePlusOne(FEOrder+1);
  unsigned int numberQuadraturePointsPlusOne = quadraturePlusOne.size();  
	FEValues<3> fe_values_plusone(dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).get_fe(), quadraturePlusOne, update_gradients | update_JxW_values);


	std::vector<double> cellJxWValuesQuadPlusOne(numberPhysicalCells*numberQuadraturePointsPlusOne);
	std::vector<double> shapeFunctionGradientValueQuadPlusOneX(numberPhysicalCells*numberQuadraturePointsPlusOne*numberDofsPerElement,0.0);
	std::vector<double> shapeFunctionGradientValueQuadPlusOneY(numberPhysicalCells*numberQuadraturePointsPlusOne*numberDofsPerElement,0.0);
	std::vector<double> shapeFunctionGradientValueQuadPlusOneZ(numberPhysicalCells*numberQuadraturePointsPlusOne*numberDofsPerElement,0.0);


	cellPtr=dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).begin_active();

	iElem=0;
	for(; cellPtr!=endcPtr; ++cellPtr)
		if(cellPtr->is_locally_owned())
		{
			fe_values_plusone.reinit (cellPtr);

			for(unsigned int q_point = 0; q_point < numberQuadraturePointsPlusOne; ++q_point)
				cellJxWValuesQuadPlusOne[iElem*numberQuadraturePointsPlusOne+q_point]=fe_values_plusone.JxW(q_point); 

			for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
				for(unsigned int q_point = 0; q_point < numberQuadraturePointsPlusOne; ++q_point)
				{
					const dealii::Tensor<1,3,double> & shape_grad=fe_values_plusone.shape_grad(iNode,q_point);

					shapeFunctionGradientValueQuadPlusOneX[iElem*numberDofsPerElement*numberQuadraturePointsPlusOne
						+iNode*numberQuadraturePointsPlusOne+q_point]=shape_grad[0];

					shapeFunctionGradientValueQuadPlusOneY[iElem*numberDofsPerElement*numberQuadraturePointsPlusOne
						+iNode*numberQuadraturePointsPlusOne+q_point]=shape_grad[1];

					shapeFunctionGradientValueQuadPlusOneZ[iElem*numberDofsPerElement*numberQuadraturePointsPlusOne
						+iNode*numberQuadraturePointsPlusOne+q_point]=shape_grad[2];
				}

			iElem++;
		}

	thrust::device_vector<double> cellJxWValuesQuadPlusOneDevice=cellJxWValuesQuadPlusOne;
	thrust::device_vector<double> shapeFunctionGradientValueQuadPlusOneXDevice=shapeFunctionGradientValueQuadPlusOneX;
	thrust::device_vector<double> shapeFunctionGradientValueQuadPlusOneYDevice=shapeFunctionGradientValueQuadPlusOneY;
	thrust::device_vector<double> shapeFunctionGradientValueQuadPlusOneZDevice=shapeFunctionGradientValueQuadPlusOneZ;

	shapeFuncCUDA::computeShapeGradNINJIntegral(d_cublasHandle,
			numberQuadraturePointsPlusOne,
			numberDofsPerElement,
			numberPhysicalCells,
			shapeFunctionGradientValueQuadPlusOneXDevice,
			shapeFunctionGradientValueQuadPlusOneYDevice,
			shapeFunctionGradientValueQuadPlusOneZDevice,
			cellJxWValuesQuadPlusOneDevice,
			d_cellShapeFunctionGradientIntegralFlattenedDevice);

	hipDeviceSynchronize();
	MPI_Barrier(MPI_COMM_WORLD);
	gpu_time = MPI_Wtime() - gpu_time;

	if (this_mpi_process==0 && dftParameters::verbosity>=2)
		std::cout<<"Time for shapeFuncCUDA::computeShapeGradNINJIntegral for FEOrder: "<<gpu_time<<std::endl;

  if (FEOrderElectro!=FEOrder)
  {
    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time=MPI_Wtime();


    QGauss<3>  quadratureElectroPlusOne(FEOrderElectro+1);
    numberQuadraturePointsPlusOne = quadratureElectroPlusOne.size();  
    FEValues<3> fe_values_electro_plusone(dftPtr->d_matrixFreeDataPRefined.get_dof_handler(dftPtr->d_baseDofHandlerIndexElectro).get_fe(), quadratureElectroPlusOne, update_gradients | update_JxW_values);


    cellJxWValuesQuadPlusOne.resize(numberPhysicalCells*numberQuadraturePointsPlusOne);
    shapeFunctionGradientValueQuadPlusOneX.resize(numberPhysicalCells*numberQuadraturePointsPlusOne*numberDofsPerElementElectro,0.0);
    shapeFunctionGradientValueQuadPlusOneY.resize(numberPhysicalCells*numberQuadraturePointsPlusOne*numberDofsPerElementElectro,0.0);
    shapeFunctionGradientValueQuadPlusOneZ.resize(numberPhysicalCells*numberQuadraturePointsPlusOne*numberDofsPerElementElectro,0.0);


    typename dealii::DoFHandler<3>::active_cell_iterator cellPtrElectro=dftPtr->d_matrixFreeDataPRefined.get_dof_handler(dftPtr->d_baseDofHandlerIndexElectro).begin_active();
    typename dealii::DoFHandler<3>::active_cell_iterator endcPtrElectro = dftPtr->d_matrixFreeDataPRefined.get_dof_handler(dftPtr->d_baseDofHandlerIndexElectro).end();

    iElem=0;
    for(; cellPtrElectro!=endcPtrElectro; ++cellPtrElectro)
      if(cellPtrElectro->is_locally_owned())
      {
        fe_values_electro_plusone.reinit (cellPtrElectro);

        for(unsigned int q_point = 0; q_point < numberQuadraturePointsPlusOne; ++q_point)
          cellJxWValuesQuadPlusOne[iElem*numberQuadraturePointsPlusOne+q_point]=fe_values_electro_plusone.JxW(q_point); 

        for(unsigned int iNode = 0; iNode < numberDofsPerElementElectro; ++iNode)
          for(unsigned int q_point = 0; q_point < numberQuadraturePointsPlusOne; ++q_point)
          {
            const dealii::Tensor<1,3,double> & shape_grad=fe_values_electro_plusone.shape_grad(iNode,q_point);

            shapeFunctionGradientValueQuadPlusOneX[iElem*numberDofsPerElementElectro*numberQuadraturePointsPlusOne
              +iNode*numberQuadraturePointsPlusOne+q_point]=shape_grad[0];

            shapeFunctionGradientValueQuadPlusOneY[iElem*numberDofsPerElementElectro*numberQuadraturePointsPlusOne
              +iNode*numberQuadraturePointsPlusOne+q_point]=shape_grad[1];

            shapeFunctionGradientValueQuadPlusOneZ[iElem*numberDofsPerElementElectro*numberQuadraturePointsPlusOne
              +iNode*numberQuadraturePointsPlusOne+q_point]=shape_grad[2];
          }

        iElem++;
      }

    cellJxWValuesQuadPlusOneDevice=cellJxWValuesQuadPlusOne;
    shapeFunctionGradientValueQuadPlusOneXDevice=shapeFunctionGradientValueQuadPlusOneX;
    shapeFunctionGradientValueQuadPlusOneYDevice=shapeFunctionGradientValueQuadPlusOneY;
    shapeFunctionGradientValueQuadPlusOneZDevice=shapeFunctionGradientValueQuadPlusOneZ;

    shapeFuncCUDA::computeShapeGradNINJIntegral(d_cublasHandle,
        numberQuadraturePointsPlusOne,
        numberDofsPerElementElectro,
        numberPhysicalCells,
        shapeFunctionGradientValueQuadPlusOneXDevice,
        shapeFunctionGradientValueQuadPlusOneYDevice,
        shapeFunctionGradientValueQuadPlusOneZDevice,
        cellJxWValuesQuadPlusOneDevice,
        d_cellShapeFunctionGradientIntegralFlattenedDeviceElectro);

    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    gpu_time = MPI_Wtime() - gpu_time;

    if (this_mpi_process==0 && dftParameters::verbosity>=2)
      std::cout<<"Time for shapeFuncCUDA::computeShapeGradNINJIntegral for FEOrderElectro: "<<gpu_time<<std::endl;
  } 

  QGaussLobatto<3>  quadratureGl(C_rhoNodalPolyOrder<FEOrder,FEOrderElectro>()+1);
  FEValues<3> fe_valuesGl(dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).get_fe(), quadratureGl, update_values | update_gradients);
  const unsigned int numberQuadraturePointsGl = quadratureGl.size();

  //
  //resize data members
  //
  std::vector<double> glShapeFunctionValueInverted(numberQuadraturePointsGl*numberDofsPerElement,0.0);

  std::vector<double> glShapeFunctionGradientValueXInverted(numberPhysicalCells*numberQuadraturePointsGl*numberDofsPerElement,0.0);

  std::vector<double> glShapeFunctionGradientValueYInverted(numberPhysicalCells*numberQuadraturePointsGl*numberDofsPerElement,0.0);

  std::vector<double> glShapeFunctionGradientValueZInverted(numberPhysicalCells*numberQuadraturePointsGl*numberDofsPerElement,0.0);


  cellPtr=dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).begin_active();
  endcPtr = dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).end();

  iElem=0;
  for(; cellPtr!=endcPtr; ++cellPtr)
    if(cellPtr->is_locally_owned())
    {
      fe_valuesGl.reinit (cellPtr);


      for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
        for(unsigned int q_point = 0; q_point < numberQuadraturePointsGl; ++q_point)
        {
          const dealii::Tensor<1,3,double> & shape_grad=fe_values.shape_grad(iNode,q_point);

          glShapeFunctionGradientValueXInverted[iElem*numberQuadraturePointsGl*numberDofsPerElement
            +q_point*numberDofsPerElement+iNode]=shape_grad[0];

          glShapeFunctionGradientValueYInverted[iElem*numberQuadraturePointsGl*numberDofsPerElement
            +q_point*numberDofsPerElement+iNode]=shape_grad[1];

          glShapeFunctionGradientValueZInverted[iElem*numberQuadraturePointsGl*numberDofsPerElement
            +q_point*numberDofsPerElement+iNode]=shape_grad[2];

        }

      if(iElem == 0)
        for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
          for(unsigned int q_point = 0; q_point < numberQuadraturePointsGl; ++q_point)
          {
            const double val=fe_valuesGl.shape_value(iNode,q_point);
            glShapeFunctionValueInverted[q_point*numberDofsPerElement+iNode] = val;
          }


      iElem++;
    }

  d_glShapeFunctionValueInvertedDevice=glShapeFunctionValueInverted;

  d_glShapeFunctionGradientValueXInvertedDevice=glShapeFunctionGradientValueXInverted;

  d_glShapeFunctionGradientValueYInvertedDevice=glShapeFunctionGradientValueYInverted;

  d_glShapeFunctionGradientValueZInvertedDevice=glShapeFunctionGradientValueZInverted;

	if (dftParameters::useHigherQuadNLP)
	{
		//QGauss<3>  quadratureNLP(C_num1DQuadNLPSP<FEOrder>());
    QIterated<3> quadratureNLP(QGauss<1>(C_num1DQuadNLPSP<FEOrder>()),C_numCopies1DQuadNLPSP());
		FEValues<3> fe_valuesNLP(dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).get_fe(), quadratureNLP, update_values|update_gradients);
		const unsigned int numberQuadraturePointsNLP = quadratureNLP.size();

		//
		//resize data members
		//
		std::vector<double> nlpShapeFunctionValueInverted(numberQuadraturePointsNLP*numberDofsPerElement,0.0);
		std::vector<double> shapeFunctionGradientValueNLPXInverted(numberPhysicalCells*numberQuadraturePointsNLP*numberDofsPerElement,0.0);
		std::vector<double> shapeFunctionGradientValueNLPYInverted(numberPhysicalCells*numberQuadraturePointsNLP*numberDofsPerElement,0.0);
		std::vector<double> shapeFunctionGradientValueNLPZInverted(numberPhysicalCells*numberQuadraturePointsNLP*numberDofsPerElement,0.0);    

		cellPtr=dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).begin_active();
		endcPtr = dftPtr->matrix_free_data.get_dof_handler(dftPtr->d_densityDofHandlerIndex).end();


    iElem=0;
		for(; cellPtr!=endcPtr; ++cellPtr)
			if(cellPtr->is_locally_owned())
			{ 
				fe_valuesNLP.reinit (cellPtr);

        for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
          for(unsigned int q_point = 0; q_point < numberQuadraturePointsNLP; ++q_point)
          {
            const dealii::Tensor<1,3,double> & shape_grad=fe_valuesNLP.shape_grad(iNode,q_point);

            shapeFunctionGradientValueNLPXInverted[iElem*numberQuadraturePointsNLP*numberDofsPerElement
              +q_point*numberDofsPerElement+iNode]=shape_grad[0];

            shapeFunctionGradientValueNLPYInverted[iElem*numberQuadraturePointsNLP*numberDofsPerElement
              +q_point*numberDofsPerElement+iNode]=shape_grad[1];

            shapeFunctionGradientValueNLPZInverted[iElem*numberQuadraturePointsNLP*numberDofsPerElement
              +q_point*numberDofsPerElement+iNode]=shape_grad[2];

          }

        if (iElem==0)
          for(unsigned int iNode = 0; iNode < numberDofsPerElement; ++iNode)
            for(unsigned int q_point = 0; q_point < numberQuadraturePointsNLP; ++q_point)
            {
              const double val=fe_valuesNLP.shape_value(iNode,q_point);
              nlpShapeFunctionValueInverted[q_point*numberDofsPerElement+iNode] = val;
            }      

        iElem++;
			}

		d_shapeFunctionValueNLPInvertedDevice=nlpShapeFunctionValueInverted;
	  d_shapeFunctionGradientValueNLPXInvertedDevice=shapeFunctionGradientValueNLPXInverted;
    d_shapeFunctionGradientValueNLPYInvertedDevice=shapeFunctionGradientValueNLPYInverted;
    d_shapeFunctionGradientValueNLPZInvertedDevice=shapeFunctionGradientValueNLPZInverted;    
	}
}
