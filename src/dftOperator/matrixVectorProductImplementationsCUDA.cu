// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2018 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Phani Motamarri, Sambit Das
//


/** @file matrixVectorProductImplementations.cc
 *  @brief Contains linear algebra operations
 *
 */


template<unsigned int FEOrder>
void kohnShamDFTOperatorCUDAClass<FEOrder>::computeLocalHamiltonianTimesXMF (const dealii::MatrixFree<3,double>  &data,
							   std::vector<vectorType>  &dst,
							   const std::vector<vectorType>  &src,
							   const std::pair<unsigned int,unsigned int> &cell_range) const
{
  VectorizedArray<double>  half = make_vectorized_array(0.5);
  VectorizedArray<double>  two = make_vectorized_array(2.0);


#ifdef USE_COMPLEX
  int kPointIndex = d_kPointIndex;
  FEEvaluation<3,FEOrder,C_num1DQuad<FEOrder>(), 2, double>  fe_eval(data, dftPtr->eigenDofHandlerIndex, 0);
  Tensor<1,2,VectorizedArray<double> > psiVal, vEffTerm, kSquareTerm, kDotGradientPsiTerm, derExchWithSigmaTimesGradRhoDotGradientPsiTerm;
  Tensor<1,2,Tensor<1,3,VectorizedArray<double> > > gradientPsiVal, gradientPsiTerm, derExchWithSigmaTimesGradRhoTimesPsi,sumGradientTerms;

  Tensor<1,3,VectorizedArray<double> > kPointCoors;
  kPointCoors[0] = make_vectorized_array(dftPtr->d_kPointCoordinates[3*kPointIndex+0]);
  kPointCoors[1] = make_vectorized_array(dftPtr->d_kPointCoordinates[3*kPointIndex+1]);
  kPointCoors[2] = make_vectorized_array(dftPtr->d_kPointCoordinates[3*kPointIndex+2]);

  double kSquareTimesHalf =  0.5*(dftPtr->d_kPointCoordinates[3*kPointIndex+0]*dftPtr->d_kPointCoordinates[3*kPointIndex+0] + dftPtr->d_kPointCoordinates[3*kPointIndex+1]*dftPtr->d_kPointCoordinates[3*kPointIndex+1] + dftPtr->d_kPointCoordinates[3*kPointIndex+2]*dftPtr->d_kPointCoordinates[3*kPointIndex+2]);
  VectorizedArray<double> halfkSquare = make_vectorized_array(kSquareTimesHalf);

  if(dftParameters::xc_id == 4)
    {
      for(unsigned int cell=cell_range.first; cell<cell_range.second; ++cell)
	{
	  fe_eval.reinit (cell);
	  for(unsigned int i = 0; i < dst.size(); ++i)
	    {
	      fe_eval.read_dof_values(src[i]);
	      fe_eval.evaluate (true,true,false);
	      for(unsigned int q = 0; q < fe_eval.n_q_points; ++q)
		{
		  //
		  //get the quadrature point values of psi and gradPsi which are complex
		  //
		  psiVal = fe_eval.get_value(q);
		  gradientPsiVal = fe_eval.get_gradient(q);

		  //
		  //compute gradientPsiTerm of the stiffnessMatrix times vector (0.5*gradientPsi)
		  //
		  gradientPsiTerm[0] = gradientPsiVal[0]*half;
		  gradientPsiTerm[1] = gradientPsiVal[1]*half;

		  //
		  //compute Veff part of the stiffness matrix times vector (Veff*psi)
		  //
		  vEffTerm[0] = psiVal[0]*vEff(cell,q);
		  vEffTerm[1] = psiVal[1]*vEff(cell,q);

		  //
		  //compute term involving dot product of k-vector and gradientPsi in stiffnessmatrix times vector
		  //
		  kDotGradientPsiTerm[0] = kPointCoors[0]*gradientPsiVal[1][0] + kPointCoors[1]*gradientPsiVal[1][1] + kPointCoors[2]*gradientPsiVal[1][2];
		  kDotGradientPsiTerm[1] = -(kPointCoors[0]*gradientPsiVal[0][0] + kPointCoors[1]*gradientPsiVal[0][1] + kPointCoors[2]*gradientPsiVal[0][2]);


		  derExchWithSigmaTimesGradRhoDotGradientPsiTerm[0] = two*(derExcWithSigmaTimesGradRho(cell,q)[0]*gradientPsiVal[0][0] + derExcWithSigmaTimesGradRho(cell,q)[1]*gradientPsiVal[0][1] + derExcWithSigmaTimesGradRho(cell,q)[2]*gradientPsiVal[0][2]);
		  derExchWithSigmaTimesGradRhoDotGradientPsiTerm[1] = two*(derExcWithSigmaTimesGradRho(cell,q)[0]*gradientPsiVal[1][0] + derExcWithSigmaTimesGradRho(cell,q)[1]*gradientPsiVal[1][1] + derExcWithSigmaTimesGradRho(cell,q)[2]*gradientPsiVal[1][2]);
		  //
		  //see if you can make this shorter
		  //
		  derExchWithSigmaTimesGradRhoTimesPsi[0][0] = two*derExcWithSigmaTimesGradRho(cell,q)[0]*psiVal[0];
		  derExchWithSigmaTimesGradRhoTimesPsi[0][1] = two*derExcWithSigmaTimesGradRho(cell,q)[1]*psiVal[0];
		  derExchWithSigmaTimesGradRhoTimesPsi[0][2] = two*derExcWithSigmaTimesGradRho(cell,q)[2]*psiVal[0];
		  derExchWithSigmaTimesGradRhoTimesPsi[1][0] = two*derExcWithSigmaTimesGradRho(cell,q)[0]*psiVal[1];
		  derExchWithSigmaTimesGradRhoTimesPsi[1][1] = two*derExcWithSigmaTimesGradRho(cell,q)[1]*psiVal[1];
		  derExchWithSigmaTimesGradRhoTimesPsi[1][2] = two*derExcWithSigmaTimesGradRho(cell,q)[2]*psiVal[1];


		  //
		  //compute kSquareTerm
		  //
		  kSquareTerm[0] = halfkSquare*psiVal[0];
		  kSquareTerm[1] = halfkSquare*psiVal[1];

		  //
		  //submit gradients and values
		  //

		  for(int i = 0; i < 3; ++i)
		    {
		      sumGradientTerms[0][i] = gradientPsiTerm[0][i] + derExchWithSigmaTimesGradRhoTimesPsi[0][i];
		      sumGradientTerms[1][i] = gradientPsiTerm[1][i] + derExchWithSigmaTimesGradRhoTimesPsi[1][i];
		    }

		  fe_eval.submit_gradient(sumGradientTerms,q);
		  fe_eval.submit_value(vEffTerm+kDotGradientPsiTerm+kSquareTerm+derExchWithSigmaTimesGradRhoDotGradientPsiTerm,q);

		}

	      fe_eval.integrate (true, true);
	      fe_eval.distribute_local_to_global (dst[i]);

	    }
	}
    }
  else
    {
      for(unsigned int cell=cell_range.first; cell<cell_range.second; ++cell)
	{
	  fe_eval.reinit (cell);
	  for(unsigned int i = 0; i < dst.size(); ++i)
	    {
	      fe_eval.read_dof_values(src[i]);
	      fe_eval.evaluate (true,true,false);
	      for(unsigned int q = 0; q < fe_eval.n_q_points; ++q)
		{
		  //
		  //get the quadrature point values of psi and gradPsi which are complex
		  //
		  psiVal = fe_eval.get_value(q);
		  gradientPsiVal = fe_eval.get_gradient(q);

		  //
		  //compute gradientPsiTerm of the stiffnessMatrix times vector (0.5*gradientPsi)
		  //
		  gradientPsiTerm[0] = gradientPsiVal[0]*half;
		  gradientPsiTerm[1] = gradientPsiVal[1]*half;

		  //
		  //compute Veff part of the stiffness matrix times vector (Veff*psi)
		  //
		  vEffTerm[0] = psiVal[0]*vEff(cell,q);
		  vEffTerm[1] = psiVal[1]*vEff(cell,q);

		  //
		  //compute term involving dot product of k-vector and gradientPsi in stiffnessmatrix times vector
		  //
		  kDotGradientPsiTerm[0] = kPointCoors[0]*gradientPsiVal[1][0] + kPointCoors[1]*gradientPsiVal[1][1] + kPointCoors[2]*gradientPsiVal[1][2];
		  kDotGradientPsiTerm[1] = -(kPointCoors[0]*gradientPsiVal[0][0] + kPointCoors[1]*gradientPsiVal[0][1] + kPointCoors[2]*gradientPsiVal[0][2]);

		  //
		  //compute kSquareTerm
		  //
		  kSquareTerm[0] = halfkSquare*psiVal[0];
		  kSquareTerm[1] = halfkSquare*psiVal[1];

		  //
		  //submit gradients and values
		  //
		  fe_eval.submit_gradient(gradientPsiTerm,q);
		  fe_eval.submit_value(vEffTerm+kDotGradientPsiTerm+kSquareTerm,q);
		}

	      fe_eval.integrate (true, true);
	      fe_eval.distribute_local_to_global (dst[i]);

	    }
	}

    }
#else
  FEEvaluation<3,FEOrder, C_num1DQuad<FEOrder>(), 1, double>  fe_eval(data, dftPtr->eigenDofHandlerIndex, 0);
  Tensor<1,3,VectorizedArray<double> > derExchWithSigmaTimesGradRhoTimesPsi,gradientPsiVal;
  VectorizedArray<double> psiVal,derExchWithSigmaTimesGradRhoDotGradientPsiTerm;
  if(dftParameters::xc_id == 4)
    {
      for(unsigned int cell = cell_range.first; cell < cell_range.second; ++cell)
	{
	  fe_eval.reinit (cell);
	  for(unsigned int i = 0; i < dst.size(); i++)
	    {
	      fe_eval.read_dof_values(src[i]);
	      fe_eval.evaluate (true,true,false);
	      for(unsigned int q = 0; q < fe_eval.n_q_points; ++q)
		{
		  psiVal = fe_eval.get_value(q);
		  gradientPsiVal = fe_eval.get_gradient(q);
		  derExchWithSigmaTimesGradRhoTimesPsi[0] = derExcWithSigmaTimesGradRho(cell,q)[0]*psiVal;
		  derExchWithSigmaTimesGradRhoTimesPsi[1] = derExcWithSigmaTimesGradRho(cell,q)[1]*psiVal;
		  derExchWithSigmaTimesGradRhoTimesPsi[2] = derExcWithSigmaTimesGradRho(cell,q)[2]*psiVal;
		  derExchWithSigmaTimesGradRhoDotGradientPsiTerm = scalar_product(derExcWithSigmaTimesGradRho(cell,q),gradientPsiVal);

		  //
		  //submit gradient and value
		  //
		  fe_eval.submit_gradient(gradientPsiVal*half + two*derExchWithSigmaTimesGradRhoTimesPsi,q);
		  fe_eval.submit_value(vEff(cell,q)*psiVal + two*derExchWithSigmaTimesGradRhoDotGradientPsiTerm,q);
		}

	      fe_eval.integrate (true, true);
	      fe_eval.distribute_local_to_global (dst[i]);
	    }
	}
    }
  else
    {
      for(unsigned int cell = cell_range.first; cell < cell_range.second; ++cell)
	{
	  fe_eval.reinit (cell);
	  for(unsigned int i = 0; i < dst.size(); i++)
	    {
	      fe_eval.read_dof_values(src[i]);
	      fe_eval.evaluate (true,true,false);
	      for(unsigned int q = 0; q < fe_eval.n_q_points; ++q)
		{
		  fe_eval.submit_gradient(fe_eval.get_gradient(q)*half, q);
		  fe_eval.submit_value(fe_eval.get_value(q)*vEff(cell,q), q);
		}

	      fe_eval.integrate (true, true);
	      fe_eval.distribute_local_to_global(dst[i]);
	    }
	}

    }
#endif
}



template<unsigned int FEOrder>
void kohnShamDFTOperatorCUDAClass<FEOrder>::computeLocalHamiltonianTimesX(const double* src,
							                  const unsigned int numberWaveFunctions,
							                  double* dst,
                                                                          const bool skipAccumulationBoundaryNodes) 
{

  const unsigned int totalLocallyOwnedCells = dftPtr->matrix_free_data.n_physical_cells();  
  
  copyCUDAKernel<<<(numberWaveFunctions+255)/256*totalLocallyOwnedCells*d_numberNodesPerElement,256>>>(numberWaveFunctions, 
												       totalLocallyOwnedCells*d_numberNodesPerElement,
												       src,
												       thrust::raw_pointer_cast(&d_cellWaveFunctionMatrix[0]),
												       thrust::raw_pointer_cast(&d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));


  const double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
  const unsigned int strideA = d_numberNodesPerElement*numberWaveFunctions;
  const unsigned int strideB = d_numberNodesPerElement*d_numberNodesPerElement; 
  const unsigned int strideC = d_numberNodesPerElement*numberWaveFunctions;


  hipblasDgemmStridedBatched(d_cublasHandle,
			    HIPBLAS_OP_N,
			    HIPBLAS_OP_N,
			    numberWaveFunctions,
			    d_numberNodesPerElement,
			    d_numberNodesPerElement,
			    &scalarCoeffAlpha,
			    thrust::raw_pointer_cast(&d_cellWaveFunctionMatrix[0]),
			    numberWaveFunctions,
			    strideA,
			    thrust::raw_pointer_cast(&d_cellHamiltonianMatrixFlattenedDevice[0]),
			    d_numberNodesPerElement,
			    strideB,
			    &scalarCoeffBeta,
			    thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0]),
			    numberWaveFunctions,
			    strideC,
			    totalLocallyOwnedCells);


  if(!(dftParameters::isPseudopotential && dftPtr->d_nonLocalAtomGlobalChargeIds.size() > 0))
  {
         if (skipAccumulationBoundaryNodes)
            daxpyAtomicAddKernelNonBoundary<<<(numberWaveFunctions+255)/256*d_numLocallyOwnedCells*d_numberNodesPerElement,256>>>
                                                                     (numberWaveFunctions,
                                                                      d_numLocallyOwnedCells*d_numberNodesPerElement,
                                                                      thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0]),
                                                                      thrust::raw_pointer_cast(&d_boundaryIdsVecDevice[0]),
                                                                      dst,
                                                                      thrust::raw_pointer_cast(&d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));
         else
            daxpyAtomicAddKernel<<<(numberWaveFunctions+255)/256*d_numLocallyOwnedCells*d_numberNodesPerElement,256>>>
                                                                     (numberWaveFunctions,
                                                                      d_numLocallyOwnedCells*d_numberNodesPerElement,
                                                                      thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0]),
                                                                      dst,
                                                                      thrust::raw_pointer_cast(&d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));

  }
    
  
}


template<unsigned int FEOrder>
void kohnShamDFTOperatorCUDAClass<FEOrder>::computeLocalHamiltonianTimesX(const dealii::LinearAlgebra::distributed::Vector<double> & src,
							              const unsigned int numberWaveFunctions,
							              dealii::LinearAlgebra::distributed::Vector<double> & dst) const
{


  //
  //element level matrix-vector multiplications
  //
  const char transA = 'N',transB = 'N';
  const double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
  const unsigned int inc = 1;
  const unsigned int totalLocallyOwnedCells = dftPtr->matrix_free_data.n_physical_cells();   

  std::vector<double> cellWaveFunctionMatrix(totalLocallyOwnedCells*d_numberNodesPerElement*numberWaveFunctions,0.0);
  std::vector<double> cellHamMatrixTimesWaveMatrix(totalLocallyOwnedCells*d_numberNodesPerElement*numberWaveFunctions,0.0);

  for(unsigned int iCell = 0; iCell < totalLocallyOwnedCells; ++iCell)
    {
      for(unsigned int iNode = 0; iNode < d_numberNodesPerElement; ++iNode)
        {
          dealii::types::global_dof_index localNodeId = d_flattenedArrayMacroCellLocalProcIndexIdMapFlattened[iCell*d_numberNodesPerElement + iNode];
          dcopy_(&numberWaveFunctions,
      	         src.begin()+localNodeId,
		 &inc,
                 &cellWaveFunctionMatrix[0]+iCell*d_numberNodesPerElement*numberWaveFunctions+iNode*numberWaveFunctions,
		 &inc);
	}
    }


   for(unsigned int iCell = 0; iCell < totalLocallyOwnedCells; ++iCell)
   { 
      dgemm_(&transA,
             &transB,
             &numberWaveFunctions,
             &d_numberNodesPerElement,
             &d_numberNodesPerElement,
             &scalarCoeffAlpha,
             &cellWaveFunctionMatrix[iCell*d_numberNodesPerElement*numberWaveFunctions],
             &numberWaveFunctions,
             &d_cellHamiltonianMatrixFlattened[iCell*d_numberNodesPerElement*d_numberNodesPerElement],
             &d_numberNodesPerElement,
             &scalarCoeffBeta,
             &cellHamMatrixTimesWaveMatrix[iCell*d_numberNodesPerElement*numberWaveFunctions],
             &numberWaveFunctions);
   }
  

 for(unsigned int iCell = 0; iCell < totalLocallyOwnedCells; ++iCell)
 {
   for(unsigned int iNode = 0; iNode < d_numberNodesPerElement; ++iNode)
   {   
      dealii::types::global_dof_index localNodeId = d_flattenedArrayMacroCellLocalProcIndexIdMapFlattened[iCell*d_numberNodesPerElement + iNode];

      daxpy_(&numberWaveFunctions,
             &scalarCoeffAlpha,
             &cellHamMatrixTimesWaveMatrix[0]+iCell*d_numberNodesPerElement*numberWaveFunctions + iNode*numberWaveFunctions,
             &inc,
             dst.begin()+localNodeId,
             &inc);

   }

 } 

}


