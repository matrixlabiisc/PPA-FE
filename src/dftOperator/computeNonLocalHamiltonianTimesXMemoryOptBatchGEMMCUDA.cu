#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2018 The Regents of the University of Michigan and DFT-FE authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das
//


template<unsigned int FEOrder>
void kohnShamDFTOperatorCUDAClass<FEOrder>::computeNonLocalHamiltonianTimesX(const double* src,
									     cudaVectorType &  projectorKetTimesVector,
									     const unsigned int numberWaveFunctions,
									     double* dst,
const bool skip1,
const bool skip2)

{
	 
  const double scalarCoeffAlpha = 1.0,scalarCoeffBeta = 0.0;
  
  //
  //compute C^{T}*X
  //
  unsigned int strideA = numberWaveFunctions*d_numberNodesPerElement;
  unsigned int strideB = d_numberNodesPerElement*d_maxSingleAtomPseudoWfc; 
  unsigned int strideC = numberWaveFunctions*d_maxSingleAtomPseudoWfc;

  if (d_totalNonlocalElems>0 && !skip1)
  { 
	  copyCUDAKernel<<<(numberWaveFunctions+255)/256*d_totalNonlocalElems*d_numberNodesPerElement,256>>>
									     (numberWaveFunctions, 
									      d_totalNonlocalElems*d_numberNodesPerElement,
									      src,
									      thrust::raw_pointer_cast(&d_cellWaveFunctionMatrixNonLocalDevice[0]),
									      thrust::raw_pointer_cast(&d_flattenedArrayCellLocalProcIndexIdFlattenedMapNonLocalDevice[0]));
	 
	 
	  
	  hipblasDgemmStridedBatched(d_cublasHandle,
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    numberWaveFunctions,
				    d_maxSingleAtomPseudoWfc,
				    d_numberNodesPerElement,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&d_cellWaveFunctionMatrixNonLocalDevice[0]),
				    numberWaveFunctions,
				    strideA,
				    thrust::raw_pointer_cast(&d_cellHamiltonianMatrixNonLocalFlattenedDevice[0]),
				    d_numberNodesPerElement,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&d_projectorKetTimesVectorAllCellsDevice[0]),
				    numberWaveFunctions,
				    strideC,
				    d_totalNonlocalElems);

	  hipblasDgemm(d_cublasHandle,
		      HIPBLAS_OP_N,
		      HIPBLAS_OP_N,
		      numberWaveFunctions,
		      d_totalPseudoWfcNonLocal,
		      d_totalNonlocalElems*d_maxSingleAtomPseudoWfc,
		      &scalarCoeffAlpha,
		      thrust::raw_pointer_cast(&d_projectorKetTimesVectorAllCellsDevice[0]),
		      numberWaveFunctions,
		      thrust::raw_pointer_cast(&d_projectorKetTimesVectorAllCellsReductionDevice[0]),
		      d_totalNonlocalElems*d_maxSingleAtomPseudoWfc,
		      &scalarCoeffBeta,
		      thrust::raw_pointer_cast(&d_projectorKetTimesVectorParFlattenedDevice[0]),
		      numberWaveFunctions);

  }

  if (skip2)
   return;

  projectorKetTimesVector=0.0;

  //std::cout<<"nonlocal l2 norm: "<<d_projectorKetTimesVectorDealiiParFlattenedDevice.l2_norm()<<std::endl;
  
  if (d_totalNonlocalElems>0)
    copyToDealiiParallelNonLocalVec<<<(numberWaveFunctions+255)/256*d_totalPseudoWfcNonLocal,256>>>
						     (numberWaveFunctions, 
						      d_totalPseudoWfcNonLocal,
                                                      thrust::raw_pointer_cast(&d_projectorKetTimesVectorParFlattenedDevice[0]),
                                                      projectorKetTimesVector.begin(),
						      thrust::raw_pointer_cast(&d_projectorIdsParallelNumberingMapDevice[0]));

  
    projectorKetTimesVector.compress(VectorOperation::add);
    projectorKetTimesVector.update_ghost_values();
  
  //std::cout<<"nonlocal l2 norm: "<<projectorKetTimesVector.l2_norm()<<std::endl;

  if (d_totalNonlocalElems>0) 
  {
	  //
	  //compute V*C^{T}*X
	  //
	  scaleCUDAKernel<<<(numberWaveFunctions+255)/256*d_totalPseudoWfcNonLocal,256>>>(numberWaveFunctions,
										 d_totalPseudoWfcNonLocal,
										 1.0,
										 projectorKetTimesVector.begin(),
										 thrust::raw_pointer_cast(&d_nonLocalPseudoPotentialConstantsDevice[0]));

	  copyFromParallelNonLocalVecToAllCellsVec<<<(numberWaveFunctions+255)/256*d_totalNonlocalElems*d_maxSingleAtomPseudoWfc,256>>>
							     (numberWaveFunctions, 
							      d_totalNonlocalElems,
							      d_maxSingleAtomPseudoWfc,
							      projectorKetTimesVector.begin(),
							      thrust::raw_pointer_cast(&d_projectorKetTimesVectorAllCellsDevice[0]),
							      thrust::raw_pointer_cast(&d_indexMapFromPaddedNonLocalVecToParallelNonLocalVecDevice[0]));
	   
	  //
	  //compute C*V*C^{T}*x
	  //

	  strideA = numberWaveFunctions*d_maxSingleAtomPseudoWfc;
	  strideB = d_maxSingleAtomPseudoWfc*d_numberNodesPerElement; 
	  strideC = numberWaveFunctions*d_numberNodesPerElement;
	  hipblasDgemmStridedBatched(d_cublasHandle,
				    HIPBLAS_OP_N,
				    HIPBLAS_OP_N,
				    numberWaveFunctions,
				    d_numberNodesPerElement,
				    d_maxSingleAtomPseudoWfc,
				    &scalarCoeffAlpha,
				    thrust::raw_pointer_cast(&d_projectorKetTimesVectorAllCellsDevice[0]),
				    numberWaveFunctions,
				    strideA,
				    thrust::raw_pointer_cast(&d_cellHamiltonianMatrixNonLocalFlattenedTransposeDevice[0]),
				    d_maxSingleAtomPseudoWfc,
				    strideB,
				    &scalarCoeffBeta,
				    thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrixNonLocalDevice[0]),
				    numberWaveFunctions,
				    strideC,
				    d_totalNonlocalElems);
	 
	   
	  for(unsigned int iAtom = 0; iAtom < d_totalNonlocalAtomsCurrentProc; ++iAtom)
	  {
		  const unsigned int accum= d_numberCellsAccumNonLocalAtoms[iAtom];
		  addNonLocalContributionCUDAKernel<<<(numberWaveFunctions+255)/256*d_numberCellsNonLocalAtoms[iAtom]*d_numberNodesPerElement,256>>>
										     (numberWaveFunctions, 
										      d_numberCellsNonLocalAtoms[iAtom]*d_numberNodesPerElement,
										      thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrixNonLocalDevice[0])
										+accum*d_numberNodesPerElement*numberWaveFunctions,
										      thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0]),
										      thrust::raw_pointer_cast(&d_cellNodeIdMapNonLocalToLocalDevice[0])
										  +accum*d_numberNodesPerElement);
		  
	  }
	  
  }   

  daxpyAtomicAddKernel<<<(numberWaveFunctions+255)/256*d_numLocallyOwnedCells*d_numberNodesPerElement,256>>>
                                                                     (numberWaveFunctions,
                                                                      d_numLocallyOwnedCells*d_numberNodesPerElement,
                                                                      thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0]),
                                                                      dst,
                                                                      thrust::raw_pointer_cast(&d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));

   
   //std::cout<<"dst norm: "<<dst.l2_norm()<<std::endl;
}
